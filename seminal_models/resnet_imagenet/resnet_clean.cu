#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hip/hip_runtime.h>
#include <stdint.h>

#include "resnet_clean.h"

#define SM_COUNT 82
#define WARP_PER_SM 4
#define THREAD_PER_WARP 32
#define MAX_THREAD_PER_BLOCK 1024
#define TILE_WIDTH 32
#define BLOCK_ROWS 8
#define CUDA_BATCH_SIZE 32
#define MAX_SHARED_MEMORY 48000
#define MAX_SHARED_MEM_FLOATS 12000
#define MAX_THREAD_PER_BLOCK_INCL_REG 512
#define BATCH_NORM_DERIV_REDUCTION_THREADS 256




// used to hide all print statements for device data
#define TO_PRINT false

#define CUDA_CALL(x) do { if((x)!=hipSuccess) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__);\
    return EXIT_FAILURE;}} while(0)
#define CURAND_CALL(x) do { if((x) != HIPRAND_STATUS_SUCCESS) { \
	printf("Error at %s:%d\n",__FILE__,__LINE__);\
	return EXIT_FAILURE;}} while(0)


/* DECLARING FUNCTIONS HERE */
void testConvolution(int in_spatial_dim, int kern_dim, int in_filters, int out_filters,  int stride, int batch_size, 
																float * input, float * weights, float * biases, float * output);


/* START OF KERNELS/FUNCTIONS */

__global__ void setVal(int size, float val, float *out){
 	int ind = blockDim.x * blockIdx.x + threadIdx.x;
 	if (ind >= size){
 		return;
 	}
 	out[ind] = val;
}

void init_weights_gaussian_device(hiprandGenerator_t * gen, int size, float *X, float mean, float var){
 	float stddev = sqrtf(var);
 	hiprandStatus_t status = hiprandGenerateNormal(*gen, X, (size_t) size, mean, stddev);
 }

// RANDOM NUMBER GENERATOR ON DEVICE CAN'T USE C LIBRARY RAND(), so use cuRAND() library instead...
// __global__ void sample_gaussian(int size, float *X, float mean, float var) {
// 	int i = blockIdx.x * blockDim.x + threadIdx.x;
// 	if (i >= size){
// 		return;
// 	}
// 	if (var == 0){
// 		X[i] = mean;
// 		return;
// 	}
// 	float x = (float)rand() / RAND_MAX;
//   	float y = (float)rand() / RAND_MAX;
//   	float z = sqrtf(-2 * logf(x)) * cosf(2 * M_PI * y);
//   	float std = sqrtf(var);
//   	float val = std * z + mean;
//   	X[i] = val;
// }

// ASSUME 1-D launch
__global__ void addVec(int size, float * A, float * B, float * out){
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= size){
		return;
	}
	out[i] = A[i] + B[i];
}

// GRID has dim (ROWS / TILE_WIDTH, COLS/TILE_WIDTH)
// each BLOCK has dim (TILE_WIDTH, TILE_WIDTH)
__global__ void matMulOptimized(const float *M, const float *N, int m, int k, int n, float *out){
	__shared__ float M_tile[TILE_WIDTH][TILE_WIDTH + 1];
	__shared__ float N_tile[TILE_WIDTH][TILE_WIDTH + 1];

	int block_x = blockIdx.x;
	int block_y = blockIdx.y;

	int thread_x = threadIdx.x;
	int thread_y = threadIdx.y;

	int row_ind = block_x * TILE_WIDTH + thread_x;
	int col_ind = block_y * TILE_WIDTH + thread_y;

	if (row_ind >= m || col_ind >= n){
		return;
	}

	float val = 0;
	for (int phase = 0; phase < ceil((float) k / float(TILE_WIDTH)); phase++) {
		if (phase * TILE_WIDTH + thread_y < k){
			M_tile[thread_x][thread_y] = M[row_ind * k + phase * TILE_WIDTH + thread_y];
		}
		else{
			M_tile[thread_x][thread_y] = 0;
		}
		if (phase * TILE_WIDTH + thread_x < k){
			N_tile[thread_x][thread_y] = N[(phase * TILE_WIDTH + thread_x) * n + col_ind];
		}
		else{
			N_tile[thread_x][thread_y] = 0;
		}

		__syncthreads();

		for (int t = 0; t < TILE_WIDTH; t++){
			val += M_tile[thread_x][t] * N_tile[t][thread_y];
		}
		__syncthreads();
	}
	out[row_ind * n + col_ind] = val;
}


// GRID has dim (ROWS / TILE_WIDTH, COLS/TILE_WIDTH)
// each BLOCK has dim (TILE_WIDTH, TILE_WIDTH)
__global__ void matMul(const float *M, const float *N, int m, int k, int n, float *out){

	
	int row_ind = blockIdx.x * TILE_WIDTH + threadIdx.x;
	int col_ind = blockIdx.y * TILE_WIDTH + threadIdx.y;

	if (row_ind >= m || col_ind >= n){
		return;
	}

	float val = 0;
	for (int z = 0; z < k; z++){
		val += M[row_ind * k + z] * N[z * n + col_ind];
	}
	out[row_ind * n + col_ind] = val;
}


// unoptimized transpose because used rarely...

// grid has dim (ROWS / TILE_WIDTH, COLS/TILE_WIDTH)
// each BLOCK has dim (TILE_WIDTH , BLOCK_ROWS) = # of threads
__global__ void transposeSharedMem(const float *in, int rows, int cols, float * out) {
  __shared__ float tile[TILE_WIDTH][TILE_WIDTH + 1];

  int row_ind = blockIdx.x * TILE_WIDTH + threadIdx.y;
  int col_ind = blockIdx.y * TILE_WIDTH + threadIdx.x;
  
  
  if (col_ind >= cols || row_ind >= rows){
  	return;
  }

  
  // each thread needs to load TILE_WIDTH / BLOCK_ROWS values
  int row_boundary = min(TILE_WIDTH, rows - row_ind);
  for (int j = 0; j < row_boundary; j += BLOCK_ROWS){
     tile[threadIdx.y + j][threadIdx.x] = in[(row_ind+j)*cols + col_ind];
  }

  __syncthreads();

  int col_boundary = min(TILE_WIDTH, cols - col_ind);
  for (int j = 0; j < col_boundary; j += BLOCK_ROWS){
     out[col_ind*rows + row_ind + j] = tile[threadIdx.y + j][threadIdx.x];
  }
}

// grid has dim (ROWS / TILE_WIDTH, COLS/TILE_WIDTH)
// each BLOCK has dim (TILE_WIDTH , TILE_WIDTH) = # of threads
__global__ void transpose(const float *in, int rows, int cols, float * out) {

  int row_ind = blockIdx.x * TILE_WIDTH + threadIdx.x;
  int col_ind = blockIdx.y * TILE_WIDTH + threadIdx.y;
  
  
  if (col_ind >= cols || row_ind >= rows){
  	return;
  }

  out[col_ind * rows + row_ind] = in[row_ind * cols + col_ind];
}


// TRIED TO OPTIMIZE: MAKE SURE THIS WORKS WITH SHARED MEM AND LAUNCH SPECS (FORGOT HOW I DID IT...)
// 48KB is maximum value for shared memory, passed into this kernel as third param <<< gridDim, blockDim, SHARED_MEM_BYTES >>>
// launch grid dimensions as (OUT_SPATIAL_DIM, OUT_SPATIAL_DIM, OUT_FILTER_CHUNK) blocks, and launch with block dim as (out_filt_rows_shared, sub_batch) threads
// thus 12k floats is max for shared memory per block
// first get as many output filter weights in shared memory as possible, but have separate blocks working on different chunks (OUT_FILTER_CHUNK * out_filt_rows_shared = out_filt)
// then stream samples in batch to compute output value for each sample and output filter. Eac sub_batch will have batch_size / dim(sub_batch) samples to go over
// __global__ void doConvolutionOptimized(const float * input, const float * weights, const float * biases, int spatial_dim, int kern_dim, int in_filters, int out_filters, int stride, int batch_size, float * out){

// 	// will consist of (shared_out_filt_rows X (kern_dim^2 * in_filt) conv_weight matrix
// 	extern __shared__ float shared_mem[];


// 	// (Calling "Kernel" a 3-D obj of weights where there is 2-D conv filter for each input channel)
// 	int kernel_size = (kern_dim * kern_dim * in_filters);

// 	int spatial_row_start = stride * blockIdx.x;
// 	int spatial_col_start = stride * blockIdx.y;
// 	int out_spatial_dim = spatial_dim / stride;

// 	int half_kernel_dim = kern_dim / 2;
// 	int out_filter_id, spatial_row, spatial_col;
// 	float out_val, spatial_val;
// 	out_filter_id = blockIdx.z * blockDim.x + threadIdx.x;
// 	if (out_filter_id >= out_filters){
// 		return;
// 	}

// 	for (int j = 0; j < kernel_size; j++){
// 		shared_mem[threadIdx.x * kernel_size + j] = weights[out_filter_id * kernel_size + j];
// 	}

// 	int samp_per_subbatch = ceil((float) batch_size / blockDim.y);
// 	int samp_start = samp_per_subbatch * threadIdx.y;
// 	int samp_end = min(batch_size, samp_start + samp_per_subbatch);
// 	int kernel_ind;
// 	// probably could be more efficient by reducing number of output filters in shared mem, and adding tiled spatial....
// 	for (int sample_ind = samp_start; sample_ind < samp_end; sample_ind++){
// 		out_val = 0;
// 		for (int row_offset = -half_kernel_dim; row_offset <= half_kernel_dim; row_offset++){
// 			for (int col_offset = -half_kernel_dim; col_offset <= half_kernel_dim; col_offset++){
// 				for (int channel = 0; channel < in_filters; channel++){
						
// 					// compute spatial value
// 					spatial_row = spatial_row_start + row_offset;
// 					spatial_col = spatial_col_start + col_offset;
// 					kernel_ind = kern_dim * in_filters * (row_offset + half_kernel_dim) + in_filters * (col_offset + half_kernel_dim) + channel;
// 					if ((spatial_row < 0) || (spatial_row >= spatial_dim) || (spatial_col < 0) || (spatial_col >= spatial_dim)) {
// 						spatial_val = 0;
// 					}
// 					else{
// 						spatial_val = input[spatial_dim * spatial_dim * in_filters * sample_ind + spatial_dim * in_filters * spatial_row + in_filters * spatial_col + channel];
// 					}

// 					// multiply with conv weight
// 					// threadIdx.x specifies the output filter id
// 					// kernel_ind specifies the (x, y, input_channel)
// 					out_val += shared_mem[threadIdx.x * kernel_size + kernel_ind] * spatial_val;
// 				}
// 			}
// 		}
// 		out[out_spatial_dim * out_spatial_dim * out_filters * sample_ind + out_spatial_dim * out_filters * blockIdx.x + out_filters * blockIdx.y + out_filter_id] = out_val + biases[out_filter_id];
// 	}
// }


// FOR NOW KEEP NAIVE (UN-OPTIMIZED)...
// not bothering with shared memory for now...

// Independent over (output_filter_id, output_spatial_row, output_spatial_col, sample)
// Launch with gridDim (out_spatial_dim, out_spatial_dim, max(1, out_filters / (MAX_THREAD_PER_BLOCK)) and blockDim (batch_size, min(MAX_THREAD_PER_BLOCK / batch_size, output_filters))
// Room to optimize a lot...
__global__ void doConvolution(const float * input, const float * weights, int spatial_dim, int kern_dim, int in_filters, int out_filters, int stride, int batch_size, float * out){

	int out_spatial_row = blockIdx.x;
	int out_spatial_col = blockIdx.y;
	int out_filter_id = blockIdx.z * blockDim.y + threadIdx.y;
	int sample_ind = threadIdx.x;
	int out_spatial_dim = spatial_dim / stride;

	// shoudn't need to check based on launch specs but will anyways
	if ((out_filter_id >= out_filters) || (sample_ind >= batch_size) || (out_spatial_row >= out_spatial_dim) || (out_spatial_col >= out_spatial_dim)) {
		return;
	}

	int in_spatial_row_start = stride * out_spatial_row;
	int in_spatial_col_start = stride * out_spatial_col;

	int half_kernel_dim = kern_dim / 2;
	int in_spatial_row, in_spatial_col, kernel_ind;
	
	// (Calling "Kernel" a 3-D obj of weights where there is 2-D conv filter for each input channel)
	int kernel_size = (kern_dim * kern_dim * in_filters);

	float out_val = 0;
	float in_spatial_val, norm_val;
	for (int row_offset = -half_kernel_dim; row_offset <= half_kernel_dim; row_offset++){
		for (int col_offset = -half_kernel_dim; col_offset <= half_kernel_dim; col_offset++){
			for (int in_channel = 0; in_channel < in_filters; in_channel++){
						
				// compute spatial value
				in_spatial_row = in_spatial_row_start + row_offset;
				in_spatial_col = in_spatial_col_start + col_offset;
				kernel_ind = kern_dim * in_filters * (row_offset + half_kernel_dim) + in_filters * (col_offset + half_kernel_dim) + in_channel;
				if ((in_spatial_row < 0) || (in_spatial_row >= spatial_dim) || (in_spatial_col < 0) || (in_spatial_col >= spatial_dim)) {
					in_spatial_val = 0;
				}
				else{
					in_spatial_val = input[spatial_dim * spatial_dim * in_filters * sample_ind + spatial_dim * in_filters * in_spatial_row + in_filters * in_spatial_col + in_channel];
				}

				// multiply with conv weight
				// threadIdx.x specifies the output filter id
				// kernel_ind specifies the (x, y, input_channel)
				out_val += weights[out_filter_id * kernel_size + kernel_ind] * in_spatial_val;
			}
		}
	}
	//out[out_spatial_dim * out_spatial_dim * out_filters * sample_ind + out_spatial_dim * out_filters * out_spatial_row + out_filters * out_spatial_col + out_filter_id] = out_val + biases[out_filter_id];
	out[out_spatial_dim * out_spatial_dim * out_filters * sample_ind + out_spatial_dim * out_filters * out_spatial_row + out_filters * out_spatial_col + out_filter_id] = out_val;
}

// FOR NOW KEEP NAIVE (UN-OPTIMIZED)...
// not bothering with shared memory for now...

// Independent over (input filter, input_x, input_y, sample)
// could use shared memory over conv weights...
// Launch with gridDim (in_spatial_dim, in_spatial_dim, max(1, input_filters / (MAX_THREAD_PER_BLOCK / batch_size))) and blockDim (batch_size, min(MAX_THREAD_PER_BLOCK / batch_size, input_filters))
// Can parallelize further with reductions, if want to optimize


__global__ void convolutionDerivInput(const float * input, const float * weights, const float * out_deriv, int spatial_dim, int kern_dim, int in_filters, int out_filters, int stride, int batch_size, 
										bool toAdd, float * input_deriv){

	int spatial_row = blockIdx.x;
	int spatial_col = blockIdx.y;
	int in_filter_id = blockIdx.z * blockDim.y + threadIdx.y;
	int sample_ind = threadIdx.x;
	// shouldn't need to check based on launch specs, but will anyways...
	if ((spatial_row >= spatial_dim) || (spatial_col >= spatial_dim) || (in_filter_id >= in_filters) || (sample_ind >= batch_size)){
		return;
	}

	int out_spatial_dim = spatial_dim / stride;
	int half_kernel_dim = kern_dim / 2;
	int out_spatial_row_start = spatial_row / stride;
	int out_spatial_col_start = spatial_col / stride;
	int kern_ind, kern_row_ind, kern_col_ind, out_spatial_ind, out_spatial_row, out_spatial_col;
	int kernel_size = (kern_dim * kern_dim * in_filters);
	float out_spatial_val_deriv;
	float total_deriv = 0;
	for (int out_filt_id = 0; out_filt_id < out_filters; out_filt_id++){
		for (int row_offset = -half_kernel_dim; row_offset <= half_kernel_dim; row_offset++){
			for (int col_offset = -half_kernel_dim; col_offset <= half_kernel_dim; col_offset++){
				// compute output spatial value that used the input spatial value
				out_spatial_row = out_spatial_row_start + row_offset;
				out_spatial_col = out_spatial_col_start + col_offset;
				// index of output spatial val (iterate over samples in batch, then rows, then columns, then channels)
				out_spatial_ind = out_spatial_dim * out_spatial_dim * out_filters * sample_ind + out_spatial_dim * out_filters * out_spatial_row + out_filters * out_spatial_col + out_filt_id;

				// get kernel index used to generate out spatial value for corresponding input spatial value
				kern_row_ind = spatial_row - out_spatial_row * stride + half_kernel_dim;
				kern_col_ind = spatial_col - out_spatial_col * stride + half_kernel_dim;
				kern_ind = kern_dim * in_filters * kern_row_ind + in_filters * kern_col_ind + in_filter_id;
				if ((kern_row_ind < 0) || (kern_row_ind >= kern_dim) || (kern_col_ind < 0) || (kern_col_ind >= kern_dim) ||
						(out_spatial_row < 0) || (out_spatial_row >= out_spatial_dim) || (out_spatial_col < 0) || (out_spatial_col >= out_spatial_dim)) {
					out_spatial_val_deriv = 0;
				}
				else{
					out_spatial_val_deriv = weights[out_filt_id * kernel_size + kern_ind] * out_deriv[out_spatial_ind];
				}
				total_deriv += out_spatial_val_deriv;
			}
		}
	}
	int input_spatial_ind = spatial_dim * spatial_dim * in_filters * sample_ind + spatial_dim * in_filters * spatial_row + in_filters * spatial_col + in_filter_id;
	// used because normal backprop + residual adds to deriv
	if (toAdd){
		input_deriv[input_spatial_ind] += total_deriv;
	}
	else{
		input_deriv[input_spatial_ind] = total_deriv;
	}
}


// // Launch with gridDim (input_filters, n_partials, batch_size) and blockDim (MAX_THREAD_PER_BLOCK_INCL_REG)

// NOT WORKING! :(

// __global__ void convolutionDerivInputPartialOptimized(const float * input, const float * weights, const float * out_deriv, int spatial_dim, int kern_dim, int in_filters, int out_filters, int total_weights, int stride, int batch_size, 
// 												int n_load_output_filters_per_thread, int n_output_filters_per_partial, int n_partials, float * partial_input_deriv){


// 	// for now sharing weights associates to a given input filter
// 	__shared__ float shared_weights[MAX_SHARED_MEM_FLOATS];


// 	// originally treat launch specs to bring in shared weights 

// 	int in_filter_id = blockIdx.x;

// 	// thread id represents (out_filter, kern_x, and kern_y)
// 	// assuming the thread indicies represents [out_filter_0: (0, 0), (0, 1), ... (kern_dim - 1, kern_dim - 1), out_filter_1: (0, 0), (0, 1), ... (kern_dim - 1, kern_dim - 1)]
// 	int thread_id = threadIdx.x;
	

// 	// blockIdx.y represents how many partial sums there will be
// 	// each parital sum computes up derivs from (blockDim.x / (kern_dim * kern_dim)) output filters
// 	int partial_id = blockIdx.y;
	
// 	// going from custom thread index scheme to semantic meaning
// 	int kern_col = thread_id % kern_dim;
// 	int kern_row = (thread_id / kern_dim) % kern_dim;

// 	int orig_out_filters_per_partial = blockDim.x / (kern_dim * kern_dim);

// 	int out_filter_start_id = (thread_id / (kern_dim * kern_dim)) + partial_id * n_output_filters_per_partial;

// 	// getting the kern ind represented in the global weight matrix
// 	int kern_ind = kern_dim * in_filters * kern_row + in_filters * kern_col + in_filter_id;

// 	int kernel_size = (kern_dim * kern_dim * in_filters);
// 	int weight_ind;

// 	for (int i = 0; i < n_load_output_filters_per_thread; i++){
// 		weight_ind = (out_filter_start_id + i * orig_out_filters_per_partial) * kernel_size + kern_ind;
// 		// if there is actually a weight to store in shared
// 		if (weight_ind < total_weights){
// 			shared_weights[thread_id + i * orig_out_filters_per_partial * kern_dim * kern_dim] = weights[weight_ind];
// 		}
// 	}

// 	__syncthreads();


// 	// now treat threads to just do ordinary work using the shared weights (independent over samples in batch & spatial dim)
	
	
// 	int start_out_filter = n_output_filters_per_partial * partial_id;

// 	int out_spatial_dim = spatial_dim / stride;
// 	int half_kernel_dim = kern_dim / 2;

// 	int spatial_row, spatial_col, in_spatial_ind, out_spatial_row, out_spatial_col, out_spatial_ind, kern_row_ind, kern_col_ind, kern_shared_mem_ind;

// 	float partial_deriv = 0;

// 	int spatial_row_col_ind = thread_id;
// 	int spatial_row_col_total = spatial_dim * spatial_dim;

// 	int sample_ind = blockIdx.z;

// 	while (spatial_row_col_ind < spatial_row_col_total){

// 		spatial_col = spatial_row_col_ind % spatial_dim;
// 		spatial_row = (spatial_row_col_ind / spatial_dim) % spatial_dim;

// 		partial_deriv = 0;

// 		for (int cur_out_filter_id = start_out_filter; cur_out_filter_id < start_out_filter + n_output_filters_per_partial && cur_out_filter_id < out_filters; cur_out_filter_id++){
// 			for (int row_offset = -half_kernel_dim; row_offset <= half_kernel_dim; row_offset++){
// 				for (int col_offset = -half_kernel_dim; col_offset <= half_kernel_dim; col_offset++){
					
// 					// compute output spatial value that used the input spatial value
// 					out_spatial_row = spatial_row / stride + row_offset;
// 					out_spatial_col = spatial_col / stride + col_offset;

// 					// get kernel index used to generate out spatial value for corresponding input spatial value
// 					kern_row_ind = spatial_row - out_spatial_row * stride + half_kernel_dim;
// 					kern_col_ind = spatial_col - out_spatial_col * stride + half_kernel_dim;

// 					if ((kern_row_ind < 0) || (kern_row_ind >= kern_dim) || (kern_col_ind < 0) || (kern_col_ind >= kern_dim) ||
// 						(out_spatial_row < 0) || (out_spatial_row >= out_spatial_dim) || (out_spatial_col < 0) || (out_spatial_col >= out_spatial_dim)) {
// 							continue;
// 					}
					
// 					// index of output spatial val (iterate over samples in batch, then rows, then columns, then channels)
// 					out_spatial_ind = out_spatial_dim * out_spatial_dim * out_filters * sample_ind + out_spatial_dim * out_filters * out_spatial_row + out_filters * out_spatial_col + cur_out_filter_id;

// 					// getting the weight using custom indexing scheme from kernel launch spec / shared mem storage
// 					kern_shared_mem_ind = (cur_out_filter_id % n_output_filters_per_partial) * (kern_dim * kern_dim) + kern_dim * kern_row_ind + kern_col_ind;

// 					partial_deriv += shared_weights[kern_shared_mem_ind] * out_deriv[out_spatial_ind];
					
// 				}
// 			}
// 		}

// 		in_spatial_ind = spatial_dim * spatial_dim * in_filters * sample_ind + spatial_dim * in_filters * spatial_row + in_filters * spatial_col + in_filter_id;

// 		partial_input_deriv[n_partials * in_spatial_ind + partial_id] = partial_deriv;

// 		// skip over all the other threads running in parallel working on other chunks of spatial/batch combos
// 		spatial_row_col_ind += blockDim.x;
// 	}
// }


// // Launch with Full Parallel: <<< ceil((float) input_size / MAX_THREAD_PER_BLOCK), MAX_THREAD_PER_BLOCK >>>
// __global__ void finalizeRecutionConvolutionInputDerivOptimized(int input_size, int n_partials, const float * partial_input_deriv, float * input_deriv, bool toAdd){

// 	int index = blockIdx.x * blockDim.x + threadIdx.x;

// 	if (index >= input_size){
// 		return;
// 	}

// 	float total_deriv = 0;
// 	for (int k = 0; k < n_partials; k++){
// 		total_deriv += partial_input_deriv[n_partials * index + k];
// 	}

// 	if (toAdd){
// 		input_deriv[index] += total_deriv;
// 	}
// 	else{
// 		input_deriv[index] = total_deriv;
// 	}

// }

// FOR NOW KEEP NAIVE (UN-OPTIMIZED)...
// not bothering with shared memory for now...

// Independent over (input filter, output filter, kern_x, kern_x)
// could use shared memory over input values...
// Launch with gridDim (kern_dim, kern_dim, output_filters) and blockDim (input_filters) [if input_filters > MAX_THREAD_PER_BLOCK switch ordering of input_filters and output_filters in launch]
__global__ void convolutionDerivWeights(const float * input, const float * weights, const float * out_deriv, int spatial_dim, int kern_dim, int in_filters, int out_filters, int stride, int batch_size,
															float * weight_deriv, bool is_block_dim_inp){

	int in_filter_id;
	int out_filter_id;
	if (is_block_dim_inp){
		in_filter_id = threadIdx.x;
		out_filter_id = blockIdx.z;
	}
	else{
		in_filter_id = blockIdx.z;
		out_filter_id = threadIdx.x;
	}
	int kern_row = blockIdx.x;
	int kern_col = blockIdx.y;

	// shouldn't need to check based on launch specs, but will anyways...
	if ((in_filter_id >= in_filters) || (out_filter_id >= out_filters) || (kern_row >= kern_dim) || (kern_col >= kern_dim)){
		return;
	}

	int kern_ind = kern_dim * in_filters * kern_row + in_filters * kern_col + in_filter_id;

	int kernel_size = (kern_dim * kern_dim * in_filters);
	int half_kernel_dim = kern_dim / 2;
	int out_spatial_dim = spatial_dim / stride;
	int in_spatial_row, in_spatial_col, in_spatial_ind, out_spatial_ind;
	float out_spatial_val_deriv = 0;
	float total_deriv = 0;
	for (int s = 0; s < batch_size; s++){
		for (int out_row = 0; out_row < out_spatial_dim; out_row++){
			for (int out_col = 0; out_col < out_spatial_dim; out_col++){

				// given out_row, out_col, kern_row, kern_col => get the input value used to generate output
				in_spatial_row = stride * out_row + kern_row - half_kernel_dim;
				in_spatial_col = stride * out_col + kern_col - half_kernel_dim;

				// accounting for input filter and sample in batch get index into the input values
				in_spatial_ind = spatial_dim * spatial_dim * in_filters * s + spatial_dim * in_filters * in_spatial_row + in_filters * in_spatial_col + in_filter_id;

				// going from sample, out_row, out_col, out_filter to get index into out_deriv values
				out_spatial_ind = out_spatial_dim * out_spatial_dim * out_filters * s + out_spatial_dim * out_filters * out_row + out_filters * out_col + out_filter_id;

				if ((in_spatial_row < 0) || (in_spatial_row >= spatial_dim) || (in_spatial_col < 0) || (in_spatial_col >= spatial_dim)){
					out_spatial_val_deriv = 0;
				}
				else{
					out_spatial_val_deriv = input[in_spatial_ind] * out_deriv[out_spatial_ind];
				}
				total_deriv += out_spatial_val_deriv;
			}
		}
	}
	weight_deriv[kernel_size * out_filter_id + kern_ind] = total_deriv;
}



// iterating over each filter separately
// launch with (OUTFILTERS) grid dim and thread dim of 1 (could easily parallelize menas + vars, with reduction, but save for later..)
// could also use shared memory here if want to be faster
// input is the output of convolution
// ASSUME reLU activation function
__global__ void getMeansAndVars(const float * input, int spatial_dim, int filters, int batch_size, float * means, float * vars){

	int filter_id = blockIdx.x * blockDim.x + threadIdx.x;
	if (filter_id >= filters){
		return;
	}

	float mean, var;
	float sum = 0;
	for (int s = 0; s < batch_size; s++){
		for (int i = 0; i < spatial_dim; i++){
			for (int j = 0; j < spatial_dim; j++){
				sum += input[spatial_dim * spatial_dim * filters * s + spatial_dim * filters * i + filters * j + filter_id];
			}
		}
	}

	mean = sum / (batch_size * spatial_dim * spatial_dim);
	means[filter_id] = mean;

	float var_sum = 0;
	int inp_index;
	for (int s = 0; s < batch_size; s++){
		for (int i = 0; i < spatial_dim; i++){
			for (int j = 0; j < spatial_dim; j++){
				inp_index = spatial_dim * spatial_dim * filters * s + spatial_dim * filters * i + filters * j + filter_id;
				var_sum += (input[inp_index] - mean) * (input[inp_index] - mean);
			}
		}
	}

	var = var_sum / (batch_size * spatial_dim * spatial_dim);
	vars[filter_id] = var;
}

// Launch with gridDim = batch_size * filters * spatial_dim * spatial_dim / MAX_THREADS and blockDim = MAX_THREADS
__global__ void doRecomputeBatchNormAndActivate(int size, const float * input, const float * gamma, const float * beta,
								int spatial_dim, int filters, int batch_size, float eps, float * means, float * vars, float * out, bool to_activate){

	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= size){
		return;
	}
	int filter_id = index % filters;

	float mean, var;
	mean = means[filter_id];
	var = vars[filter_id];
	float normalized_temp_val, normalized_val;
	normalized_temp_val = (input[index] - mean) / sqrtf(var + eps);
	normalized_val = gamma[filter_id] * normalized_temp_val + beta[filter_id];
	if (to_activate){
		out[index] = fmaxf(normalized_val, 0); 
	}
	else{
		out[index] = normalized_val;
	}
}

// iterating over each filter separately
// launch with (OUTFILTERS) grid dim and thread dim of 1 (could easily parallelize menas + vars, with reduction, but save for later..)
// could also use shared memory here if want to be faster
// input is the output of convolution
// ASSUME reLU activation function
// __global__ void activationAndBatchNormDerivOld(const float * input, const float * gamma, const float * beta, 
// 									int spatial_dim, int filters, int batch_size, float eps, const float * means, const float * vars, const float * activated,
// 									const float * out_layer_deriv, float * normalized_temp_deriv, float * gamma_deriv, float * beta_deriv, float * input_deriv, bool to_activate_deriv){
	
	
// 	int filter_id = blockIdx.x * blockDim.x + threadIdx.x;
// 	if (filter_id >= filters){
// 		return;
// 	}

// 	float n_samples = batch_size * spatial_dim * spatial_dim;
// 	float gamma_val = gamma[filter_id];
// 	float mean_val = means[filter_id];
// 	float var_val = vars[filter_id];

// 	// first compute dL/activated (relu deriv) and then dL/dNormalized_Temp (== x hat)
// 	// also can compute dL/dGamma and dL/dBeta (parameters of batch norm)
// 	int index;
// 	float dGamma = 0;
// 	float dBeta = 0;
// 	float activated_val, out_layer_deriv_val, normalized_temp_val;
// 	for (int s = 0; s < batch_size; s++){
// 		for (int i = 0; i < spatial_dim; i++){
// 			for (int j = 0; j < spatial_dim; j++){
// 				index = spatial_dim * spatial_dim * filters * s + spatial_dim * filters * i + filters * j + filter_id;
// 				activated_val = activated[index];
// 				if (to_activate_deriv && (activated_val <= 0)) {
// 					normalized_temp_deriv[index] = 0;
// 				}
// 				else{
// 					out_layer_deriv_val = out_layer_deriv[index];
// 					normalized_temp_val = (input[index] - mean_val) / sqrtf(var_val + eps);
// 					normalized_temp_deriv[index] = out_layer_deriv_val * gamma_val;
// 					dGamma += out_layer_deriv_val * normalized_temp_val;
// 					dBeta += out_layer_deriv_val;
// 				}
// 			}
// 		}
// 	}

// 	// save down dGamma and dBeta so optimzer can update parameters
// 	gamma_deriv[filter_id] = dGamma;
// 	beta_deriv[filter_id] = dBeta;

// 	// compute dL/dVar and most of dL/dMean
// 	float dVar = 0;
// 	float dMean = 0;
// 	float partial_var_deriv = 0; 
// 	float norm_temp_deriv_val;
// 	float filt_var_three_halfs_power = -0.5 * powf(var_val + eps, -1.5);
// 	float neg_filt_var_recip_sqrt = -1.0 / sqrtf(var_val + eps);
// 	for (int s = 0; s < batch_size; s++){
// 		for (int i = 0; i < spatial_dim; i++){
// 			for (int j = 0; j < spatial_dim; j++){
// 				index = spatial_dim * spatial_dim * filters * s + spatial_dim * filters * i + filters * j + filter_id;
// 				norm_temp_deriv_val = normalized_temp_deriv[index];
// 				dVar += norm_temp_deriv_val * (input[index] - mean_val) * filt_var_three_halfs_power;
// 				dMean += norm_temp_deriv_val * neg_filt_var_recip_sqrt;
// 				partial_var_deriv += -2 * (input[index] - mean_val);
// 			}
// 		}
// 	}

// 	// finish off dL/dMean
// 	dMean += dVar * partial_var_deriv / n_samples;

// 	// compute dL/dX (aka w.r.t. to input to batch norm which is typically the output of a conv)
// 	// saving input_deriv so backprop can continue to previous layer
// 	for (int s = 0; s < batch_size; s++){
// 		for (int i = 0; i < spatial_dim; i++){
// 			for (int j = 0; j < spatial_dim; j++){
// 				index = spatial_dim * spatial_dim * filters * s + spatial_dim * filters * i + filters * j + filter_id;
// 				input_deriv[index] = normalized_temp_deriv[index] * (-1 * neg_filt_var_recip_sqrt) + dVar * (2 * (input[index] - mean_val)) / n_samples + dMean / n_samples;
// 			}
// 		}
// 	}
// }


__device__ void warpReduce(volatile float * shared_mem_data, int thread_id){
	shared_mem_data[thread_id] += shared_mem_data[thread_id + 32];
	shared_mem_data[thread_id] += shared_mem_data[thread_id + 16];
	shared_mem_data[thread_id] += shared_mem_data[thread_id + 8];
	shared_mem_data[thread_id] += shared_mem_data[thread_id + 4];
	shared_mem_data[thread_id] += shared_mem_data[thread_id + 2];
	shared_mem_data[thread_id] += shared_mem_data[thread_id + 1];
}


// computing partial sums for every stride'th element in list (e.g. partialSums for all index % filters == 0, index % filters == 1, etc.)
// want to reduce over batch_size * spatial_dim * spatial_dim per filter
// each block will get partialSum of portion of batch_size * spatial_dim * spatial_dim elements for given filter
// thus there will be filters * ((batch_size * spatial_dim * spatial_dim) / BATCH_NORM_DERIV_REDUCTION_THREADS) blocks

// LAUNCH SPEC: <<< filters * ceil(((batch_size * spatial_dim * spatial_dim) / BATCH_NORM_DERIV_REDUCTION_THREADS)), BATCH_NORM_DERIV_REDUCTION_THREADS >>>


// partial sums has gridDim * stride entries, that further need to be reduced to stride entries for the full sum
// each block will compute partial sum of 256 elements

// NOT MEMORY COALESCED! Should change the order of arrays to have filter be the highest order, rather than lowest
__global__ void batchNormDerivReduction(int size, const float * gammas, const float * betas, const float * input,
														int spatial_dim, int filters, int batch_size, float eps, const float * means, const float * vars, const float * activated_derivs,
														float * global_partialDerivSum, float * global_partialDerivMultSum, float * global_partialDGamma, float * global_partialDBeta, float * normalized_temp_deriv){
	
	
	__shared__ float partialDerivSum[BATCH_NORM_DERIV_REDUCTION_THREADS];
	__shared__ float partialDerivMultSum[BATCH_NORM_DERIV_REDUCTION_THREADS];
	__shared__ float partialDGamma[BATCH_NORM_DERIV_REDUCTION_THREADS];
	__shared__ float partialDBeta[BATCH_NORM_DERIV_REDUCTION_THREADS];

	unsigned int filter_id = blockIdx.x % filters;
	unsigned int n_partials_per_filt = gridDim.x / filters;
	unsigned int filter_partial_ind = blockIdx.x / filters;
	unsigned int block_start = filter_id + filter_partial_ind * blockDim.x * filters;
	unsigned int thread_id = threadIdx.x;
	unsigned int index = block_start + thread_id * filters;
	
	if (index >= size){
		return;
	}

	float gamma = gammas[filter_id];
	float beta = betas[filter_id];
	float mean = means[filter_id];
	float var = vars[filter_id];

	float activated_deriv = activated_derivs[index];
	float normalized_temp_val = (input[index] - mean) / sqrtf(var + eps);
	float normalized_temp_deriv_val = activated_deriv * gamma;
	normalized_temp_deriv[index] = normalized_temp_deriv_val;

	partialDerivSum[thread_id] = normalized_temp_deriv_val;
	partialDerivMultSum[thread_id] = normalized_temp_deriv_val * normalized_temp_val;
	partialDGamma[thread_id] = activated_deriv * normalized_temp_val;
	partialDBeta[thread_id] = activated_deriv;

	__syncthreads();

	// now each block has 256 elements from the same filter loaded consecutively into shared memory.
	// do reduction of these and save to the global partial memory to then finalized sum over partials

	for (unsigned int stride = (blockDim.x/2); stride > 32; stride>>=1){
		if (thread_id < stride){
			partialDerivSum[thread_id] += partialDerivSum[thread_id + stride];
			partialDerivMultSum[thread_id] += partialDerivMultSum[thread_id + stride];
			partialDGamma[thread_id] += partialDGamma[thread_id + stride];
			partialDBeta[thread_id] += partialDBeta[thread_id + stride];
		}
		__syncthreads();
	}

	// only 1 warp left so manually unrolling 
	if (thread_id < 32){
		warpReduce(partialDerivSum, thread_id);
		warpReduce(partialDerivMultSum, thread_id);
		warpReduce(partialDGamma, thread_id);
		warpReduce(partialDBeta, thread_id);
	}

	int partial_ind_in_global = filter_id * n_partials_per_filt + filter_partial_ind;
	if (thread_id == 0){
		global_partialDerivSum[partial_ind_in_global] = partialDerivSum[0];
		global_partialDerivMultSum[partial_ind_in_global] = partialDerivMultSum[0];
		global_partialDGamma[partial_ind_in_global] = partialDGamma[0];
		global_partialDBeta[partial_ind_in_global] = partialDBeta[0];
	}
}


// LAUNCH WITH < filters, 1 >
// still more room for reductions over partials...
__global__ void finalizeReductionBatchNormDeriv(int filters, int n_partials_per_filt, float * global_partialDerivSum, float * global_partialDerivMultSum, float * global_partialDGamma, float * global_partialDBeta, 
												 float * gamma_deriv, float * beta_deriv, float * deriv_sums, float * deriv_mult_sums){

	int filter_id = blockIdx.x;

	if (filter_id > filters){
		return;
	}

	float dSum = 0;
	float dMultSum = 0;
	float dGamma = 0;
	float dBeta = 0;

	int filt_start_ind = filter_id * n_partials_per_filt;

	for (int i = 0; i < n_partials_per_filt; i++){
		dSum += global_partialDerivSum[filt_start_ind + i];
		dMultSum += global_partialDerivMultSum[filt_start_ind + i];
		dGamma += global_partialDGamma[filt_start_ind + i];
		dBeta += global_partialDBeta[filt_start_ind + i];
	}

	gamma_deriv[filter_id] = dGamma;
	beta_deriv[filter_id] = dBeta;
	deriv_sums[filter_id] = dSum;
	deriv_mult_sums[filter_id] = dMultSum;
}


// LAUNCH WITH < ceil(filters * batch_size * spatial_dim * spatial_dim / (MAX_THREAD_PER_BLOCK)), MAX_THREAD_PER_BLOCK >
// fully parallel
__global__ void computeBatchNormInputDerivFromReduction(int total_size, const float * input, int spatial_dim, int filters, int batch_size, float eps, const float * means, const float * vars,
									const float * normalized_temp_deriv, const float * deriv_sums, const float * deriv_mult_sums, float * input_deriv){

	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= total_size){
		return;
	}
	int filter_id = index % filters;
	float mean = means[filter_id];
	float var = vars[filter_id];
	float deriv_sum = deriv_sums[filter_id];
	float deriv_mult_sum = deriv_mult_sums[filter_id];
	int n_samples = batch_size * spatial_dim * spatial_dim;

	float normalized_temp_val = (input[index] - mean) / sqrtf(var + eps);
	input_deriv[index] = (n_samples * normalized_temp_deriv[index] - deriv_sum - normalized_temp_val * deriv_mult_sum) / (n_samples * sqrtf(var + eps));

}

__global__ void activationAndBatchNormDeriv(const float * input, const float * gamma, const float * beta, 
									int spatial_dim, int filters, int batch_size, float eps, const float * means, const float * vars, const float * activated,
									const float * out_layer_deriv, float * normalized_temp_deriv, float * gamma_deriv, float * beta_deriv, float * deriv_sums, float * deriv_mult_sums, float * input_deriv, bool to_activate_deriv){
	
	
	int filter_id = blockIdx.x * blockDim.x + threadIdx.x;
	if (filter_id >= filters){
		return;
	}

	float n_samples = batch_size * spatial_dim * spatial_dim;
	float gamma_val = gamma[filter_id];
	float mean_val = means[filter_id];
	float var_val = vars[filter_id];

	// first compute dL/activated (relu deriv) and then dL/dNormalized_Temp (== x hat)
	// also can compute dL/dGamma and dL/dBeta (parameters of batch norm)
	int index;
	float dGamma = 0;
	float dBeta = 0;
	float deriv_sum = 0;
	float deriv_mult_sum = 0;
	float activated_val, out_layer_deriv_val, normalized_temp_val, normalized_temp_deriv_val;
	for (int s = 0; s < batch_size; s++){
		for (int i = 0; i < spatial_dim; i++){
			for (int j = 0; j < spatial_dim; j++){
				index = spatial_dim * spatial_dim * filters * s + spatial_dim * filters * i + filters * j + filter_id;
				activated_val = activated[index];
				if (to_activate_deriv && (activated_val <= 0)) {
					normalized_temp_deriv[index] = 0;
				}
				else{
					out_layer_deriv_val = out_layer_deriv[index];
					normalized_temp_val = (input[index] - mean_val) / sqrtf(var_val + eps);
					normalized_temp_deriv_val = out_layer_deriv_val * gamma_val;
					deriv_sum += normalized_temp_deriv_val;
					deriv_mult_sum += normalized_temp_deriv_val * normalized_temp_val;
					dGamma += out_layer_deriv_val * normalized_temp_val;
					dBeta += out_layer_deriv_val;
					normalized_temp_deriv[index] = normalized_temp_deriv_val;
				}
			}
		}
	}

	// save down dGamma and dBeta so optimzer can update parameters
	gamma_deriv[filter_id] = dGamma;
	beta_deriv[filter_id] = dBeta;
	deriv_sums[filter_id] = deriv_sum;
	deriv_mult_sums[filter_id] = deriv_mult_sum;
}

// assume grid launch of (SPATIAL_OUT_DIM, SPATIAL_OUT_DIM) and block dim of (FILTERS)
// could parallelize over batches as well, but probably ok. 
// *runs into issues if #filters greater than threads per block
__global__ void doMaxPool(const float * input, int kern_dim, int stride, int batch_size, int * max_inds, float * out){

	int filter_id = threadIdx.x;

	// know this because of launch specification
	int filters = blockDim.x;
	int in_spatial_dim = stride * gridDim.x;
	int out_spatial_dim = gridDim.x;

	int spatial_row_start = stride * blockIdx.x;
	int spatial_col_start = stride * blockIdx.y;

	int half_kernel_dim = kern_dim / 2;

	float max_val, inp_val;
	int spatial_row, spatial_col, max_ind, inp_ind, out_ind;
	for (int s = 0; s < batch_size; s++){
		max_val = -1024;
		max_ind = -1024;
		for (int row_off = -half_kernel_dim; row_off <= half_kernel_dim; row_off++){
			for (int col_off = -half_kernel_dim; col_off <= half_kernel_dim; col_off++){
				spatial_row = spatial_row_start + row_off;
				spatial_col = spatial_col_start + col_off;
				if ((spatial_row < 0) || (spatial_row >= in_spatial_dim) || (spatial_col < 0) || (spatial_col >= in_spatial_dim)){
					continue;
				}
				inp_ind = in_spatial_dim * in_spatial_dim * filters * s + in_spatial_dim * filters * spatial_row + filters * spatial_col + filter_id;
				inp_val = input[inp_ind];
				if (inp_val > max_val){
					max_val = inp_val;
					max_ind = inp_ind;
				}
			}
		}
		out_ind = out_spatial_dim * out_spatial_dim * filters * s + out_spatial_dim * filters * blockIdx.x + filters * blockIdx.y + filter_id;
		max_inds[out_ind] = max_ind;
		out[out_ind] = max_val;
	}
}

// assume grid launch of (SPATIAL_OUT_DIM, SPATIAL_OUT_DIM, OUT_FILTERS) and block dim of (BATCH_SIZE)
// max_inds_populated is mapping from max_pool_out_index -> associated max_index of input (populated from forward pass)
// also assume max_pool_inp_deriv is populated with all 0's to begin with and we overwrite non-zero values
__global__ void maxPoolDeriv(const int *max_inds_populated, const float *out_deriv, int kern_dim, int in_spatial_dim, int stride, int filters, int batch_size, float * max_pool_inp_deriv){

	int out_spatial_dim = in_spatial_dim / stride;

	int out_spatial_row = blockIdx.x;
	int out_spatial_col = blockIdx.y;
	int out_filter_id = blockIdx.z;
	int sample_ind = threadIdx.x;

	// based on launch spec should be ok, but check anyways
	if ((out_spatial_row >= out_spatial_dim) || (out_spatial_col >= out_spatial_dim) || (out_filter_id >= filters) || (sample_ind >= batch_size)){
		return;
	}

	int out_ind = out_spatial_dim * out_spatial_dim * filters * sample_ind + out_spatial_dim * filters * out_spatial_row + filters * out_spatial_col + out_filter_id;
	int max_ind_for_out = max_inds_populated[out_ind];

	max_pool_inp_deriv[max_ind_for_out] = out_deriv[out_ind];
}


// assume grid launch of (# Filters) and block dim of (batch size)
// could parallelize over batches as well, but probably ok. 
// *runs into issues if #filters greater than threads per block
__global__ void doFilterAvgPool(const float * input, int spatial_dim, float * out){

	int filter_id = blockIdx.x;
	int sample_ind = threadIdx.x;

	// know this because of launch specification
	int filters = gridDim.x;

	float sum = 0;
	for (int row = 0; row < spatial_dim; row++){
		for (int col = 0; col < spatial_dim; col++){
			sum += input[spatial_dim * spatial_dim * filters * sample_ind + spatial_dim * filters * row + filters * col + filter_id];
		}
	}

	float avg_val = sum / (spatial_dim * spatial_dim);
	out[filters * sample_ind + filter_id] = avg_val;
}

// assume grid launch of (# Filters) and block dim of (batch size)
// could parallelize over batches as well, but probably ok. 
// *runs into issues if #filters greater than threads per block
__global__ void filterAvgPoolDeriv(const float * pooled_deriv, int filters, int batch_size, int spatial_dim, float * out){

	int filter_id = blockIdx.x;
	int sample_ind = threadIdx.x;

	// unnecessary because of launch conditions, but putting anyways...
	if ((filter_id >= filters) || (sample_ind >= batch_size)){
		return;
	}

	// indexing into (N, 2048) = (batch_size, filters) matrix 
	float pooled_filt_deriv = pooled_deriv[sample_ind * filters + filter_id];
	float avg_pooled_filt_deriv = pooled_filt_deriv / (spatial_dim * spatial_dim);

	// populating the pre-pooled conv block output
	for (int row = 0; row < spatial_dim; row++){
		for (int col = 0; col < spatial_dim; col++){
			out[spatial_dim * spatial_dim * filters * sample_ind + spatial_dim * filters * row + filters * col + filter_id] = avg_pooled_filt_deriv;
		}
	}
}



__global__ void doActivation(int size, const float * input, float * output){
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i >= size){
		return;
	}
	output[i] = fmaxf(0, input[i]);
}

__global__ void doActivationDeriv(int size, const float *input, const float * upstream_deriv, float * output){
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i >= size){
		return;
	}
	if (input[i] > 0){
		output[i] = upstream_deriv[i];
	}
	else{
		output[i] = 0;
	}
}

// assume pass in 1-D block with batch size blocks and 1 thread per block
// could exploit more parallelism here but shouldnt be bottleneck for now...
// assume X is a matrix where # rows = batch size and # columns = output dim
__global__ void softMax(const float * X, int batch_size, int output_len, float * out){
  int i = threadIdx.x;
  if (i < batch_size){
    float sum = 0;
    for (int j = 0; j < output_len; j++){
      sum += expf(X[i * output_len + j]);
    }
    for (int j = 0; j < output_len; j++){
      out[i * output_len + j] = expf(X[i * output_len + j]) / sum;
    }
  }
}

// launch with gridDim (output_dim) and threadDim (batch_size)
__global__ void averageDerivOverBatchSize(float * output_deriv, int output_dim, int batch_size){

	int output_class = blockIdx.x;
	int sample_ind = threadIdx.x;

	// shouldn't happen because of launch spec but check anyways...
	if ((output_class >= output_dim) || (sample_ind >= batch_size)){
		return;
	}
	output_deriv[sample_ind * output_dim + output_class] /= batch_size;
}


// launch with gridDim = (batch_size), blockDim = (1)
__global__ void crossEntropyDeriv(float * output_deriv, const int * correct_classes, int output_dim, int batch_size){
	int i = threadIdx.x;
	if (i < batch_size){
		output_deriv[i * output_dim + correct_classes[i]] -= 1;
	}
}

// assume large 1-D launch
__global__ void updateMeans(int size, const float * gradients, const float * model_params, float base_mean_decay, float weight_decay, float * prev_means, int loc_ind){
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i >= size){
		return;
	}
	if (isnan(gradients[i])){
		printf("ERROR in Update Means for Parameter at location: %d\nGradient is NAN at index: %d...keeping same running mean\n\n", loc_ind, i);
		return;
	}
	if (isinf(gradients[i])){
		printf("ERROR in Update Means for Parameter at location: %d\nGradient is INF at index: %d...keeping same running mean\n\n", loc_ind, i);
		return;
	}
	float grad_with_decay = gradients[i] + weight_decay * model_params[i];
	prev_means[i] = base_mean_decay * prev_means[i] + (1 - base_mean_decay) * grad_with_decay;
	
}

// assume large 1-D launch
__global__ void updateVars(int size, const float * gradients, const float * model_params, float base_var_decay, float weight_decay, float * prev_vars, int loc_ind){
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i >= size){
		return;
	}
	if (isnan(gradients[i])){
		printf("ERROR in Update Vars for Parameter at location: %d\nGradient is NAN at index: %d...keeping same running var\n", loc_ind, i);
		return;
	}
	if (isinf(gradients[i])){
		printf("ERROR in Update Vars for Parameter at location: %d\nGradient is INF at index: %d...keeping same running var\n", loc_ind, i);
		return;
	}
	float grad_with_decay = gradients[i] + weight_decay * model_params[i];
	prev_vars[i] = base_var_decay * prev_vars[i] + (1 - base_var_decay) * grad_with_decay * grad_with_decay;
}

// assume large 1-D launch
__global__ void updateParams(int size, float * model_params, const float * means, const float * vars, float learning_rate, float weight_decay, float cur_mean_decay, float cur_var_decay, float eps, int loc_ind){
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i >= size){
		return;
	}
	float mean_adj = means[i] / (1 - cur_mean_decay);
	float var_adj = vars[i] / (1 - cur_var_decay);
	float old_model_param = model_params[i];
	model_params[i] = model_params[i] - (learning_rate * (mean_adj / (sqrtf(var_adj) + eps)) + weight_decay * old_model_param);
	if (isnan(model_params[i])){
		printf("ERROR: for Parameter at location: %d\nto NAN at index: %d...resetting to prev value\n", loc_ind, i);
		model_params[i] = old_model_param;
		return;
	}
	if (isinf(model_params[i])){
		printf("ERROR: for Parameter at location: %d\nto INF at index: %d...resetting to prev value\n", loc_ind, i);
		model_params[i] = old_model_param;
		return;
	}
}

/* INITIALIZE CORE MODEL STRUCTURES */

Dims * init_dimensions(int input, int init_kernel_dim, int init_conv_filters, int init_conv_stride, int init_maxpool_dim, int init_maxpool_stride, 
							int n_conv_blocks, int * is_block_spatial_reduction, int final_depth, int output){
	
	Dims * dims = (Dims *) malloc(sizeof(Dims));
	dims -> input = input;
	dims -> init_kernel_dim = init_kernel_dim;
	dims -> init_conv_filters = init_conv_filters;
	dims -> init_conv_stride = init_conv_stride;
	dims -> init_maxpool_dim = init_maxpool_dim;
	dims -> init_maxpool_stride = init_maxpool_stride;
	dims -> n_conv_blocks = n_conv_blocks;
	dims -> is_block_spatial_reduction = is_block_spatial_reduction;
	dims -> final_depth = final_depth;
	dims -> output = output;

	return dims;
}

BatchNorm * init_batch_norm(int spatial_dim, int depth, float gamma_val, bool is_zero){
	
	BatchNorm * batch_norm = (BatchNorm *) malloc(sizeof(BatchNorm));

	batch_norm -> spatial_dim = spatial_dim;
	batch_norm -> depth = depth;

	float * gamma, * beta;

	hipMalloc(&gamma, depth * sizeof(float));
	hipMemset(gamma, 0, depth * sizeof(float));
	// ZERO-GAMMA INITIALIZE TO SOLVE PROBLEM OF EXPLODING GRADIENTS (Goyal et al. 2017)
	if (!is_zero){
		setVal <<< ceil((float) depth / MAX_THREAD_PER_BLOCK), MAX_THREAD_PER_BLOCK >>> (depth, gamma_val, gamma);
	}

	hipMalloc(&beta, depth * sizeof(float));
	hipMemset(beta, 0, depth * sizeof(float));

	batch_norm -> gamma = gamma;
	batch_norm -> beta = beta;

	return batch_norm;

}

ConvBlock * init_conv_block(int incoming_filters, int incoming_spatial_dim, int reduced_depth, int expanded_depth, int stride, hiprandGenerator_t * gen, bool is_zero){
	
	ConvBlock * conv_block = (ConvBlock *) malloc(sizeof(ConvBlock));
	conv_block -> incoming_filters = incoming_filters;
	conv_block -> incoming_spatial_dim = incoming_spatial_dim;
	conv_block -> reduced_depth = reduced_depth;
	conv_block -> expanded_depth = expanded_depth;
	conv_block -> stride = stride;

	float * depth_reduction, *spatial, *depth_expansion;
	int depth_reduction_size, spatial_size, depth_expansion_size;
	float depth_reduction_fan_in_plus_fan_out, spatial_fan_in_plus_fan_out, depth_expansion_fan_in_plus_fan_out;

	BatchNorm *norm_depth_reduction, *norm_spatial, *norm_expansion, *norm_projection;

	depth_reduction_size = incoming_filters * reduced_depth;
	depth_reduction_fan_in_plus_fan_out = incoming_filters + reduced_depth;
	hipMalloc(&depth_reduction, depth_reduction_size * sizeof(float));
	hipMemset(depth_reduction, 0, depth_reduction_size * sizeof(float));
	if (!is_zero){
		init_weights_gaussian_device(gen, depth_reduction_size, depth_reduction, 0, 2.0 / depth_reduction_fan_in_plus_fan_out);
	}

	norm_depth_reduction = init_batch_norm(incoming_spatial_dim, reduced_depth, 1.0, is_zero);


	spatial_size = reduced_depth * reduced_depth * 3 * 3;
	spatial_fan_in_plus_fan_out = (3 * 3) * (reduced_depth + reduced_depth);
	hipMalloc(&spatial, spatial_size * sizeof(float));
	hipMemset(spatial, 0, spatial_size * sizeof(float));
	if (!is_zero){
		init_weights_gaussian_device(gen, spatial_size, spatial, 0, 2.0 / spatial_fan_in_plus_fan_out);
	}

	// the spatial decrease happens at middle 3x3 layer, to the last layer of stride block will receive lower spatial dim input
	if (stride == 2){
		incoming_spatial_dim /= 2;
	}
	norm_spatial = init_batch_norm(incoming_spatial_dim, reduced_depth, 1.0, is_zero);

	depth_expansion_size = expanded_depth * reduced_depth;
	depth_expansion_fan_in_plus_fan_out = reduced_depth + expanded_depth;
	hipMalloc(&depth_expansion, depth_expansion_size * sizeof(float));
	hipMemset(depth_expansion, 0, depth_expansion_size * sizeof(float));
	if (!is_zero){
		init_weights_gaussian_device(gen, depth_expansion_size, depth_expansion, 0, 2.0 / depth_expansion_fan_in_plus_fan_out);
	}

	conv_block -> depth_reduction = depth_reduction;
	conv_block -> norm_depth_reduction = norm_depth_reduction;

	conv_block -> spatial = spatial;
	conv_block -> norm_spatial = norm_spatial;


	conv_block -> depth_expansion = depth_expansion;

	norm_expansion = init_batch_norm(incoming_spatial_dim, expanded_depth, 1.0, is_zero);
	conv_block -> norm_expansion = norm_expansion;

	float * projection;
	int projection_size;
	if (stride == 2){
		projection_size = 3 * 3 * incoming_filters * expanded_depth;
	}
	else{
		projection_size = incoming_filters * expanded_depth;
	}

	// assuming only project when depths are different (all projections in resnet-50 this way)
	// could later change to adapt to just spatial transform...
	int projection_fan_in_plus_fan_out;
	if (incoming_filters != expanded_depth){
		hipMalloc(&projection, projection_size * sizeof(float));
		hipMemset(projection, 0, projection_size * sizeof(float));
		if (stride == 2){
			projection_fan_in_plus_fan_out = 3 * 3 * (incoming_filters + expanded_depth);
		}
		else{
			projection_fan_in_plus_fan_out = incoming_filters + expanded_depth;
		}
		if (!is_zero){
			init_weights_gaussian_device(gen, projection_size, projection, 0, 2.0 / (projection_fan_in_plus_fan_out));
		}
		norm_projection = init_batch_norm(incoming_spatial_dim, expanded_depth, 1.0, is_zero);
	}
	else{
		projection = NULL;
		norm_projection = NULL;
	}

	conv_block -> projection = projection;
	conv_block -> norm_projection = norm_projection;

	return conv_block;
}

Params * init_model_parameters(Dims * model_dims, hiprandGenerator_t * gen, bool is_zero){

	Params * params = (Params *) malloc(sizeof(Params));

	// dimensions unpacked
	int input_dim = model_dims -> input;
	int n_conv_blocks = model_dims -> n_conv_blocks;
	int init_kernel_dim = model_dims -> init_kernel_dim;
	int init_conv_filters = model_dims -> init_conv_filters;
	int * is_block_spatial_reduction = model_dims -> is_block_spatial_reduction;
	int output_dim = model_dims -> output;

	// init array to hold pointers to weights
	// 3 * 3 weight arrays per conv block (weights, gamma, beta per layer in block) + 3 * inital + fully connected + 4 projections * 3
	// ignoring biases + batch norm weights for now...
	int n_locations = 16 + 9 * n_conv_blocks;
	params -> n_locations = n_locations;

	float ** locations = (float **) malloc(n_locations * sizeof(float *));
	int * sizes = (int *) malloc(n_locations * sizeof(int));
	// tracking location ind as we start allocating...
	


	// init first 7 * 7 conv_layer
	float * init_conv_layer;
	int init_conv_size = init_kernel_dim * init_kernel_dim * init_conv_filters * 3;
	float init_conv_fan_in_plus_fan_out = 7 * 7 * (3 + init_conv_filters);
	hipError_t malloc_err = hipMalloc(&init_conv_layer,  init_conv_size * sizeof(float));
	hipError_t memset_err = hipMemset(init_conv_layer, 0, init_conv_size * sizeof(float));
	if (!is_zero){
		init_weights_gaussian_device(gen, init_conv_size, init_conv_layer, 0, 2.0 / init_conv_fan_in_plus_fan_out);
	}
	params -> init_conv_layer = init_conv_layer;
	int loc_ind = 0;
	locations[loc_ind] = init_conv_layer;
	sizes[loc_ind] = init_kernel_dim * init_kernel_dim * init_conv_filters * 3;
	loc_ind++;

	BatchNorm * norm_init_conv = init_batch_norm(input_dim / model_dims -> init_conv_stride, init_conv_filters, 1.0, is_zero);
	params -> norm_init_conv = norm_init_conv;

	locations[loc_ind] = norm_init_conv -> gamma;
	sizes[loc_ind] = init_conv_filters;
	loc_ind++;

	locations[loc_ind] = norm_init_conv -> beta;
	sizes[loc_ind] = init_conv_filters;
	loc_ind++;
	

	// init conv blocks
	ConvBlock ** conv_blocks = (ConvBlock **) malloc(n_conv_blocks * sizeof(ConvBlock *));
	int incoming_filters = init_conv_filters;
	// assume stride 2 initial conv layer then stride 2 pool before entering conv_blocks
	int incoming_spatial_dim = input_dim / 4;
	int stride = 1;
	int reduced_depth = init_conv_filters;
	int expanded_depth = 4 * init_conv_filters;
	for (int i = 0; i < n_conv_blocks; i++){
		if (is_block_spatial_reduction[i] == 1){
			stride = 2;
			reduced_depth *= 2;
			expanded_depth *= 2;
		}
		else{
			stride = 1;
		}
		conv_blocks[i] = init_conv_block(incoming_filters, incoming_spatial_dim, reduced_depth, expanded_depth, stride, gen, is_zero);
		locations[loc_ind] = conv_blocks[i] -> depth_reduction;
		sizes[loc_ind] = incoming_filters * reduced_depth;
		loc_ind++;
		locations[loc_ind] = conv_blocks[i] -> norm_depth_reduction -> gamma;
		sizes[loc_ind] = reduced_depth;
		loc_ind++;
		locations[loc_ind] = conv_blocks[i] -> norm_depth_reduction -> beta;
		sizes[loc_ind] = reduced_depth;
		loc_ind++;

		locations[loc_ind] = conv_blocks[i] -> spatial;
		sizes[loc_ind] = reduced_depth * reduced_depth * 3 * 3;
		loc_ind++;
		locations[loc_ind] = conv_blocks[i] -> norm_spatial -> gamma;
		sizes[loc_ind] = reduced_depth;
		loc_ind++;
		locations[loc_ind] = conv_blocks[i] -> norm_spatial -> beta;
		sizes[loc_ind] = reduced_depth;
		loc_ind++;

		locations[loc_ind] = conv_blocks[i] -> depth_expansion;
		sizes[loc_ind] = expanded_depth * reduced_depth;
		loc_ind++;
		locations[loc_ind] = conv_blocks[i] -> norm_expansion -> gamma;
		sizes[loc_ind] = expanded_depth;
		loc_ind++;
		locations[loc_ind] = conv_blocks[i] -> norm_expansion -> beta;
		sizes[loc_ind] = expanded_depth;
		loc_ind++;
		
		// if the block needed a projection to make input dim = output dim
		if (conv_blocks[i] -> projection){
			locations[loc_ind] = conv_blocks[i] -> projection;
			if (stride == 2){
				sizes[loc_ind] = 3 * 3 * incoming_filters * expanded_depth;
			}
			else{
				sizes[loc_ind] = incoming_filters * expanded_depth;
			}
			loc_ind++;
			locations[loc_ind] = conv_blocks[i] -> norm_projection -> gamma;
			sizes[loc_ind] = expanded_depth;
			loc_ind++;
			locations[loc_ind] = conv_blocks[i] -> norm_projection -> beta;
			sizes[loc_ind] = expanded_depth;
			loc_ind++;
		}

		// after stride 2 block then reduce spatial dim for next block
		if (is_block_spatial_reduction[i] == 1){
			incoming_spatial_dim /= 2;
		}
		incoming_filters = expanded_depth;
	}
	params -> conv_blocks = conv_blocks;

	float * fully_connected;
	// here expanded depth is the last layer's filters which will go through average pool before FC layer
	// expanded depth should equal dims -> final_depth
	int fully_connected_size = expanded_depth * output_dim;
	float fully_connected_fan_in = expanded_depth;
	hipMalloc(&fully_connected, fully_connected_size * sizeof(float));
	hipMemset(fully_connected, 0, fully_connected_size * sizeof(float));
	if (!is_zero){
		init_weights_gaussian_device(gen, fully_connected_size, fully_connected, 0, 0.0001);
	}

	params -> fully_connected = fully_connected;
	locations[loc_ind] = fully_connected;
	sizes[loc_ind] = expanded_depth * output_dim;

	params -> locations = locations;
	params -> sizes = sizes;

	return params;
}

ResNet * init_resnet(Dims * dims, hiprandGenerator_t * gen){
	ResNet * model = (ResNet *) malloc(sizeof(ResNet));
	model -> dims = dims;
	Params * params = init_model_parameters(dims, gen, false);
	model -> params = params;
	return model;
}


/* INITIALIZE TRAINING STRUCTURES */

Cache_BatchNorm * init_cache_batchnorm(int input_size, int feature_size){
	Cache_BatchNorm * cache_batchnorm = (Cache_BatchNorm *) malloc(sizeof(Cache_BatchNorm));

	cache_batchnorm -> input_size = input_size;
	cache_batchnorm -> feature_size = feature_size;

	float * means, *vars;

	hipMalloc(&means, feature_size * sizeof(float));
	hipMalloc(&vars, feature_size * sizeof(float));

	cache_batchnorm -> means = means;
	cache_batchnorm -> vars = vars;

	return cache_batchnorm;
}

Activation_ConvBlock * init_activation_convblock(ConvBlock * conv_block, int batch_size){
	Activation_ConvBlock * activation_conv_block = (Activation_ConvBlock *) malloc(sizeof(Activation_ConvBlock));

	int incoming_filters = conv_block -> incoming_filters;
	int incoming_spatial_dim = conv_block -> incoming_spatial_dim;
	int stride = conv_block -> stride;
	int reduced_depth = conv_block -> reduced_depth;
	int expanded_depth = conv_block -> expanded_depth;

	activation_conv_block -> incoming_filters = incoming_filters;
	activation_conv_block -> incoming_spatial_dim = incoming_spatial_dim;
	activation_conv_block -> reduced_depth = reduced_depth;
	activation_conv_block -> expanded_depth = expanded_depth;
	activation_conv_block -> stride = stride;

	float * post_reduced, *post_spatial, *post_expanded, *transformed_residual, *post_projection_norm_vals, *output_activated;
	int post_reduced_size, post_spatial_size, output_size;
	Cache_BatchNorm * norm_post_reduced, *norm_post_spatial, *norm_post_expanded, *norm_post_projection;
	

	post_reduced_size = reduced_depth * incoming_spatial_dim * incoming_spatial_dim * batch_size;
	hipMalloc(&post_reduced, post_reduced_size * sizeof(float));
	activation_conv_block -> post_reduced = post_reduced;

	norm_post_reduced = init_cache_batchnorm(post_reduced_size, reduced_depth);
	activation_conv_block -> norm_post_reduced = norm_post_reduced;

	post_spatial_size = reduced_depth * incoming_spatial_dim * incoming_spatial_dim / (stride * stride) * batch_size;
	hipMalloc(&post_spatial, post_spatial_size * sizeof(float));
	activation_conv_block -> post_spatial = post_spatial;

	norm_post_spatial = init_cache_batchnorm(post_spatial_size, reduced_depth);
	activation_conv_block -> norm_post_spatial = norm_post_spatial;

	output_size = expanded_depth * incoming_spatial_dim * incoming_spatial_dim / (stride * stride) * batch_size;
	
	hipMalloc(&post_expanded, output_size * sizeof(float));
	activation_conv_block -> post_expanded = post_expanded;

	norm_post_expanded = init_cache_batchnorm(output_size, expanded_depth);
	activation_conv_block -> norm_post_expanded = norm_post_expanded;

	// only allocate space if transformed, otherwise it will be assumed to be identity of input
	transformed_residual = NULL;
	norm_post_projection = NULL;
	post_projection_norm_vals = NULL;
	if (incoming_filters != expanded_depth){
		hipMalloc(&transformed_residual, output_size * sizeof(float));
		norm_post_projection = init_cache_batchnorm(output_size, expanded_depth);
		hipMalloc(&post_projection_norm_vals, output_size * sizeof(float));
	}
	activation_conv_block -> transformed_residual = transformed_residual;
	activation_conv_block -> norm_post_projection = norm_post_projection;
	activation_conv_block -> post_projection_norm_vals = post_projection_norm_vals;

	hipMalloc(&output_activated, output_size * sizeof(float));
	activation_conv_block -> output_activated = output_activated;

	return activation_conv_block;
}

Activations * init_activations(Dims * dims, ConvBlock ** conv_blocks, int batch_size){
	
	Activations * activations = (Activations *) malloc(sizeof(Activations));

	int input_dim = dims -> input;
	int init_conv_filters = dims -> init_conv_filters;
	int init_conv_stride = dims -> init_conv_stride;
	int maxpool_stride = dims -> init_maxpool_stride;

	float * init_conv_applied;
	int init_conv_applied_size = init_conv_filters * input_dim * input_dim / (init_conv_stride * init_conv_stride) * batch_size; 
	hipMalloc(&init_conv_applied, init_conv_applied_size * sizeof(float));
	activations -> init_conv_applied = init_conv_applied;

	Cache_BatchNorm * norm_init_conv = init_cache_batchnorm(init_conv_applied_size, init_conv_filters);
	activations -> norm_init_conv = norm_init_conv;

	float * init_conv_activated;
	hipMalloc(&init_conv_activated, init_conv_applied_size * sizeof(float));
	activations -> init_conv_activated = init_conv_activated;

	int init_convblock_input_size = init_conv_filters * input_dim * input_dim / (init_conv_stride * init_conv_stride) / (maxpool_stride * maxpool_stride) * batch_size;

	int * max_inds;
	hipMalloc(&max_inds, init_convblock_input_size * sizeof(int));
	activations -> max_inds = max_inds;

	float *init_convblock_input;
	hipMalloc(&init_convblock_input, init_convblock_input_size * sizeof(float));
	activations -> init_convblock_input = init_convblock_input;

	int n_conv_blocks = dims -> n_conv_blocks;

	Activation_ConvBlock ** activation_conv_blocks = (Activation_ConvBlock **) malloc(n_conv_blocks * sizeof(Activation_ConvBlock *));
	for (int i = 0; i < n_conv_blocks; i++){
		ConvBlock * conv_block = conv_blocks[i];
		activation_conv_blocks[i] = init_activation_convblock(conv_block, batch_size);
	}

	activations -> activation_conv_blocks = activation_conv_blocks;
	activations -> n_conv_blocks = n_conv_blocks;

	int final_depth = dims -> final_depth;
	float * final_conv_output_pooled;
	int final_conv_output_pooled_size = final_depth * batch_size;
	hipMalloc(&final_conv_output_pooled, final_conv_output_pooled_size * sizeof(float));
	activations -> final_conv_output_pooled = final_conv_output_pooled;

	int output_dim = dims -> output;
	int output_size = output_dim * batch_size;

	float * linear_output;
	hipMalloc(&linear_output, output_size * sizeof(float));
	activations -> linear_output = linear_output;

	return activations;
}


Forward_Buffer * init_forward_buffer(Dims * dims, ConvBlock ** conv_blocks, int batch_size){

	Forward_Buffer * forward_buffer = (Forward_Buffer *) malloc(sizeof(Forward_Buffer));

	forward_buffer -> activations = init_activations(dims, conv_blocks, batch_size);

	int output_dim = dims -> output;
	int output_size = output_dim * batch_size;

	float * pred;
	hipMalloc(&pred, output_size * batch_size * sizeof(float));
	forward_buffer -> pred = pred;

	// will be copied to cpu to be able to print values and compute loss on cpu side
	float * pred_cpu = (float *) malloc(output_size * batch_size * sizeof(float));
	forward_buffer -> pred_cpu = pred_cpu;

	return forward_buffer;
}


Backprop_Buffer * init_backprop_buffer(Dims * dims, ConvBlock ** conv_blocks, int batch_size){

	Backprop_Buffer * backprop_buffer = (Backprop_Buffer *) malloc(sizeof(Backprop_Buffer));

	int output_dim = dims -> output;
	int output_size = output_dim * batch_size;

	float * output_layer_deriv;
	hipMalloc(&output_layer_deriv, output_size * sizeof(float));
	backprop_buffer -> output_layer_deriv = output_layer_deriv;

	backprop_buffer -> param_derivs = init_model_parameters(dims, NULL, true);
	backprop_buffer -> prev_means = init_model_parameters(dims, NULL, true);
	backprop_buffer -> prev_vars = init_model_parameters(dims, NULL, true);

	return backprop_buffer;
}


Train_ResNet * init_trainer(ResNet * model, Batch * cur_batch, int batch_size, float learning_rate, float weight_decay, float mean_decay, float var_decay, float eps, int n_epochs, int total_images){
	Train_ResNet * trainer = (Train_ResNet *) malloc(sizeof(Train_ResNet));

	trainer -> model = model;

	trainer -> cur_batch = cur_batch;
	trainer -> batch_size = batch_size;

	Dims * dims = model -> dims;
	ConvBlock ** conv_blocks = model -> params -> conv_blocks;
	trainer -> forward_buffer = init_forward_buffer(dims, conv_blocks, batch_size);
	trainer -> backprop_buffer = init_backprop_buffer(dims, conv_blocks, batch_size);

	trainer -> learning_rate = learning_rate;
	trainer -> weight_decay = weight_decay;
	trainer -> base_mean_decay = mean_decay;
	trainer -> base_var_decay = var_decay;
	trainer -> cur_mean_decay = 1;
	trainer -> cur_var_decay = 1;
	
	trainer -> eps = eps;

	trainer -> cur_epoch = 0;
	trainer -> n_epochs = n_epochs;
	trainer -> total_images = total_images;

	trainer -> cur_dump_id = -1;

	trainer -> loss_per_epoch = (float *) calloc(n_epochs, sizeof(float));
	trainer -> accuracy_per_epoch = (float *) calloc(n_epochs, sizeof(float));

	return trainer;
}

Batch * init_general_batch(int n_images, int image_size, int image_dim, int shard_n_images){
	Batch * batch = (Batch *) malloc(sizeof(Batch));

	batch -> n_images = n_images;
	// in resnet-50 will be 224 * 224 * 3
	batch -> image_size = image_size;
	batch -> image_dim = image_dim;
	float * images_float_cpu;
	// load batch by first brining into cpu, pinned memory
	hipError_t status_images_pinned = hipHostMalloc((float **)&images_float_cpu, (size_t) n_images * (size_t) image_size * sizeof(float));
	batch -> images_float_cpu = images_float_cpu;
	
	// allocate memory on gpu so that after loaded on cpu can bring in
	// will be converting from uint8 on CPU to float on GPU
	float * images;
	hipMalloc(&images, (size_t) n_images * (size_t) image_size * sizeof(float));
	batch -> images = images;

	// pinned memory for correct_classes_cpu
	int * correct_classes_cpu;
	hipError_t status_classes_pinned = hipHostMalloc((int **)&correct_classes_cpu, n_images * sizeof(int));
	batch -> correct_classes_cpu = correct_classes_cpu;

	int * correct_classes;
	hipMalloc(&correct_classes, n_images * sizeof(int));
	batch -> correct_classes = correct_classes;

	batch -> cur_shard_id = -1;
	batch -> cur_batch_in_shard = -1;
	batch -> shard_n_images = shard_n_images;
	batch -> full_shard_images = (float *) malloc((size_t) shard_n_images * (size_t) image_size * sizeof(float));
	batch -> full_shard_correct_classes = (int *) malloc(shard_n_images * sizeof(int));

	return batch;
}

// (if this takes too long, can do it in parallel with separate process on cpu)
// ASSUMING shard_n_images % batch_size = 0
void load_new_batch(Train_ResNet * trainer, Class_Metadata * class_metadata, Batch * batch_buffer){
	
	int batch_size = batch_buffer -> n_images;
	int image_size = batch_buffer -> image_size;
	size_t total_pixels = (size_t) batch_size * (size_t) image_size;
	
	float * full_shard_images = batch_buffer -> full_shard_images;
	int * full_shard_correct_classes = batch_buffer -> full_shard_correct_classes;	

	float * images_float_cpu = batch_buffer -> images_float_cpu;
	float * images = batch_buffer -> images;

	int * correct_classes_cpu = batch_buffer -> correct_classes_cpu;
	int * correct_classes = batch_buffer -> correct_classes;

	int cur_shard_id = batch_buffer -> cur_shard_id;
	int cur_batch_in_shard = batch_buffer -> cur_batch_in_shard;
	int shard_n_images = batch_buffer -> shard_n_images;
	
	int cur_dump_id = trainer -> cur_dump_id;



	int start_img_num = cur_batch_in_shard * batch_size;
	int n_read;
	int print_ret;

	char * shard_images_filepath, * shard_labels_filepath;
	// cur_shard_id = -1 implies first iteration
	if ((cur_shard_id == -1) || (start_img_num >= shard_n_images)) {

		// update new shard id
		cur_shard_id += 1;
		batch_buffer -> cur_shard_id = cur_shard_id;

		// load new shard into RAM
		print_ret = asprintf(&shard_images_filepath, "/mnt/storage/data/vision/imagenet/2012/train_data_shards/%03d.images", cur_shard_id);
		FILE * shard_images_file = fopen(shard_images_filepath, "rb");
		n_read = fread(full_shard_images, sizeof(float), ((size_t) shard_n_images) * ((size_t) image_size), shard_images_file);
		fclose(shard_images_file);
		free(shard_images_filepath);

		print_ret = asprintf(&shard_labels_filepath, "/mnt/storage/data/vision/imagenet/2012/train_data_shards/%03d.labels", cur_shard_id);
		FILE * shard_labels_file = fopen(shard_labels_filepath, "rb");
		n_read = fread(full_shard_correct_classes, sizeof(int), shard_n_images, shard_labels_file);
		fclose(shard_labels_file);
		free(shard_labels_filepath);

		// reset cur batch in shard to 0
		cur_batch_in_shard = 0;
		batch_buffer -> cur_batch_in_shard = cur_batch_in_shard;
	}

	// load current batch
	memcpy(images_float_cpu, full_shard_images + cur_batch_in_shard * total_pixels, total_pixels * sizeof(float));
	memcpy(correct_classes_cpu, full_shard_correct_classes + cur_batch_in_shard * batch_size, batch_size * sizeof(int));
	
	/* SAVING BATCH TO FILES FOR INSPECTION... */
	// if (cur_batch_in_shard == 0){
	// 	FILE * test_images_file = fopen("images.buffer", "wb");
	// 	fwrite(images_float_cpu, sizeof(float), total_pixels, test_images_file);
	// 	fclose(test_images_file);

	// 	FILE * test_labels_file = fopen("labels.buffer", "wb");
	// 	fwrite(correct_classes_cpu, sizeof(int), (size_t) batch_size, test_labels_file);
	// 	fclose(test_labels_file);
	// 	exit(0);
	// }

	// copy current batch to GPU

	hipMemcpy(images, images_float_cpu, total_pixels * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(correct_classes, correct_classes_cpu, batch_size * sizeof(int), hipMemcpyHostToDevice);

	// update cur batch for next iteration of loading
	cur_batch_in_shard++;
	batch_buffer -> cur_batch_in_shard = cur_batch_in_shard;

	cur_dump_id++;
	trainer -> cur_dump_id = cur_dump_id;


}


// READ CLASSES AND LABELS!
// reading a text file line by line into a buffer
// pre-allocate buffer and specify type
void text_file_to_buffer(void * buffer, char * filename, const char * type){

	char ** my_text_buffer = (char **) buffer;
	int * my_int_buffer = (int *) buffer;
	
	FILE * fp;
    char * line = NULL;
    size_t len = 0;

    fp = fopen(filename, "r");
    if (fp == NULL)
        exit(EXIT_FAILURE);
    int cnt = 0;
    while (getline(&line, &len, fp) != -1) {
    	if (strcmp(type, "TEXT") == 0){
        	my_text_buffer[cnt] = strdup(line);
        }
        else if (strcmp(type, "INT") == 0){
        	my_int_buffer[cnt] = atoi(line);
        }
        else{
        	// pass
        }
        cnt++;
    }

    fclose(fp);
    if (line){
    	free(line);
    }
}

Class_Metadata * populate_class_info(char * label_filename, char * synset_filename, char * class_size_filename, int n_classes){
	
	Class_Metadata * classes = (Class_Metadata *) malloc(sizeof(Class_Metadata));

	char ** labels = (char **) malloc(n_classes * sizeof(char *));
	char ** synsets = (char **) malloc(n_classes * sizeof(char *));
	int * counts = (int *) malloc(n_classes * sizeof(int));

	text_file_to_buffer(labels, label_filename, "TEXT");
	text_file_to_buffer(synsets, synset_filename, "TEXT");
	text_file_to_buffer(counts, class_size_filename, "INT");

	classes -> labels = labels;
	classes -> synsets = synsets;
	classes -> counts = counts;
	classes -> n_classes = n_classes;

	return classes;
}


/* PREP AND LAUNCHING CUDA KERNELS! */


// tried to write optimized conv kernel. will wait to debug...
// void prepareAndDoConvolutionOptimized(int in_spatial_dim, int kern_dim, int in_filters, int out_filters,  int stride, int batch_size, 
// 																float * input, float * weights, float * biases, float * output){

// 	int out_filter_row_size = kern_dim * kern_dim * in_filters;
// 	int max_out_filter_rows = MAX_SHARED_MEM_FLOATS / out_filter_row_size;
// 	int out_filter_chunks = ceil((float) out_filters / max_out_filter_rows);
// 	int shared_mem_size = out_filter_row_size * max_out_filter_rows;
// 	int out_spatial_dim = in_spatial_dim / stride;
// 	int max_subatch_size = MAX_THREAD_PER_BLOCK / max_out_filter_rows;

// 	dim3 gridDimConv(out_spatial_dim, out_spatial_dim, out_filter_chunks);
// 	dim3 blockDimConv(max_out_filter_rows, max_subatch_size);

// 	doConvolution <<< gridDimConv, blockDimConv, shared_mem_size >>> (input, weights, biases, in_spatial_dim, kern_dim, in_filters, out_filters, stride, batch_size, output);

// }



void prepareAndDoConvolution(int in_spatial_dim, int kern_dim, int in_filters, int out_filters,  int stride, int batch_size, 
																float * input, float * weights, float * output){
	int out_spatial_dim = in_spatial_dim / stride;
	int out_filters_block = min(MAX_THREAD_PER_BLOCK_INCL_REG / batch_size, out_filters);
	int out_filters_grid = max(1, (int) ceil((float) out_filters / (float) out_filters_block));

	dim3 gridDimConv(out_spatial_dim, out_spatial_dim, out_filters_grid);
	dim3 blockDimConv(batch_size, out_filters_block);

	doConvolution <<< gridDimConv, blockDimConv>>> (input, weights, in_spatial_dim, kern_dim, in_filters, out_filters, stride, batch_size, output);
}


void prepareAndDoConvolutionDeriv(int in_spatial_dim, int kern_dim, int in_filters, int out_filters, int stride, int batch_size, bool toAdd,
												float * input, float * weights, float * out_deriv,
												float * input_deriv, float * weight_deriv, bool toComputeInputDeriv){
	
	// first layer conv doesn't take deriv w.r.t input
	int in_filters_block = min(MAX_THREAD_PER_BLOCK_INCL_REG / batch_size, in_filters);
	int in_filters_grid = max(1, (int) ceil((float) in_filters / (float) in_filters_block));

	dim3 gridDimDerivInput(in_spatial_dim, in_spatial_dim, in_filters_grid);
	dim3 blockDimDerivInput(batch_size, in_filters_block);
	if (toComputeInputDeriv){
		convolutionDerivInput <<< gridDimDerivInput, blockDimDerivInput >>> (input, weights, out_deriv, in_spatial_dim, kern_dim, in_filters, out_filters, stride, batch_size, toAdd, input_deriv);
	}

	int block_dim, grid_dim;
	bool is_block_dim_inp;
	if (in_filters > MAX_THREAD_PER_BLOCK){
		block_dim = out_filters;
		grid_dim = in_filters;
		is_block_dim_inp = false;
	}
	else{
		block_dim = in_filters;
		grid_dim = out_filters;
		is_block_dim_inp = true;
	}
	
	dim3 gridDimDerivWeights(kern_dim, kern_dim, grid_dim);
	dim3 blockDimDerivWeights(block_dim);
	convolutionDerivWeights <<< gridDimDerivWeights, blockDimDerivWeights >>> (input, weights, out_deriv, in_spatial_dim, kern_dim, in_filters, out_filters, stride, batch_size, weight_deriv, is_block_dim_inp);
}



// NOT WORKING! :(

// void prepareAndDoConvolutionDerivOptimized(int in_spatial_dim, int kern_dim, int in_filters, int out_filters, int stride, int batch_size, bool toAdd,
// 												float * input, float * weights, float * out_deriv,
// 												float * input_deriv, float * weight_deriv, bool toComputeInputDeriv){
	
	

// 	int output_filters_per_partial = min(out_filters, MAX_SHARED_MEM_FLOATS / (kern_dim * kern_dim));
// 	int n_partials = ceil((float) out_filters / (float) output_filters_per_partial);
// 	int n_load_output_filters_per_thread = max(1, output_filters_per_partial / MAX_THREAD_PER_BLOCK_INCL_REG);
// 	int shared_mem_usage = output_filters_per_partial * kern_dim * kern_dim * sizeof(float);
// 	int total_weights = out_filters * in_filters * kern_dim * kern_dim;
	
// 	int input_size = batch_size * in_filters * in_spatial_dim * in_spatial_dim;

// 	dim3 gridDimDerivInput(in_filters, n_partials, batch_size);
// 	dim3 blockDimDerivInput(MAX_THREAD_PER_BLOCK_INCL_REG);

// 	dim3 gridDimTotalInputMax(ceil((float) (input_size) / (float) MAX_THREAD_PER_BLOCK));
// 	dim3 blockDimTotalInputMax(MAX_THREAD_PER_BLOCK);
	
// 	// first layer conv doesn't take deriv w.r.t input;
// 	if (toComputeInputDeriv){
// 		float * partial_input_deriv;
// 		hipMalloc(&partial_input_deriv, input_size * n_partials * sizeof(float));

// 		convolutionDerivInputPartial <<< gridDimDerivInput, blockDimDerivInput >>> (input, weights, out_deriv, in_spatial_dim, kern_dim, in_filters, out_filters, total_weights, stride, batch_size, n_load_output_filters_per_thread, output_filters_per_partial, n_partials, partial_input_deriv);

// 		finalizeRecutionConvolutionInputDeriv <<< gridDimTotalInputMax, blockDimTotalInputMax >>> (input_size, n_partials, partial_input_deriv, input_deriv, toAdd);

// 		hipFree(partial_input_deriv);
// 	}

// 	int block_dim, grid_dim;
// 	bool is_block_dim_inp;
// 	if (in_filters > MAX_THREAD_PER_BLOCK){
// 		block_dim = out_filters;
// 		grid_dim = in_filters;
// 		is_block_dim_inp = false;
// 	}
// 	else{
// 		block_dim = in_filters;
// 		grid_dim = out_filters;
// 		is_block_dim_inp = true;
// 	}
	
// 	dim3 gridDimDerivWeights(kern_dim, kern_dim, grid_dim);
// 	dim3 blockDimDerivWeights(block_dim);
// 	convolutionDerivWeights <<< gridDimDerivWeights, blockDimDerivWeights >>> (input, weights, out_deriv, in_spatial_dim, kern_dim, in_filters, out_filters, stride, batch_size, weight_deriv, is_block_dim_inp);
// }


void prepareAndDoBatchNormAndActivate(BatchNorm * batch_norm_params, Cache_BatchNorm * batch_norm_cache, int batch_size, float eps, float * input, float * activated_out, bool to_activate){
	// reading values from batch norm params
	int filters = batch_norm_params -> depth;
	int spatial_dim = batch_norm_params -> spatial_dim;
	float * gamma = batch_norm_params -> gamma;
	float * beta = batch_norm_params -> beta;

	// read the output device pointers from batch_norm_cache
	float * means_out = batch_norm_cache -> means;
	float * vars_out = batch_norm_cache -> vars;

	int num_threads = min(MAX_THREAD_PER_BLOCK_INCL_REG, filters);
	int num_blocks = 1;
	if (filters > num_threads){
		num_blocks = ceil((float) filters / (float) MAX_THREAD_PER_BLOCK_INCL_REG);
	}

	dim3 gridDimMeansAndVars(num_blocks);
	dim3 blockDimMeansAndVars(num_threads);
	getMeansAndVars <<< gridDimMeansAndVars, blockDimMeansAndVars >>> (input, spatial_dim, filters, batch_size, means_out, vars_out);

	int total_size = batch_size * filters * spatial_dim * spatial_dim;
	num_threads = MAX_THREAD_PER_BLOCK;
	num_blocks = ceil((float) (total_size) / (float) MAX_THREAD_PER_BLOCK);

	dim3 gridDimBatchNorm(num_blocks);
	dim3 blockDimBatchNorm(num_threads);
	doRecomputeBatchNormAndActivate<<< gridDimBatchNorm, blockDimBatchNorm >>> (total_size, input, gamma, beta, spatial_dim, filters, batch_size, eps, means_out, vars_out, activated_out, to_activate);
}

void prepareAndDoRecomputeBNActivation(float * input, BatchNorm * batch_norm_params, Cache_BatchNorm * batch_norm_cache, int batch_size, float eps, float * out, bool to_activate){

	// reading values from batch norm params
	int filters = batch_norm_params -> depth;
	int spatial_dim = batch_norm_params -> spatial_dim;
	float * gamma = batch_norm_params -> gamma;
	float * beta = batch_norm_params -> beta;

	// read the output device pointers from batch_norm_cache
	float * means_out = batch_norm_cache -> means;
	float * vars_out = batch_norm_cache -> vars;

	int total_size = batch_size * filters * spatial_dim * spatial_dim;
	int num_threads = MAX_THREAD_PER_BLOCK;
	int num_blocks = ceil((float) (total_size) / (float) MAX_THREAD_PER_BLOCK);

	dim3 gridDimBatchNorm(num_blocks);
	dim3 blockDimBatchNorm(num_threads);
	doRecomputeBatchNormAndActivate<<< gridDimBatchNorm, blockDimBatchNorm >>> (total_size, input, gamma, beta, spatial_dim, filters, batch_size, eps, means_out, vars_out, out, to_activate);
}

// void prepareAndDoActivationAndBatchNormDerivOld(BatchNorm * batch_norm_params, Cache_BatchNorm * batch_norm_cache, BatchNorm * batch_norm_param_derivs, 
// 																								int batch_size, float eps, float * input, float * activated, float * out_layer_deriv, float * input_deriv, bool to_activate_deriv){
// 	int filters = batch_norm_params -> depth;
// 	int spatial_dim = batch_norm_params -> spatial_dim;
// 	float * gamma = batch_norm_params -> gamma;
// 	float * beta = batch_norm_params -> beta;
// 	float * means = batch_norm_cache -> means;
// 	float * vars = batch_norm_cache -> vars;

// 	float * normalized_temp_deriv;
// 	hipMalloc(&normalized_temp_deriv, batch_size * filters * spatial_dim * spatial_dim * sizeof(float));

// 	float * deriv_sums;
// 	hipMalloc(&deriv_sums, filters * sizeof(float));

// 	float * deriv_mult_sums;
// 	hipMalloc(&deriv_mult_sums, filters * sizeof(float));

// 	float * gamma_deriv = batch_norm_param_derivs -> gamma;
// 	float * beta_deriv = batch_norm_param_derivs -> beta;

// 	int num_threads = min(MAX_THREAD_PER_BLOCK_INCL_REG, filters);
// 	int num_blocks = 1;
// 	if (filters > num_threads){
// 		num_blocks = ceil((float) filters / (float) MAX_THREAD_PER_BLOCK_INCL_REG);
// 	}

// 	dim3 gridDimBatchNormDeriv(num_blocks);
// 	dim3 blockDimBatchNormDeriv(num_threads);
// 	activationAndBatchNormDeriv <<< gridDimBatchNormDeriv, blockDimBatchNormDeriv >>> (input, gamma, beta, spatial_dim, filters, batch_size, eps, means, vars, activated, out_layer_deriv, normalized_temp_deriv, gamma_deriv, beta_deriv, deriv_sums, deriv_mult_sums, input_deriv, to_activate_deriv);

// 	int total_size = batch_size * filters * spatial_dim * spatial_dim;
// 	num_threads = MAX_THREAD_PER_BLOCK;
// 	num_blocks = ceil((float) (total_size) / (float) MAX_THREAD_PER_BLOCK);

// 	dim3 gridDimBatchNormInpDeriv(num_blocks);
// 	dim3 blockDimBatchNormInpDeriv(num_threads);

// 	batchNormInputDeriv <<< gridDimBatchNormInpDeriv, blockDimBatchNormInpDeriv >>> (total_size, input, spatial_dim, filters, batch_size, eps, means, vars, normalized_temp_deriv, deriv_sums, deriv_mult_sums, input_deriv);

// 	hipFree(normalized_temp_deriv);
// 	hipFree(deriv_sums);
// 	hipFree(deriv_mult_sums);

// }


void prepareAndDoActivationAndBatchNormDeriv(BatchNorm * batch_norm_params, Cache_BatchNorm * batch_norm_cache, BatchNorm * batch_norm_param_derivs, 
																								int batch_size, float eps, float * input, float * activated, float * out_layer_deriv, float * input_deriv, bool to_activate_deriv){
	int filters = batch_norm_params -> depth;
	int spatial_dim = batch_norm_params -> spatial_dim;
	int total_size = batch_size * filters * spatial_dim * spatial_dim;

	float * activation_deriv;
	dim3 gridDimTotalSizeMax(ceil((float) total_size / (float) MAX_THREAD_PER_BLOCK));
	dim3 blockDimTotalSizeMax(MAX_THREAD_PER_BLOCK);

	if (to_activate_deriv){
		hipMalloc(&activation_deriv, total_size * sizeof(float));
		doActivationDeriv <<< gridDimTotalSizeMax, blockDimTotalSizeMax >>> (total_size, activated, out_layer_deriv, activation_deriv);
	}
	else{
		activation_deriv = out_layer_deriv;
	}

	float * gammas = batch_norm_params -> gamma;
	float * betas = batch_norm_params -> beta;
	float * means = batch_norm_cache -> means;
	float * vars = batch_norm_cache -> vars;

	float * normalized_temp_deriv;
	hipMalloc(&normalized_temp_deriv, batch_size * filters * spatial_dim * spatial_dim * sizeof(float));

	

	int n_partials_per_filt = ceil((float) (batch_size * spatial_dim * spatial_dim) / (float) BATCH_NORM_DERIV_REDUCTION_THREADS);
	

	float * global_partialDerivSum, *global_partialDerivMultSum, *global_partialDGamma, *global_partialDBeta;
	hipMalloc(&global_partialDerivSum, filters * n_partials_per_filt * sizeof(float));
	hipMalloc(&global_partialDerivMultSum, filters * n_partials_per_filt * sizeof(float));
	hipMalloc(&global_partialDGamma, filters * n_partials_per_filt * sizeof(float));
	hipMalloc(&global_partialDBeta, filters * n_partials_per_filt * sizeof(float));


	dim3 gridDimBatchNormReduction(filters * n_partials_per_filt);
	dim3 blockDimBatchNormReduction(BATCH_NORM_DERIV_REDUCTION_THREADS);

	batchNormDerivReduction <<< gridDimBatchNormReduction, blockDimBatchNormReduction >>> (total_size, gammas, betas, input, spatial_dim, filters, batch_size, eps, means, vars, activation_deriv, 
																							global_partialDerivSum, global_partialDerivMultSum, global_partialDGamma, global_partialDBeta, normalized_temp_deriv);

	if (to_activate_deriv){
		hipFree(activation_deriv);
	}


	float * gamma_deriv = batch_norm_param_derivs -> gamma;
	float * beta_deriv = batch_norm_param_derivs -> beta;
	
	float * deriv_sums;
	hipMalloc(&deriv_sums, filters * sizeof(float));

	float * deriv_mult_sums;
	hipMalloc(&deriv_mult_sums, filters * sizeof(float));


	dim3 gridDimFinalizeBNReduction(filters);
	dim3 blockDimFinalizeBNReduction(1);

	finalizeReductionBatchNormDeriv <<< gridDimFinalizeBNReduction, blockDimFinalizeBNReduction >>> (filters, n_partials_per_filt, global_partialDerivSum, global_partialDerivMultSum, global_partialDGamma, global_partialDBeta,
																										gamma_deriv, beta_deriv, deriv_sums, deriv_mult_sums);
	hipFree(global_partialDerivSum);
	hipFree(global_partialDerivMultSum);
	hipFree(global_partialDGamma);
	hipFree(global_partialDBeta);

	computeBatchNormInputDerivFromReduction <<< gridDimTotalSizeMax, blockDimTotalSizeMax >>> (total_size, input, spatial_dim, filters, batch_size, eps, means, vars,
																								normalized_temp_deriv, deriv_sums, deriv_mult_sums, input_deriv);
	hipFree(normalized_temp_deriv);
	hipFree(deriv_sums);
	hipFree(deriv_mult_sums);

}


void prepareAndDoMatMulLeftTranspose(const float * left_orig, const float * right, int left_orig_rows, int left_orig_cols, int right_rows, int right_cols, float * out){
	float * temp_left;
	hipMalloc(&temp_left, left_orig_rows * left_orig_cols * sizeof(float));

	dim3 gridDimTranspose(ceil((float) left_orig_rows / TILE_WIDTH), ceil((float)left_orig_cols / TILE_WIDTH));
	dim3 blockDimTranspose(TILE_WIDTH, TILE_WIDTH);
	transpose <<< gridDimTranspose, blockDimTranspose >>> (left_orig, left_orig_rows, left_orig_cols, temp_left);

	dim3 gridDimMatMul(ceil((float) left_orig_cols / TILE_WIDTH), ceil((float) right_cols / TILE_WIDTH));
	dim3 blockDimMatMul(TILE_WIDTH, TILE_WIDTH);
	matMul <<< gridDimMatMul, blockDimMatMul >>> (temp_left, right, left_orig_cols, right_rows, right_cols, out);
	hipFree(temp_left);
}

void prepareAndDoMatMulRightTranspose(const float * left, const float * right_orig, int left_rows, int left_cols, int right_orig_rows, int right_orig_cols, float * out){
	float * temp_right;
	hipMalloc(&temp_right, right_orig_rows * right_orig_cols * sizeof(float));
	
	dim3 gridDimTranspose(ceil((float) right_orig_rows / TILE_WIDTH), ceil((float)right_orig_cols / TILE_WIDTH));
	dim3 blockDimTranspose(TILE_WIDTH, TILE_WIDTH);

	transpose <<< gridDimTranspose, blockDimTranspose >>> (right_orig, right_orig_rows, right_orig_cols, temp_right);
	
	dim3 gridDimMatMul(ceil((float) left_rows / TILE_WIDTH), ceil((float) right_orig_rows / TILE_WIDTH));
	dim3 blockDimMatMul(TILE_WIDTH, TILE_WIDTH);
	matMul <<< gridDimMatMul, blockDimMatMul >>> (left, temp_right, left_rows, left_cols, right_orig_rows, out);
	hipFree(temp_right);
}


void forward_pass(Train_ResNet * trainer){

	Dims * dims = trainer -> model -> dims;

	float eps = trainer -> eps;
	int batch_size = trainer -> batch_size;

	float * input = trainer -> cur_batch -> images;
	float * first_conv = trainer -> model -> params -> init_conv_layer;
	float * first_conv_output = trainer -> forward_buffer -> activations -> init_conv_applied;
	// first apply the convolutions
	// launch grid dimensions as (OUT_SPATIAL_DIM, OUT_SPATIAL_DIM, OUT_FILTER_CHUNK) blocks, and launch with block dim as (out_filt_rows_shared, sub_batch) threads
	
	// 3 colors
	int init_in_filters = 3;
	int init_spatial_dim = dims -> input;
	int init_kernel_dim = dims -> init_kernel_dim;
	int init_out_filters = dims -> init_conv_filters;
	int init_stride = dims -> init_conv_stride;
	int init_out_spatial_dim = init_spatial_dim / init_stride;

	prepareAndDoConvolution(init_spatial_dim, init_kernel_dim, init_in_filters, init_out_filters, init_stride, batch_size, input, first_conv, first_conv_output);

	BatchNorm * norm_init_conv_params = trainer -> model -> params -> norm_init_conv;
	Cache_BatchNorm * norm_init_conv_cache = trainer -> forward_buffer -> activations -> norm_init_conv;
	float * init_activated = trainer -> forward_buffer -> activations -> init_conv_activated;

	prepareAndDoBatchNormAndActivate(norm_init_conv_params, norm_init_conv_cache, batch_size, eps, first_conv_output, init_activated, true);


	int init_maxpool_dim = dims -> init_maxpool_dim;
	int init_maxpool_stride = dims -> init_maxpool_stride;
	int init_maxpool_out_dim = init_out_spatial_dim / init_maxpool_stride;
	float * init_convblock_input = trainer -> forward_buffer -> activations -> init_convblock_input;
	int * max_ind_buff = trainer -> forward_buffer -> activations -> max_inds;

	dim3 gridDimMaxPool(init_maxpool_out_dim, init_maxpool_out_dim);
	dim3 blockDimMaxPool(init_out_filters);
	doMaxPool <<< gridDimMaxPool , blockDimMaxPool >>> (init_activated, init_maxpool_dim, init_maxpool_stride, batch_size, max_ind_buff, init_convblock_input);

	/* NOW CAN MOVE ONTO TO CONV_BLOCK LAYERS! */

	int n_conv_blocks = dims -> n_conv_blocks;

	
	ConvBlock ** params_conv_blocks = trainer -> model -> params -> conv_blocks;
	Activation_ConvBlock ** activation_conv_blocks = trainer -> forward_buffer -> activations -> activation_conv_blocks;
	ConvBlock * cur_conv_block_params;
	Activation_ConvBlock * cur_conv_block_activation;
	int in_spatial_dim, kern_dim, in_filters, out_filters, stride, out_spatial_dim, total_size_conv_block_output;

	float * conv_block_input = init_convblock_input;
	float *conv_input, * conv_weights, * conv_output, *norm_input, * norm_output, * conv_block_output;
	float *projection_weights, *transformed_residual, *post_projection_norm_vals;
	BatchNorm * cur_batch_norm_params;
	Cache_BatchNorm * cur_batch_norm_cache;
	for (int i = 0; i < n_conv_blocks; i++){
		cur_conv_block_params = params_conv_blocks[i];
		cur_conv_block_activation = activation_conv_blocks[i];

		// do first 1x1 depth_reduce convolution
		in_spatial_dim = cur_conv_block_params -> incoming_spatial_dim;
		in_filters = cur_conv_block_params -> incoming_filters;
		out_filters = cur_conv_block_params -> reduced_depth;
		kern_dim = 1;
		stride = 1;
		// either intialized first time above loop from the maxpool
		// every other block will be the non-activated output of previous conv block (previous iteration output) 
		conv_input = conv_block_input;
		conv_weights = cur_conv_block_params -> depth_reduction;
		conv_output = cur_conv_block_activation -> post_reduced;

		prepareAndDoConvolution(in_spatial_dim, kern_dim, in_filters, out_filters, stride, batch_size, conv_input, conv_weights, conv_output);

		norm_input = conv_output;
		cur_batch_norm_cache = cur_conv_block_activation -> norm_post_reduced;
		cur_batch_norm_params = cur_conv_block_params -> norm_depth_reduction;
		hipMalloc(&norm_output, batch_size * (cur_batch_norm_params -> spatial_dim) * (cur_batch_norm_params -> spatial_dim) * (cur_batch_norm_params -> depth) * sizeof(float));

		prepareAndDoBatchNormAndActivate(cur_batch_norm_params, cur_batch_norm_cache, batch_size, eps, norm_input, norm_output, true);

		// do 3x3 spatial convolution

		// same as in first conv
		in_spatial_dim = cur_conv_block_params -> incoming_spatial_dim;
		// now is output filters of 1st conv, which is reduced depth filters
		in_filters = cur_conv_block_params -> reduced_depth;
		// keeps depth the same, just spatial conv
		out_filters = cur_conv_block_params -> reduced_depth;
		kern_dim = 3;
		// if stride is occurring in conv block happens at this kernel
		stride = cur_conv_block_params -> stride;

		// going to process norm&activation within conv kernel
		conv_input = norm_output;

		conv_weights = cur_conv_block_params -> spatial;
		conv_output = cur_conv_block_activation -> post_spatial;

		prepareAndDoConvolution(in_spatial_dim, kern_dim, in_filters, out_filters, stride, batch_size, conv_input, conv_weights, conv_output);

		hipFree(norm_output);

		norm_input = conv_output;
		cur_batch_norm_cache = cur_conv_block_activation -> norm_post_spatial;
		cur_batch_norm_params = cur_conv_block_params -> norm_spatial;
		hipMalloc(&norm_output, batch_size * (cur_batch_norm_params -> spatial_dim) * (cur_batch_norm_params -> spatial_dim) * (cur_batch_norm_params -> depth) * sizeof(float));

		prepareAndDoBatchNormAndActivate(cur_batch_norm_params, cur_batch_norm_cache, batch_size, eps, norm_input, norm_output, true);

		// do 1x1 depth expansion convolution

		// if stride happened now would need to take that into account
		in_spatial_dim = (cur_conv_block_params -> incoming_spatial_dim) / (cur_conv_block_params -> stride);
		// prev 3x3 conv kept out filters as reduced depth
		in_filters = cur_conv_block_params -> reduced_depth;
		// now creating expanded depth out filters
		out_filters = cur_conv_block_params -> expanded_depth;
		kern_dim = 1;
		stride = 1;
		conv_input = norm_output;
		conv_weights = cur_conv_block_params -> depth_expansion;
		conv_output = cur_conv_block_activation -> post_expanded;

		prepareAndDoConvolution(in_spatial_dim, kern_dim, in_filters, out_filters, stride, batch_size, conv_input, conv_weights, conv_output);

		hipFree(norm_output);

		norm_input = conv_output;
		cur_batch_norm_params = cur_conv_block_params -> norm_expansion;
		cur_batch_norm_cache = cur_conv_block_activation -> norm_post_expanded;
		norm_output = cur_conv_block_activation -> output_activated;

		// do not activate because first need to add to (projection) residual
		prepareAndDoBatchNormAndActivate(cur_batch_norm_params, cur_batch_norm_cache, batch_size, eps, norm_input, norm_output, false);

		// now need to add identity of conv_block_input (if same dimensions), or project=convolve (different dimensions) and add to conv_output
		// projection is a incoming block filters X expanded depth matrix
		// if stride of 2 in additon to depth change, then 3x3 kernel with stride 2 applied to block input
		// works as a depth-wise 1x1 convolution where in_filters = incoming_filters and out_filters = expanded_depth

		// already updated
		in_spatial_dim = (cur_conv_block_params -> incoming_spatial_dim);
		out_spatial_dim = (cur_conv_block_params -> incoming_spatial_dim) / (cur_conv_block_params -> stride);
		in_filters = cur_conv_block_params -> incoming_filters;
		out_filters = cur_conv_block_params -> expanded_depth;
		stride = cur_conv_block_params -> stride;
		if (stride == 2){
			kern_dim = 3;
		}
		else{
			kern_dim = 1;
		}
		projection_weights = cur_conv_block_params -> projection;

		total_size_conv_block_output = out_spatial_dim * out_spatial_dim * out_filters * batch_size;
		
				
		// the conv_block initializer already handled if we need projection, and if so it allocated weights
		// if there is a projection needed we will do convolution with the above parameters
		if (projection_weights){
			// allocated device memory to store output
			transformed_residual = cur_conv_block_activation -> transformed_residual;
			prepareAndDoConvolution(in_spatial_dim, kern_dim, in_filters, out_filters, stride, batch_size, conv_block_input, projection_weights, transformed_residual);
			post_projection_norm_vals = cur_conv_block_activation -> post_projection_norm_vals;
			prepareAndDoBatchNormAndActivate(cur_conv_block_params -> norm_projection, cur_conv_block_activation -> norm_post_projection, batch_size, eps, transformed_residual, post_projection_norm_vals, false);
		}
		else{
			// would've been null, so renaming for semantic clarity
			post_projection_norm_vals = conv_block_input;
		}

		dim3 gridDimConvOutput(ceil((float) total_size_conv_block_output / MAX_THREAD_PER_BLOCK));
		dim3 blockDimConvOutput(MAX_THREAD_PER_BLOCK);

		conv_block_output = cur_conv_block_activation -> output_activated;
		// add identity residual connection (or projected residual connection) to the prior batch norm output
		addVec <<< gridDimConvOutput, blockDimConvOutput >>> (total_size_conv_block_output, norm_output, post_projection_norm_vals, conv_block_output);

		doActivation <<< gridDimConvOutput, blockDimConvOutput >>> (total_size_conv_block_output, conv_block_output, conv_block_output);
		
		// prepare for next block...
		// don't bother with activation because can do that within first conv kernel of next block
		conv_block_input = conv_block_output;
	}

	int final_filters = dims -> final_depth;
	int final_spatial_dim = params_conv_blocks[n_conv_blocks - 1] -> incoming_spatial_dim;
	float * final_conv_block_output = activation_conv_blocks[n_conv_blocks - 1] -> output_activated;
	float * final_avg_pool_values = trainer -> forward_buffer -> activations -> final_conv_output_pooled;

	// NEED TO DO AVERAGE POOL OF LAST LAYER to go from (batch_size, 7, 7, 2048) to (batch size, 1, 1, 2048)

	// format of output is each row is a sample and has a row size of 2048
	dim3 gridDimAvgPool(final_filters);
	dim3 blockDimAvgPool(batch_size);
	// again assuming that we didn't actiate teh final conv block yet...
	doFilterAvgPool <<< gridDimAvgPool, blockDimAvgPool >>> (final_conv_block_output, final_spatial_dim, final_avg_pool_values);

	// APPLY FULLY CONNECTED LAYER BETWEEN (2048, 1000)
	float * fc_weights = trainer -> model -> params -> fully_connected;
	float * fc_output = trainer -> forward_buffer -> activations -> linear_output;
	int output_dim = dims -> output;

	// matrix multiply between (N, 2048) and fc weights of (2048, 1000), yields output of (N, 1000)
	// output is each row is a unique sample

	// GRID has dim (OUT_ROWS / TILE_WIDTH, OUT_COLS/TILE_WIDTH)
	// each BLOCK has dim (TILE_WIDTH, TILE_WIDTH)
	dim3 gridDimFCOutput(ceil((float) batch_size / TILE_WIDTH), ceil((float) output_dim / TILE_WIDTH));
	dim3 blockDimFCOutput(TILE_WIDTH, TILE_WIDTH);

	matMul <<< (gridDimFCOutput), (blockDimFCOutput) >>> (final_avg_pool_values, fc_weights, batch_size, final_filters, output_dim, fc_output);

	// DO SOFTMAX
	float * pred = trainer -> forward_buffer -> pred;
	dim3 gridDimSoftMax(1);
	dim3 blockDimSoftMax(batch_size);
	softMax <<< gridDimSoftMax, blockDimSoftMax >>> (fc_output, batch_size, output_dim, pred);

	// FINISH UP BY POPULATING PREDICTIONS ONTO CPU
	float * pred_cpu = trainer -> forward_buffer -> pred_cpu;
	hipMemcpy(pred_cpu, pred, batch_size * output_dim * sizeof(float), hipMemcpyDeviceToHost);
}

void backwards_pass(Train_ResNet * trainer){
	
	Dims * dims = trainer -> model -> dims;
	int batch_size = trainer -> batch_size;
	int output_dim = dims -> output;
	float eps = trainer -> eps;
	Activations * activations = trainer -> forward_buffer -> activations;
	Params * model_params = trainer -> model -> params;
	Backprop_Buffer * backprop_buffer = trainer -> backprop_buffer;
	Params * param_derivs = backprop_buffer -> param_derivs;

	/* STEP 1: LAST LAYER DERIVATIVE */

	// layer has output_dim * batch_size values
	// End of network was: fully connected layer -> softmax
	// Derivative of cross entropy loss w.r.t to fully connected values is: s - y where s is softmax value
	// thus copy softmax values and subtract 1 from the correct index (we know labels y are 0 except correct label of 1)
	int * correct_classes = trainer -> cur_batch -> correct_classes;
	float * pred = trainer -> forward_buffer -> pred;
	float * output_layer_deriv = backprop_buffer -> output_layer_deriv;
	hipMemcpy(output_layer_deriv, pred, batch_size * output_dim * sizeof(float), hipMemcpyDeviceToDevice);

	dim3 gridDimCrossDeriv(1);
	dim3 blockDimCrossDeriv(batch_size);
	crossEntropyDeriv <<< gridDimCrossDeriv, blockDimCrossDeriv >>> (output_layer_deriv, correct_classes, output_dim, batch_size);

	// divide by the batch size because loss is sum across all batches...
	// NOT SURE IF WE WANT TO DO AVERAGE HERE OR NOT...?
	
	// dim3 gridDimTakeAvgDeriv(output_dim);
	// dim3 blockDimTakeAvgDeriv(batch_size);
	// averageDerivOverBatchSize <<< gridDimTakeAvgDeriv, blockDimTakeAvgDeriv >>> (output_layer_deriv, output_dim, batch_size);

	/* STEP 2: FC WEIGHT DERIV AND FINAL AVG POOL (SECOND LAST ACTIVTION LAYER) DERIVATIVE */

	// TODO: MAKE SURE THE DIMENSIONS ARE CORRECT ORDER...

	// FC WEIGHTS (2048, 1000) DERIV = matMul(transpose(final_conv_output_pooled), output_layer_deriv)
	int final_depth = dims -> final_depth;
	float * fc_deriv = param_derivs -> fully_connected;
	float * final_conv_output_pooled = activations -> final_conv_output_pooled;
	prepareAndDoMatMulLeftTranspose(final_conv_output_pooled, output_layer_deriv, batch_size, final_depth, batch_size, output_dim, fc_deriv);

	// FINAL AVG POOL (N, 2048) DERIV = matMul(output_layer_deriv, transpose(FC Weight))
	float * fc_weights = model_params -> fully_connected;
	
	float * final_avg_pool_deriv;
	hipMalloc(&final_avg_pool_deriv, batch_size * final_depth * sizeof(float));
	prepareAndDoMatMulRightTranspose(output_layer_deriv, fc_weights, batch_size, output_dim, final_depth, output_dim, final_avg_pool_deriv);


	/* CONV BLOCK DATA FROM FORWARD PASS */
	int n_conv_blocks = dims -> n_conv_blocks;
	Activation_ConvBlock ** activation_conv_blocks = activations -> activation_conv_blocks;
	ConvBlock ** conv_block_params = model_params -> conv_blocks;

	ConvBlock ** conv_block_param_derivs = param_derivs -> conv_blocks;

	int final_spatial_dim = conv_block_params[n_conv_blocks - 1] -> incoming_spatial_dim;
	
	/* STEP 3: AVG POOL DERIV */

	// get the location for the deriv of final conv block output
	float * final_conv_block_output_deriv;
	hipMalloc(&final_conv_block_output_deriv, batch_size * final_spatial_dim * final_spatial_dim * final_depth * sizeof(float));

	// using final_avg_pool_deriv (batch_size, 2048) to populate final_conv_block_output_deriv (batch_size, 7, 7, 2048)
	// each expanded (prior to pooling) spatial index takes on value of given filter's avg_pool_deriv / (spatial_dim^2)
	dim3 gridDimAvgPoolDeriv(final_depth);
	dim3 blockDimAvgPoolDeriv(batch_size);
	filterAvgPoolDeriv <<< gridDimAvgPoolDeriv, blockDimAvgPoolDeriv >>> (final_avg_pool_deriv, final_depth, batch_size, final_spatial_dim, final_conv_block_output_deriv);

	hipFree(final_avg_pool_deriv);

	
	/* STEP 4: CONV BLOCK & BATCH NORM DERIVS  */
	

	// we are starting with deriv of last conv block output...

	// To go backwards for each block we:
		// 1.) Get deriv of output activated (ReLU so just 0 or 1)
		// 2.) Get deriv projection filter & transformed (if there is a projection of residual, otherwise both derivs are 1)
		// 3.) Multiply the deriv of output activation * deriv of transformed residual and add to the deriv of first layer of conv block (= output activated of prior block)
		// 4.) Multiply the deriv of output activation * deriv of batch norm for expanded conv output (with respect to both its own parameters and also the input to batch norm = expanded conv output)
		// 5.) Get deriv of expanded convolution & deriv of input to expanded convolution (= batch norm output of spatial conv)
		// 6.) Get deriv of batch norm for spatial conv output (with respect to both its own parameters and also the input to batch norm = spatial conv output)
		// 7.) Get deriv of sptial convolution & deriv of input to spatial convolution (= batch norm output of reduced conv)
		// 8.) Get deriv of batch norm for reduced conv output (with respect to both its own parameters and also the input to batch norm = reduced conv output)
		// 9.) Get deriv of reduced convolution & deriv of input to reduced convolution, which is the first layer of conv block (= batch norm output of prior conv block)
		// Items 3.) and 9.) provide the derivative used to repeat process for prior block

	

	// will update these variables throughout loop to pass to batch norm deriv
	float *bn_input, *bn_activated, *bn_out_layer_deriv, *bn_input_deriv;
	BatchNorm *cur_batch_norm_params, *cur_batch_norm_param_derivs;
	Cache_BatchNorm *cur_batch_norm_cache;

	// will update these every iteration through conv_blocks
	ConvBlock * cur_conv_block_params, *cur_conv_block_param_derivs;
	Activation_ConvBlock * cur_conv_block_activation;

	// will update these within every iteration through conv_blocks
	// because multiple convolutions per block, but keep params same for easy calls to functions
	int in_spatial_dim, kern_dim, in_filters, out_filters, stride;
	float *conv_input, *conv_weight, *conv_out_deriv;
	float *conv_input_deriv, *conv_weight_deriv;


	int input_filters, input_spatial_dim;

	// STARTING POINT FROM BACKPROP COMING FROM UPSTREAM LAYERS IS AT LAST CONV BLOCK ACTIVATION -> OUTPUT_ACTIVATED
	float *conv_block_input, *conv_block_input_deriv, *prev_conv_block_input_deriv, * upstream_deriv, *block_activation_deriv, *final_output, *prev_bn_activated, *conv_block_input_activated;

	// extra temp variables
	int total_size, output_size, input_size;

	for (int i = n_conv_blocks - 1; i >= 0; i--){

		// getting current conv block parameters and buffers to hold derivs
		cur_conv_block_params = conv_block_params[i];
		cur_conv_block_param_derivs = conv_block_param_derivs[i];

		// getting current conv block activation values and buffers to hold derivs
		cur_conv_block_activation = activation_conv_blocks[i];

		input_filters = cur_conv_block_params -> incoming_filters;
		input_spatial_dim = cur_conv_block_params -> incoming_spatial_dim;

		input_size = batch_size * input_filters * input_spatial_dim * input_spatial_dim;

		hipMalloc(&conv_block_input_deriv, input_size * sizeof(float));


		// residual deriv and normal backprop deriv added to this
		if (i == 0){
			conv_block_input = activations -> init_convblock_input;
		}
		else{
			conv_block_input = activation_conv_blocks[i - 1] -> output_activated;
		}

		/* 1: Conv Block Output Activation */
		
		// GIVEN
		if (i == n_conv_blocks - 1){
			upstream_deriv = final_conv_block_output_deriv;
		}
		else{
			upstream_deriv = prev_conv_block_input_deriv;
		}
		
		final_output = cur_conv_block_activation -> output_activated;

		
		output_size = batch_size * cur_conv_block_params -> expanded_depth * cur_conv_block_params -> incoming_spatial_dim * cur_conv_block_params -> incoming_spatial_dim / ((cur_conv_block_params -> stride) * (cur_conv_block_params -> stride));

		// to fill in the ReLU deriv location
		hipMalloc(&block_activation_deriv, output_size * sizeof(float));

		dim3 gridDimOutput(ceil((float) output_size / MAX_THREAD_PER_BLOCK));
		dim3 blockDimOutput(MAX_THREAD_PER_BLOCK);
		doActivationDeriv <<< gridDimOutput, blockDimOutput >>> (output_size, final_output, upstream_deriv, block_activation_deriv);

		if (i == n_conv_blocks - 1){
			hipFree(final_conv_block_output_deriv);
		}
		else{
			hipFree(prev_conv_block_input_deriv);
		}



		/* 2: (Transformed) Residual Derivs & Chained/Added to Conv Block Input Deriv (= prior_block_output_deriv) */

		// check if there is a projection (aka convolution over depth/kern_dim=1 or possibly stride=2/kern_dim=3), otherwise the projection deriv is 1
		// If there is a projection need to compute derivative of the projection convolution kernel weights and deriv w.r.t. projection convolution input=conv_block_input=prior_block_output_activated
		if (cur_conv_block_params -> projection){


			// DEAL WITH BATCH NORM
			// update the current batch norm layer pointers
			cur_batch_norm_params = cur_conv_block_params -> norm_projection;
			cur_batch_norm_param_derivs = cur_conv_block_param_derivs -> norm_projection;

			cur_batch_norm_cache = cur_conv_block_activation -> norm_post_projection;

			// fill in details about backprop I/O
			// dL/dBN_Output (given)
			bn_out_layer_deriv = block_activation_deriv;
			
			// dL/dBN_Input (to fill in)
			hipMalloc(&bn_input_deriv, output_size * sizeof(float));
			
			// input to batch norm layer from forward pass
			bn_input = cur_conv_block_activation -> transformed_residual;
			
			// activated output of batch norm layer from forward pass
			bn_activated = cur_conv_block_activation -> post_projection_norm_vals;
		
			prepareAndDoActivationAndBatchNormDeriv(cur_batch_norm_params, cur_batch_norm_cache, cur_batch_norm_param_derivs,
																						batch_size, eps, bn_input, bn_activated, bn_out_layer_deriv, bn_input_deriv, false);
			// CONVOLUTION DIMENSIONS
			in_spatial_dim = (cur_conv_block_params -> incoming_spatial_dim);
			in_filters = cur_conv_block_params -> incoming_filters;
			out_filters = cur_conv_block_params -> expanded_depth;
			stride = cur_conv_block_params -> stride;
			if (stride == 2){
				kern_dim = 3;
			}
			else{
				kern_dim = 1;
			}


			// CONVOLUTION FORWARD DATA
			// transformed residual convolution input is the value at first step of conv block => activated output from previous block
			conv_input = conv_block_input;
			conv_weight = cur_conv_block_params -> projection;
			// from backprop
			conv_out_deriv = bn_input_deriv;

			// CONVOLUTION BACKWARDS DERIV DATA BUFFERS
			// because residual
			conv_input_deriv = conv_block_input_deriv;
			conv_weight_deriv = cur_conv_block_param_derivs -> projection;

			prepareAndDoConvolutionDeriv(in_spatial_dim, kern_dim, in_filters, out_filters, stride, batch_size, false,
													conv_input, conv_weight, conv_out_deriv,
													conv_input_deriv, conv_weight_deriv, true);

			hipFree(bn_input_deriv);

		}
		else{
			total_size = batch_size * (cur_conv_block_params -> incoming_spatial_dim) * (cur_conv_block_params -> incoming_spatial_dim) * (cur_conv_block_params -> incoming_filters);
			dim3 gridDimResidual(ceil((float) total_size / MAX_THREAD_PER_BLOCK));
			dim3 blockDimResidual(MAX_THREAD_PER_BLOCK);
			setVal <<< gridDimResidual, blockDimResidual >>> (total_size, 0, conv_block_input_deriv);
			addVec <<< gridDimResidual, blockDimResidual >>> (total_size, conv_block_input_deriv, block_activation_deriv, conv_block_input_deriv);
		}
		

		/* 3: Expanded Convolution And Batch Norm Derivs */

		// update the current batch norm layer pointers
		cur_batch_norm_params = cur_conv_block_params -> norm_expansion;
		cur_batch_norm_param_derivs = cur_conv_block_param_derivs -> norm_expansion;

		cur_batch_norm_cache = cur_conv_block_activation -> norm_post_expanded;

		// Re-compute BN Activation from Expanded
		hipMalloc(&prev_bn_activated, batch_size * cur_batch_norm_params -> spatial_dim * cur_batch_norm_params -> spatial_dim * cur_batch_norm_params -> depth * sizeof(float));

		prepareAndDoRecomputeBNActivation(cur_conv_block_activation -> post_expanded, cur_batch_norm_params, cur_batch_norm_cache, batch_size, eps, prev_bn_activated, false);

		// fill in details about backprop I/O
		// dL/dBN_Output (given)
		bn_out_layer_deriv = block_activation_deriv;
		// dL/dBN_Input (to fill in)
		hipMalloc(&bn_input_deriv, output_size * sizeof(float));
		// input to batch norm layer from forward pass
		bn_input = cur_conv_block_activation -> post_expanded;
		
		prepareAndDoActivationAndBatchNormDeriv(cur_batch_norm_params, cur_batch_norm_cache, cur_batch_norm_param_derivs,
																						batch_size, eps, bn_input, prev_bn_activated, bn_out_layer_deriv, bn_input_deriv, false);
		hipFree(block_activation_deriv);
		hipFree(prev_bn_activated);

		// CONVOLUTION DIMENSIONS
		in_spatial_dim = (cur_conv_block_params -> incoming_spatial_dim) / (cur_conv_block_params -> stride);
		in_filters = cur_conv_block_params -> reduced_depth;
		out_filters = cur_conv_block_params -> expanded_depth;
		stride = 1;
		kern_dim = 1;

		// CONVOLUTION FORWARD DATA
		conv_input = cur_conv_block_activation -> post_spatial;
		cur_batch_norm_params = cur_conv_block_params -> norm_spatial;
		cur_batch_norm_cache = cur_conv_block_activation -> norm_post_spatial;
		conv_weight = cur_conv_block_params -> depth_expansion;
		// from backprop
		conv_out_deriv = bn_input_deriv;

		// CONVOLUTION BACKWARDS DERIV DATA BUFFERS
		// because residual
		hipMalloc(&conv_input_deriv, batch_size * in_spatial_dim * in_spatial_dim * in_filters * sizeof(float));
		conv_weight_deriv = cur_conv_block_param_derivs -> depth_expansion;


		// Re-compute BN Activation from Spatial
		hipMalloc(&prev_bn_activated, batch_size * in_spatial_dim * in_spatial_dim * in_filters * sizeof(float));

		prepareAndDoRecomputeBNActivation(conv_input, cur_batch_norm_params, cur_batch_norm_cache, batch_size, eps, prev_bn_activated, true);

		prepareAndDoConvolutionDeriv(in_spatial_dim, kern_dim, in_filters, out_filters, stride, batch_size, false,
														prev_bn_activated, conv_weight, conv_out_deriv,
														conv_input_deriv, conv_weight_deriv, true);

		hipFree(bn_input_deriv);
		

		/* 4: Spatial Convolution Activation and Batch Norm Derivs */

		// update the current batch norm layer pointers
		cur_batch_norm_params = cur_conv_block_params -> norm_spatial;
		cur_batch_norm_param_derivs = cur_conv_block_param_derivs -> norm_spatial;

		cur_batch_norm_cache = cur_conv_block_activation -> norm_post_spatial;

		// fill in details about backprop I/O
		// dL/dBN_Output (given)
		bn_out_layer_deriv = conv_input_deriv;
		// dL/dBN_Input (to fill in)
		hipMalloc(&bn_input_deriv, batch_size * in_spatial_dim * in_spatial_dim * cur_conv_block_params -> reduced_depth * sizeof(float));
		// input to batch norm layer from forward pass
		bn_input = cur_conv_block_activation -> post_spatial;
		
		prepareAndDoActivationAndBatchNormDeriv(cur_batch_norm_params, cur_batch_norm_cache, cur_batch_norm_param_derivs,
																						batch_size, eps, bn_input, prev_bn_activated, bn_out_layer_deriv, bn_input_deriv, true);

		hipFree(conv_input_deriv);
		hipFree(prev_bn_activated);

		/* 5: Spatial Convolution Derivs */



		// CONVOLUTION DIMENSIONS
		in_spatial_dim = cur_conv_block_params -> incoming_spatial_dim;
		in_filters = cur_conv_block_params -> reduced_depth;
		out_filters = cur_conv_block_params -> reduced_depth;
		stride = cur_conv_block_params -> stride;
		kern_dim = 3;

		// CONVOLUTION FORWARD DATA
		conv_input = cur_conv_block_activation -> post_reduced;
		cur_batch_norm_params = cur_conv_block_params -> norm_depth_reduction;
		cur_batch_norm_cache = cur_conv_block_activation -> norm_post_reduced;

		conv_weight = cur_conv_block_params -> spatial;
		// from backprop
		conv_out_deriv = bn_input_deriv;

		// CONVOLUTION BACKWARDS DERIV DATA BUFFERS
		// because residual
		hipMalloc(&conv_input_deriv, batch_size * in_spatial_dim * in_spatial_dim * in_filters * sizeof(float));
		conv_weight_deriv = cur_conv_block_param_derivs -> spatial;

		// Re-compute BN Activation from Reduction
		hipMalloc(&prev_bn_activated, batch_size * in_spatial_dim * in_spatial_dim * in_filters * sizeof(float));

		prepareAndDoRecomputeBNActivation(conv_input, cur_batch_norm_params, cur_batch_norm_cache, batch_size, eps, prev_bn_activated, true);

		prepareAndDoConvolutionDeriv(in_spatial_dim, kern_dim, in_filters, out_filters, stride, batch_size, false,
														prev_bn_activated, conv_weight, conv_out_deriv,
														conv_input_deriv, conv_weight_deriv, true);

		hipFree(bn_input_deriv);

		/* 6: Reduced Convolution Activation and Batch Norm Derivs */

		// update the current batch norm layer pointers
		cur_batch_norm_params = cur_conv_block_params -> norm_depth_reduction;
		cur_batch_norm_param_derivs = cur_conv_block_param_derivs -> norm_depth_reduction;

		cur_batch_norm_cache = cur_conv_block_activation -> norm_post_reduced;

		// fill in details about backprop I/O
		// dL/dBN_Output (given)
		bn_out_layer_deriv = conv_input_deriv;
		// dL/dBN_Input (to fill in)
		hipMalloc(&bn_input_deriv, batch_size * in_spatial_dim * in_spatial_dim * cur_conv_block_params -> reduced_depth * sizeof(float));
		// input to batch norm layer from forward pass
		bn_input = cur_conv_block_activation -> post_reduced;
		
		prepareAndDoActivationAndBatchNormDeriv(cur_batch_norm_params, cur_batch_norm_cache, cur_batch_norm_param_derivs,
																						batch_size, eps, bn_input, prev_bn_activated, bn_out_layer_deriv, bn_input_deriv, true);

		hipFree(conv_input_deriv);
		hipFree(prev_bn_activated);


		/* 7: Reduced Convolution Derivs */


		// CONVOLUTION DIMENSIONS
		in_spatial_dim = cur_conv_block_params -> incoming_spatial_dim;
		in_filters = cur_conv_block_params -> incoming_filters;
		out_filters = cur_conv_block_params -> reduced_depth;
		stride = 1;
		kern_dim = 1;

		// CONVOLUTION FORWARD DATA
		conv_input = conv_block_input;
		conv_weight = cur_conv_block_params -> depth_reduction;
		// from backprop
		conv_out_deriv = bn_input_deriv;

		// CONVOLUTION BACKWARDS DERIV DATA BUFFERS
		// because residual
		conv_input_deriv = conv_block_input_deriv;
		conv_weight_deriv = cur_conv_block_param_derivs -> depth_reduction;

		prepareAndDoConvolutionDeriv(in_spatial_dim, kern_dim, in_filters, out_filters, stride, batch_size, true,
													conv_input, conv_weight, conv_out_deriv,
													conv_input_deriv, conv_weight_deriv, true);

		hipFree(bn_input_deriv);
		

		prev_conv_block_input_deriv = conv_block_input_deriv;

	}


	/* STEP 5: MAX POOL DERIV */

	// maxpool dimensions (used in forward pass)
	int maxpool_kern_dim = dims -> init_maxpool_dim;
	int maxpool_stride = dims -> init_maxpool_stride;
	int maxpool_in_spatial_dim = dims -> input / dims -> init_conv_stride;
	int maxpool_out_spatial_dim = maxpool_in_spatial_dim / maxpool_stride;
	int maxpool_filters = dims -> init_conv_filters;

	// backprop up through the init convblock input has been done. the gradient is at:
	float * maxpool_out_deriv = conv_block_input_deriv;

	// getting the max inds cached from forward pass to easily do backprop
	int * max_inds = activations -> max_inds;

	// populating the gradient of input to max_pool located at:
	float * maxpool_inp_deriv;

	// ensure that gradient has 0's, so that maxPoolDeriv kernel can overwrite only at max ind locations
	int maxpool_inp_size = maxpool_in_spatial_dim * maxpool_in_spatial_dim * maxpool_filters * batch_size;
	hipMalloc(&maxpool_inp_deriv, maxpool_inp_size * sizeof(float));
	hipMemset(maxpool_inp_deriv, 0, maxpool_inp_size * sizeof(float));

	dim3 gridDimMaxPoolDeriv(maxpool_out_spatial_dim, maxpool_out_spatial_dim, maxpool_filters);
	dim3 blockDimMaxPoolDeriv(batch_size);

	// compute max pool deriv (i.e. populate maxpool_inp_deriv)
	maxPoolDeriv <<< gridDimMaxPoolDeriv, blockDimMaxPoolDeriv >>> (max_inds, maxpool_out_deriv, maxpool_kern_dim, maxpool_in_spatial_dim, maxpool_stride, maxpool_filters, batch_size, maxpool_inp_deriv);

	hipFree(conv_block_input_deriv);

	/* STEP 6: INIT BATCH NORM & CONV DERIV */

	// BACKPROP OVER THE BATCH NORM OF FIRST CONV LAYER

	// update the current batch norm layer pointers
	cur_batch_norm_params = model_params -> norm_init_conv;
	cur_batch_norm_param_derivs = param_derivs -> norm_init_conv;

	cur_batch_norm_cache = activations -> norm_init_conv;

	// fill in details about backprop I/O
	// dL/dBN_Output (given)
	bn_out_layer_deriv = maxpool_inp_deriv;
	// dL/dBN_Input (to fill in)
	hipMalloc(&bn_input_deriv, batch_size * dims -> init_conv_filters * (dims -> input) * (dims -> input) / ((dims -> init_conv_stride) * (dims -> init_conv_stride)) * sizeof(float));
	// input to batch norm layer from forward pass
	bn_input = activations -> init_conv_applied;
	// activated output of batch norm layer from forward pass
	bn_activated = activations -> init_conv_activated;
		
	prepareAndDoActivationAndBatchNormDeriv(cur_batch_norm_params, cur_batch_norm_cache, cur_batch_norm_param_derivs,
																						batch_size, eps, bn_input, bn_activated, bn_out_layer_deriv, bn_input_deriv, true);

	hipFree(maxpool_inp_deriv);

	// BACKPROP OVER FIRST CONV LAYER

	// CONVOLUTION DIMENSIONS
	// hardcoded to 3 for the colors
	in_filters = 3;
	out_filters = dims -> init_conv_filters;
	in_spatial_dim = dims -> input;
	stride = dims -> init_conv_stride;
	kern_dim = dims -> init_kernel_dim;

	// CONVOLUTION FORWARD DATA
	conv_input = trainer -> cur_batch -> images;
	conv_weight = model_params -> init_conv_layer;
	// from backprop
	conv_out_deriv = bn_input_deriv;

	// CONVOLUTION BACKWARDS DERIV DATA BUFFERS
	// because residual
	conv_input_deriv = NULL;
	conv_weight_deriv = param_derivs -> init_conv_layer;

	prepareAndDoConvolutionDeriv(in_spatial_dim, kern_dim, in_filters, out_filters, stride, batch_size, false,
													conv_input, conv_weight, conv_out_deriv,
													conv_input_deriv, conv_weight_deriv, false);

	hipFree(bn_input_deriv);
}

void dump_parameters(int dump_id, Train_ResNet * trainer){

	Params * model_params = trainer -> model -> params;
	float ** model_params_locations = model_params -> locations;
	int * param_sizes = model_params -> sizes;
	int n_locations = model_params -> n_locations;

	// values calculated from backprop, will reset these before returning
	Params * current_gradients = trainer -> backprop_buffer -> param_derivs;
	float ** current_gradient_locations = current_gradients -> locations;
	
	// running history values that the optimizer needs, will update these before returning
	Params * prev_grad_means = trainer -> backprop_buffer -> prev_means;
	float ** prev_grad_means_locations = prev_grad_means -> locations;
	Params * prev_grad_vars = trainer -> backprop_buffer -> prev_vars;
	float ** prev_grad_vars_locations = prev_grad_vars -> locations;

	int param_size;
	float *model_location, *grad_location, * mean_location, * var_location;

	float * cpu_param_buff;
	FILE * fp;

	char * model_params_filepath;
	char * gradients_filepath;
	char * means_filepath;
	char * vars_filepath;

	int n_read, print_ret;
	for (int i = n_locations - 1; i >= 0; i--){
		param_size = param_sizes[i];
		cpu_param_buff = (float *) malloc(param_size * sizeof(float));

		model_location = model_params_locations[i];
		hipMemcpy(cpu_param_buff, model_location, param_size * sizeof(float), hipMemcpyDeviceToHost);
		print_ret = asprintf(&model_params_filepath, "/mnt/storage/data/vision/imagenet/training_dumps/%08d/model_params/%03d.buffer", dump_id, i);
		fp = fopen(model_params_filepath, "wb");
		n_read = fwrite(cpu_param_buff, sizeof(float), (size_t) param_size, fp);
		fclose(fp);
		free(model_params_filepath);


		grad_location = current_gradient_locations[i];
		hipMemcpy(cpu_param_buff, grad_location, param_size * sizeof(float), hipMemcpyDeviceToHost);
		print_ret = asprintf(&gradients_filepath, "/mnt/storage/data/vision/imagenet/training_dumps/%08d/gradients/%03d.buffer", dump_id, i);
		fp = fopen(gradients_filepath, "wb");
		n_read = fwrite(cpu_param_buff, sizeof(float), (size_t) param_size, fp);
		fclose(fp);
		free(gradients_filepath);

		mean_location = prev_grad_means_locations[i];
		hipMemcpy(cpu_param_buff, mean_location, param_size * sizeof(float), hipMemcpyDeviceToHost);
		print_ret = asprintf(&means_filepath, "/mnt/storage/data/vision/imagenet/training_dumps/%08d/means/%03d.buffer", dump_id, i);
		fp = fopen(means_filepath, "wb");
		n_read = fwrite(cpu_param_buff, sizeof(float), (size_t) param_size, fp);
		fclose(fp);
		free(means_filepath);

		var_location = prev_grad_vars_locations[i];
		hipMemcpy(cpu_param_buff, var_location, param_size * sizeof(float), hipMemcpyDeviceToHost);
		print_ret = asprintf(&vars_filepath, "/mnt/storage/data/vision/imagenet/training_dumps/%08d/vars/%03d.buffer", dump_id, i);
		fp = fopen(vars_filepath, "wb");
		n_read = fwrite(cpu_param_buff, sizeof(float), (size_t) param_size, fp);
		fclose(fp);
		free(vars_filepath);

		free(cpu_param_buff);
	}
}


void dump_batch_norm_cache(Train_ResNet * trainer, char * filepath, Cache_BatchNorm * cache_batchnorm){

	FILE * fp;
	int n_wrote, print_ret;

	int input_size = cache_batchnorm -> input_size;
	int filters = cache_batchnorm -> feature_size;

	char * filepath_new = NULL;

	print_ret = asprintf(&filepath_new, "%smeans.buffer", filepath);
	float * cpu_means = (float *) malloc(filters * sizeof(float));
	hipMemcpy(cpu_means, cache_batchnorm -> means, filters * sizeof(float), hipMemcpyDeviceToHost);
	fp = fopen(filepath_new, "wb");
	n_wrote = fwrite(cpu_means, sizeof(float), filters, fp);
	fclose(fp);
	free(cpu_means);
	free(filepath_new);

	print_ret = asprintf(&filepath_new, "%svars.buffer", filepath);
	float * cpu_vars = (float *) malloc(filters * sizeof(float));
	hipMemcpy(cpu_vars, cache_batchnorm -> vars, filters * sizeof(float), hipMemcpyDeviceToHost);
	fp = fopen(filepath_new, "wb");
	n_wrote = fwrite(cpu_vars, sizeof(float), filters, fp);
	fclose(fp);
	free(cpu_vars);
	free(filepath_new);
}

void dump_conv_block_activation(int dump_id, Train_ResNet * trainer, Activation_ConvBlock * activation_conv_block, int conv_block_ind, bool is_deriv){
	FILE * fp;
	int n_wrote, print_ret;

	char * filepath = NULL;

	if (is_deriv){
		print_ret = asprintf(&filepath, "/mnt/storage/data/vision/imagenet/training_dumps/%08d/activation_derivs/conv_blocks/%02d/", dump_id, conv_block_ind);
	}
	else{
		print_ret = asprintf(&filepath, "/mnt/storage/data/vision/imagenet/training_dumps/%08d/activations/conv_blocks/%02d/", dump_id, conv_block_ind);
	}

	char * filepath_dup = NULL;
	
	char * batchnorm_filepath = NULL;
	if (is_deriv){
		print_ret = asprintf(&batchnorm_filepath, "/mnt/storage/data/vision/imagenet/training_dumps/%08d/activation_derivs/batch_norms/%02d/", dump_id, conv_block_ind);
	}
	else{
		print_ret = asprintf(&batchnorm_filepath, "/mnt/storage/data/vision/imagenet/training_dumps/%08d/activations/batch_norms/%02d/", dump_id, conv_block_ind);
	}

	char * batchnorm_filepath_dup = NULL; 

	int batch_size = trainer -> batch_size;
	int incoming_spatial_dim = activation_conv_block -> incoming_spatial_dim;
	int reduced_depth = activation_conv_block -> reduced_depth;
	int expanded_depth = activation_conv_block -> expanded_depth;
	int stride = activation_conv_block -> stride;


	/* REDUCTION CONV APPLIED */
	int reduction_size = incoming_spatial_dim * incoming_spatial_dim * reduced_depth * batch_size;
	float * cpu_reduction_applied = (float *) malloc(reduction_size * sizeof(float));
	hipMemcpy(cpu_reduction_applied, activation_conv_block -> post_reduced, reduction_size * sizeof(float), hipMemcpyDeviceToHost);
	print_ret = asprintf(&filepath_dup, "%sreduction_applied.buffer", filepath);
	fp = fopen(filepath_dup, "wb");
	n_wrote = fwrite(cpu_reduction_applied, sizeof(float), reduction_size, fp);
	fclose(fp);
	free(filepath_dup);
	free(cpu_reduction_applied);



	/* REDUCTION BATCH NORM */
	print_ret = asprintf(&batchnorm_filepath_dup, "%sreduced/", batchnorm_filepath);
	dump_batch_norm_cache(trainer, batchnorm_filepath_dup, activation_conv_block -> norm_post_reduced);
	free(batchnorm_filepath_dup);


	/* SPATIAL CONV APPLIED */
	int spatial_size = incoming_spatial_dim * incoming_spatial_dim * reduced_depth * batch_size / (stride * stride);
	float * cpu_spatial_applied = (float *) malloc(spatial_size * sizeof(float));
	hipMemcpy(cpu_spatial_applied, activation_conv_block -> post_spatial, spatial_size * sizeof(float), hipMemcpyDeviceToHost);
	print_ret = asprintf(&filepath_dup, "%sspatial_applied.buffer", filepath);
	fp = fopen(filepath_dup, "wb");
	n_wrote = fwrite(cpu_spatial_applied, sizeof(float), spatial_size, fp);
	fclose(fp);
	free(filepath_dup);
	free(cpu_spatial_applied);


	/* SPATIAL BATCH NORM */
	print_ret = asprintf(&batchnorm_filepath_dup, "%sspatial/", batchnorm_filepath);
	dump_batch_norm_cache(trainer, batchnorm_filepath_dup, activation_conv_block -> norm_post_spatial);
	free(batchnorm_filepath_dup);


	/* EXPANDED CONV APPLIED */
	int expanded_size = incoming_spatial_dim * incoming_spatial_dim * expanded_depth * batch_size / (stride * stride);
	float * cpu_expanded_applied = (float *) malloc(expanded_size * sizeof(float));
	hipMemcpy(cpu_expanded_applied, activation_conv_block -> post_expanded, expanded_size * sizeof(float), hipMemcpyDeviceToHost);
	print_ret = asprintf(&filepath_dup, "%sexpanded_applied.buffer", filepath);
	fp = fopen(filepath_dup, "wb");
	n_wrote = fwrite(cpu_expanded_applied, sizeof(float), expanded_size, fp);
	fclose(fp);
	free(filepath_dup);
	free(cpu_expanded_applied);

	/* POST EXPANDED NORM */
	print_ret = asprintf(&batchnorm_filepath_dup, "%sexpanded/", batchnorm_filepath);
	dump_batch_norm_cache(trainer, batchnorm_filepath_dup, activation_conv_block -> norm_post_expanded);
	free(batchnorm_filepath_dup);

	/* (TRANSFORMED) RESIDUAL */

	// only blocks with projection weights haved a transformed residual. otherwise identity to input
	if (activation_conv_block -> transformed_residual) {
		float * cpu_residual = (float *) malloc(expanded_size * sizeof(float));
		hipMemcpy(cpu_residual, activation_conv_block -> transformed_residual, expanded_size * sizeof(float), hipMemcpyDeviceToHost);
		print_ret = asprintf(&filepath_dup, "%stransformed_residual.buffer", filepath);
		fp = fopen(filepath_dup, "wb");
		n_wrote = fwrite(cpu_residual, sizeof(float), expanded_size, fp);
		fclose(fp);
		free(filepath_dup);
		free(cpu_residual);

		print_ret = asprintf(&batchnorm_filepath_dup, "%sprojected/", batchnorm_filepath);
		dump_batch_norm_cache(trainer, batchnorm_filepath_dup, activation_conv_block -> norm_post_projection);
		free(batchnorm_filepath_dup);

	}

	/* EXPANDED + RESIDUAL, and Activated */
	float * cpu_combined_output = (float *) malloc(expanded_size * sizeof(float));
	hipMemcpy(cpu_combined_output, activation_conv_block -> output_activated, expanded_size * sizeof(float), hipMemcpyDeviceToHost);
	print_ret = asprintf(&filepath_dup, "%soutput_activated.buffer", filepath);
	fp = fopen(filepath_dup, "wb");
	n_wrote = fwrite(cpu_combined_output, sizeof(float), expanded_size, fp);
	fclose(fp);
	free(filepath_dup);
	free(cpu_combined_output);

}

void dump_activations(int dump_id, Train_ResNet * trainer, Activations * activations, bool is_deriv){

	size_t batch_size = trainer -> batch_size;
	Dims * dims = trainer -> model -> dims;

	char * filepath = NULL;
	FILE * fp;
	int n_wrote, print_ret;

	// input
	size_t input_size = trainer -> cur_batch -> image_size * batch_size;
	if (!is_deriv){
		float * cpu_images = (float *) malloc(input_size * sizeof(float));
		hipMemcpy(cpu_images, trainer -> cur_batch -> images, input_size * sizeof(float), hipMemcpyDeviceToHost);
		print_ret = asprintf(&filepath, "/mnt/storage/data/vision/imagenet/training_dumps/%08d/activations/input.buffer", dump_id);
		fp = fopen(filepath, "wb");
		n_wrote = fwrite(cpu_images, sizeof(float), input_size, fp);
		fclose(fp);
		free(cpu_images);
		free(filepath);
	}


	/* 1. INIT CONV */

	size_t init_conv_applied_size = batch_size * dims -> init_conv_filters * (dims -> input / dims -> init_conv_stride) * (dims -> input / dims -> init_conv_stride);
	float * cpu_init_conv_applied = (float *) malloc(init_conv_applied_size * sizeof(float));
	hipMemcpy(cpu_init_conv_applied, activations -> init_conv_applied, init_conv_applied_size * sizeof(float), hipMemcpyDeviceToHost);
	if (is_deriv){
		print_ret = asprintf(&filepath, "/mnt/storage/data/vision/imagenet/training_dumps/%08d/activation_derivs/init_conv_applied.buffer", dump_id);
	}
	else{
		print_ret = asprintf(&filepath, "/mnt/storage/data/vision/imagenet/training_dumps/%08d/activations/init_conv_applied.buffer", dump_id);
	}
	fp = fopen(filepath, "wb");
	n_wrote = fwrite(cpu_init_conv_applied, sizeof(float), init_conv_applied_size, fp);
	fclose(fp);
	free(filepath);
	free(cpu_init_conv_applied);


	/* 2. INIT BATCH NORM */
	if (is_deriv){
		print_ret = asprintf(&filepath, "/mnt/storage/data/vision/imagenet/training_dumps/%08d/activation_derivs/batch_norms/init/", dump_id);
	}
	else{
		print_ret = asprintf(&filepath, "/mnt/storage/data/vision/imagenet/training_dumps/%08d/activations/batch_norms/init/", dump_id);
	}

	dump_batch_norm_cache(trainer, filepath, activations -> norm_init_conv);
	free(filepath);

	/* 3. ACTIVATED BATCH NORM */
	float * cpu_init_conv_activated = (float *) malloc(init_conv_applied_size * sizeof(float));
	hipMemcpy(cpu_init_conv_activated, activations -> init_conv_activated, init_conv_applied_size * sizeof(float), hipMemcpyDeviceToHost);
	if (is_deriv){
		print_ret = asprintf(&filepath, "/mnt/storage/data/vision/imagenet/training_dumps/%08d/activation_derivs/init_conv_activated.buffer", dump_id);
	}
	else{
		print_ret = asprintf(&filepath, "/mnt/storage/data/vision/imagenet/training_dumps/%08d/activations/init_conv_activated.buffer", dump_id);
	}
	fp = fopen(filepath, "wb");
	n_wrote = fwrite(cpu_init_conv_activated, sizeof(float), init_conv_applied_size, fp);
	fclose(fp);
	free(filepath);
	free(cpu_init_conv_activated);

	/* 4. MAX POOL */
	size_t maxpool_size = init_conv_applied_size / (dims -> init_maxpool_stride * dims -> init_maxpool_stride);
	// max inds only populated on forward pass
	if (!is_deriv){
		int * cpu_max_inds = (int *) malloc(maxpool_size * sizeof(int));
		hipMemcpy(cpu_max_inds, activations -> max_inds, maxpool_size * sizeof(int), hipMemcpyDeviceToHost);
		print_ret = asprintf(&filepath, "/mnt/storage/data/vision/imagenet/training_dumps/%08d/activations/max_inds.buffer", dump_id);
		fp = fopen(filepath, "wb");
		n_wrote = fwrite(cpu_max_inds, sizeof(int), maxpool_size, fp);
		fclose(fp);
		free(filepath);
		free(cpu_max_inds);
	}

	float * cpu_init_convblock_input = (float *) malloc(maxpool_size * sizeof(float));
	hipMemcpy(cpu_init_convblock_input, activations -> init_convblock_input, maxpool_size * sizeof(float), hipMemcpyDeviceToHost);
	if (is_deriv){
		print_ret = asprintf(&filepath, "/mnt/storage/data/vision/imagenet/training_dumps/%08d/activation_derivs/init_convblock_input.buffer", dump_id);
	}
	else{
		print_ret = asprintf(&filepath, "/mnt/storage/data/vision/imagenet/training_dumps/%08d/activations/init_convblock_input.buffer", dump_id);
	}
	fp = fopen(filepath, "wb");
	n_wrote = fwrite(cpu_init_convblock_input, sizeof(float), maxpool_size, fp);
	fclose(fp);
	free(filepath);
	free(cpu_init_convblock_input);


	/* 5. CONV BLOCKS */
	int n_conv_blocks = activations -> n_conv_blocks;
	Activation_ConvBlock ** conv_blocks = activations -> activation_conv_blocks;
	Activation_ConvBlock * cur_conv_block;
	for (int i = 0; i < n_conv_blocks; i++){
		cur_conv_block = conv_blocks[i];
		dump_conv_block_activation(dump_id, trainer, cur_conv_block, i, is_deriv);
	}


	/* 6. FINAL AVG POOL */
	int final_avg_pool_size = dims -> final_depth * batch_size;
	float * cpu_final_avg_pool = (float *) malloc(final_avg_pool_size * sizeof(float));
	hipMemcpy(cpu_final_avg_pool, activations -> final_conv_output_pooled, final_avg_pool_size * sizeof(float), hipMemcpyDeviceToHost);
	if (is_deriv){
		print_ret = asprintf(&filepath, "/mnt/storage/data/vision/imagenet/training_dumps/%08d/activation_derivs/final_avg_pool.buffer", dump_id);
	}
	else{
		print_ret = asprintf(&filepath, "/mnt/storage/data/vision/imagenet/training_dumps/%08d/activations/final_avg_pool.buffer", dump_id);
	}
	fp = fopen(filepath, "wb");
	n_wrote = fwrite(cpu_final_avg_pool, sizeof(float), final_avg_pool_size, fp);
	fclose(fp);
	free(filepath);
	free(cpu_final_avg_pool);

	/* 7. Fully Connected Output */
	int output_size = dims -> output * batch_size;
	float * cpu_linear_output = (float *) malloc(output_size * sizeof(float));
	hipMemcpy(cpu_linear_output, activations -> linear_output, output_size * sizeof(float), hipMemcpyDeviceToHost);
	if (is_deriv){
		print_ret = asprintf(&filepath, "/mnt/storage/data/vision/imagenet/training_dumps/%08d/activation_derivs/fc_output.buffer", dump_id);
	}
	else{
		print_ret = asprintf(&filepath, "/mnt/storage/data/vision/imagenet/training_dumps/%08d/activations/fc_output.buffer", dump_id);
	}
	fp = fopen(filepath, "wb");
	n_wrote = fwrite(cpu_linear_output, sizeof(float), output_size, fp);
	fclose(fp);
	free(filepath);
	free(cpu_linear_output);


	/* 8. Softmax Prediction */
	float * cpu_softmax = (float *) malloc(output_size * sizeof(float));
	if (is_deriv){
		hipMemcpy(cpu_softmax, trainer -> backprop_buffer -> output_layer_deriv, output_size * sizeof(float), hipMemcpyDeviceToHost);
		print_ret = asprintf(&filepath, "/mnt/storage/data/vision/imagenet/training_dumps/%08d/activation_derivs/softmax.buffer", dump_id);
	}
	else{
		hipMemcpy(cpu_softmax, trainer -> forward_buffer -> pred, output_size * sizeof(float), hipMemcpyDeviceToHost);
		print_ret = asprintf(&filepath, "/mnt/storage/data/vision/imagenet/training_dumps/%08d/activations/softmax.buffer", dump_id);
	}
	fp = fopen(filepath, "wb");
	n_wrote = fwrite(cpu_softmax, sizeof(float), output_size, fp);
	fclose(fp);
	free(filepath);
	free(cpu_softmax);


	/* 9. Correct Classes */
	if (!is_deriv){
		int * correct_classes_cpu = trainer -> cur_batch -> correct_classes_cpu;
		print_ret = asprintf(&filepath, "/mnt/storage/data/vision/imagenet/training_dumps/%08d/activations/correct_classes.buffer", dump_id);
		fp = fopen(filepath, "wb");
		n_wrote = fwrite(correct_classes_cpu, sizeof(int), batch_size, fp);
		free(filepath);
		fclose(fp);
	}
}

void dump_trainer(int dump_id, Train_ResNet * trainer){

	/* DUMP PARAMETERS */
	dump_parameters(dump_id, trainer);
	
	/* DUMP FORWARD ACTIVATIONS */
	dump_activations(dump_id, trainer, trainer -> forward_buffer -> activations, false);

}


// takes in pointers to GPU memory
void check_errors(Train_ResNet * trainer, int param_size, float * model_location, float * grad_location, float * mean_location, float * var_location, int location_ind){

	float * cpu_param_model = (float *) malloc(param_size * sizeof(float));
	hipMemcpy(cpu_param_model, model_location, param_size * sizeof(float), hipMemcpyDeviceToHost);

	float * cpu_param_grad = (float *) malloc(param_size * sizeof(float));
	hipMemcpy(cpu_param_grad, grad_location, param_size * sizeof(float), hipMemcpyDeviceToHost);

	float * cpu_param_mean = (float *) malloc(param_size * sizeof(float));
	hipMemcpy(cpu_param_mean, mean_location, param_size * sizeof(float), hipMemcpyDeviceToHost);

	float * cpu_param_var = (float *) malloc(param_size * sizeof(float));
	hipMemcpy(cpu_param_var, var_location, param_size * sizeof(float), hipMemcpyDeviceToHost);

	for (int i = 0; i < param_size; i++){
		if ((isnan(cpu_param_model[i])) || (isnan(cpu_param_grad[i])) || (isnan(cpu_param_mean[i])) || (isnan(cpu_param_var[i]))
				|| (isinf(cpu_param_model[i])) || (isinf(cpu_param_grad[i])) || (isinf(cpu_param_mean[i])) || (isinf(cpu_param_var[i]))){
			printf("ERROR: nan or inf found at location: %d\n", location_ind);
			printf("Dumping data to id=99999999 and exiting...\n");
			dump_trainer(99999999, trainer);
			exit(1);
		}
	}

	free(cpu_param_model);
	free(cpu_param_grad);
	free(cpu_param_mean);
	free(cpu_param_var);
}

// doing ADAM optimizer
void update_parameters(Train_ResNet * trainer){
	
	size_t batch_size = (size_t) trainer -> batch_size;
	size_t image_size = (size_t) trainer -> cur_batch -> image_size;

	float learning_rate = trainer -> learning_rate;
	float weight_decay = trainer -> weight_decay;
	float base_mean_decay = trainer -> base_mean_decay;
	float base_var_decay = trainer -> base_var_decay;
	// update the running decays here...
	float cur_mean_decay = trainer -> cur_mean_decay * base_mean_decay;
	float cur_var_decay = trainer -> cur_var_decay * base_var_decay;
	float eps = trainer -> eps;
	int cur_epoch = trainer -> cur_epoch;

	Params * model_params = trainer -> model -> params;
	float ** model_params_locations = model_params -> locations;
	int * param_sizes = model_params -> sizes;
	int n_locations = model_params -> n_locations;

	// values calculated from backprop, will reset these before returning
	Params * current_gradients = trainer -> backprop_buffer -> param_derivs;
	float ** current_gradient_locations = current_gradients -> locations;
	
	// running history values that the optimizer needs, will update these before returning
	Params * prev_grad_means = trainer -> backprop_buffer -> prev_means;
	float ** prev_grad_means_locations = prev_grad_means -> locations;
	Params * prev_grad_vars = trainer -> backprop_buffer -> prev_vars;
	float ** prev_grad_vars_locations = prev_grad_vars -> locations;

	int param_size;
	float *model_location, *grad_location, * mean_location, * var_location;

	/* DUMP THE STATE OF TRAINING PROCESS! */
	// dumping every 10 batches, before update
	// also dump when nan or inf occurs (data dumped to id=99999999)
	int cur_dump_id = trainer -> cur_dump_id;
	// if (cur_dump_id % 1000 == 0){
	// 	printf("DUMPING TRAINER... (ID = %d)!\n\n", cur_dump_id);
	// 	dump_trainer(cur_dump_id, trainer);
	// }
	
	for (int i = n_locations - 1; i >= 0; i--){
		param_size = param_sizes[i];
		model_location = model_params_locations[i];
		grad_location = current_gradient_locations[i];
		mean_location = prev_grad_means_locations[i];
		var_location = prev_grad_vars_locations[i];

		check_errors(trainer, param_size, model_location, grad_location, mean_location, var_location, i);

		dim3 gridDimUpdate(ceil((float) param_size / MAX_THREAD_PER_BLOCK));
		dim3 blockDimUpdate(MAX_THREAD_PER_BLOCK);
		updateMeans <<< gridDimUpdate, blockDimUpdate >>> (param_size, grad_location, model_location, base_mean_decay, weight_decay, mean_location, i);
		updateVars <<< gridDimUpdate, blockDimUpdate >>> (param_size, grad_location, model_location, base_var_decay, weight_decay, var_location, i);
		updateParams <<< gridDimUpdate, blockDimUpdate >>> (param_size, model_location, mean_location, var_location, learning_rate, weight_decay, cur_mean_decay, cur_var_decay, eps, i);
	}


	

	/* RESET ALL VALUES TO 0 FOR NEXT PASS THROUGH BACKPROP */
	for (int i = 0; i < n_locations; i++){
		param_size = param_sizes[i];
		grad_location = current_gradient_locations[i];
		hipMemset(grad_location, 0, param_size * sizeof(float));
		// reset_forward_buffer(trainer);
		// reset_backward_buffer(trainer);
	}

	// reset images and classes before next hipMemcpy
	hipMemset(trainer -> cur_batch -> images, 0, batch_size * image_size * sizeof(float));
	hipMemset(trainer -> cur_batch -> correct_classes, 0, batch_size * sizeof(int));

	// change the current mean and var decay...
	trainer -> cur_mean_decay = cur_mean_decay;
	trainer -> cur_var_decay = cur_var_decay;
}


void testTranspose(){

	int orig_rows = 2048;
	int orig_cols = 1000;

	float * origMat_host = (float *) malloc(orig_rows * orig_cols * sizeof(float));
	for (int i = 0; i < orig_rows; i++){
		for (int j = 0; j < orig_cols; j++){
			origMat_host[i * orig_cols + j] = ((float)(rand())/(float)(RAND_MAX));
		}
	}

	float * devOrigMat;
	hipMalloc(&devOrigMat, orig_rows * orig_cols * sizeof(float));
	hipMemcpy(devOrigMat, origMat_host, orig_rows * orig_cols * sizeof(float), hipMemcpyHostToDevice);

	float * devTrans;
	hipMalloc(&devTrans, orig_cols * orig_rows * sizeof(float));

	dim3 gridDimTranspose(ceil((float) orig_rows / TILE_WIDTH), ceil((float) orig_cols / TILE_WIDTH));
	dim3 blockDimTranspose(TILE_WIDTH, TILE_WIDTH);
	transpose <<< gridDimTranspose, blockDimTranspose >>> (devOrigMat, orig_rows, orig_cols, devTrans);

	float *matTrans_host = (float *) malloc(orig_cols * orig_rows * sizeof(float));

	hipMemcpy(matTrans_host, devTrans, orig_cols * orig_rows * sizeof(float), hipMemcpyDeviceToHost);

	hipFree(devOrigMat);
	hipFree(devTrans);

	for (int i = 0; i < orig_cols; i++){
		for (int j = 0; j < orig_rows; j++){
			if (origMat_host[j * orig_cols + i] != matTrans_host[i * orig_rows + j]){
				printf("TRANSPOSE ERROR: @ original row: %d, original col: %d\n", j, i);
			}
		}
	}

	free(origMat_host);
	free(matTrans_host);
}


void testMatMul(){

	int m = 32;
	int k = 2048;
	int n = 1000;

	float * A_host = (float *) malloc(m * k * sizeof(float));
	float * B_host = (float *) malloc(k * n * sizeof(float));
	float * C_host = (float *) calloc(m * n, sizeof(float));

	for (int i = 0; i < m; i++){
		for (int j = 0; j < k; j++){
			A_host[i * k + j] = ((float)(rand())/(float)(RAND_MAX)) * (((int)(rand()) % 2) * 2 - 1);
		}
	}

	for (int i = 0; i < k; i++){
		for (int j = 0; j < n; j++){
			B_host[i * n + j] = ((float)(rand())/(float)(RAND_MAX))  * (((int)(rand()) % 2) * 2 - 1);
		}
	}

	for (int i = 0; i < m; i++){
		for (int j = 0; j < n; j++){
			for (int c = 0; c < k; c++){
				C_host[i * n + j] += A_host[i * k + c] * B_host[c * n + j];
			}
		}
	}

	float * A_dev, *B_dev, *C_dev;
	hipMalloc(&A_dev, m * k * sizeof(float));
	hipMemcpy(A_dev, A_host, m * k * sizeof(float), hipMemcpyHostToDevice);

	hipMalloc(&B_dev, k * n * sizeof(float));
	hipMemcpy(B_dev, B_host, k * n * sizeof(float), hipMemcpyHostToDevice);


	hipMalloc(&C_dev, m * n * sizeof(float));


	dim3 gridDimMatMul(ceil((float) m / TILE_WIDTH), ceil((float) n / TILE_WIDTH));
	dim3 blockDimMatMul(TILE_WIDTH, TILE_WIDTH);

	matMul <<< gridDimMatMul, blockDimMatMul >>> (A_dev, B_dev, m, k, n, C_dev);

	float * C_kern_result = (float *) malloc(m * n * sizeof(float));

	float eps = 0.00001;

	hipMemcpy(C_kern_result, C_dev, m * n * sizeof(float), hipMemcpyDeviceToHost);

	float cpu_val, gpu_val;
	for (int i = 0; i < m; i++){
		for (int j = 0; j < n; j++){
			gpu_val = C_kern_result[i * n + j];
			cpu_val = C_host[i * n + j];
			if ( (gpu_val < (cpu_val - eps)) || (gpu_val > (cpu_val + eps)) ){
				printf("MatMul ERROR: @ row: %d, col: %d\n", j, i);
				printf("CPU Result: %f\n", cpu_val);
				printf("GPU Result: %f\n\n", gpu_val);
			}
		}
	}

	hipFree(A_dev);
	hipFree(B_dev);
	hipFree(C_dev);

	free(A_host);
	free(B_host);
	free(C_host);
	free(C_kern_result);

}

void testConvolution(int in_spatial_dim, int kern_dim, int in_filters, int out_filters,  int stride, int batch_size, 
																float * input, float * weights, float * output){

	printf("\n\n* TESTING THE CONVOLUTION KERNEL *\n\n");
	/* FIRST DO COMPUTATION ON GPU */

	int out_spatial_dim = in_spatial_dim / stride;
	int out_filters_block = min(MAX_THREAD_PER_BLOCK / batch_size, out_filters);
	int out_filters_grid = max(1, (int) ceil((float) out_filters / (float) out_filters_block));

	printf("Conv Params -- Batch Size: %d, In Spatial: %d, Stride: %d, Kern Dim: %d, In Filters: %d, Out Filters %d\n", batch_size, in_spatial_dim, stride, kern_dim, in_filters, out_filters);
	printf("Launch Params -- Out Filters Block: %d, Out Filters Grid: %d\n", out_filters_block, out_filters_grid);
	dim3 gridDimConv(out_spatial_dim, out_spatial_dim, out_filters_grid);
	dim3 blockDimConv(batch_size, out_filters_block);

	printf("Computing Convolution on GPU...\n");

	doConvolution <<< gridDimConv, blockDimConv>>> (input, weights, in_spatial_dim, kern_dim, in_filters, out_filters, stride, batch_size, output);

	hipDeviceSynchronize();

	float * gpu_output_on_cpu = (float *) malloc(batch_size * out_spatial_dim * out_spatial_dim * out_filters * sizeof(float));

	hipMemcpy(gpu_output_on_cpu, output, batch_size * out_spatial_dim * out_spatial_dim * out_filters * sizeof(float), hipMemcpyDeviceToHost);

	/* DO COMPUTATION ON CPU */
	
	// COPYING VALUES FROM GPU TO THE CPU...
	float * input_cpu = (float *) malloc(batch_size * in_spatial_dim * in_spatial_dim * in_filters * sizeof(float));
	hipMemcpy(input_cpu, input, batch_size * in_spatial_dim * in_spatial_dim * in_filters * sizeof(float), hipMemcpyDeviceToHost);

	float * weights_cpu = (float *) malloc(kern_dim * kern_dim * in_filters * out_filters * sizeof(float));
	hipMemcpy(weights_cpu, weights, kern_dim * kern_dim * in_filters * out_filters * sizeof(float), hipMemcpyDeviceToHost);

	float * cpu_output = (float *) malloc(batch_size * out_spatial_dim * out_spatial_dim * out_filters * sizeof(float));

	printf("Computing Convolution on CPU...\n");

	int output_ind, in_spatial_row_start, in_spatial_col_start, in_spatial_row, in_spatial_col, input_ind, kernel_ind;
	int half_kernel_dim = kern_dim / 2;
	int kernel_size = in_filters * kern_dim * kern_dim;
	float in_spatial_val;
	for (int samp = 0; samp < batch_size; samp++){
		for (int out_filt = 0; out_filt < out_filters; out_filt++){
			for (int out_i = 0; out_i < out_spatial_dim; out_i++){
				for (int out_j = 0; out_j < out_spatial_dim; out_j++){
					output_ind = out_spatial_dim * out_spatial_dim * out_filters * samp + out_spatial_dim * out_filters * out_i + out_filters * out_j + out_filt;
					cpu_output[output_ind] = 0;
					in_spatial_row_start = out_i * stride;
					in_spatial_col_start = out_j * stride;
					for (int in_filt = 0; in_filt < in_filters; in_filt++){
						for (int row_offset = -half_kernel_dim; row_offset <= half_kernel_dim; row_offset++){
							for (int col_offset = -half_kernel_dim; col_offset <= half_kernel_dim; col_offset++){
								// compute spatial value
								in_spatial_row = in_spatial_row_start + row_offset;
								in_spatial_col = in_spatial_col_start + col_offset;
								input_ind = in_spatial_dim * in_spatial_dim * in_filters * samp + in_spatial_dim * in_filters * in_spatial_row + in_filters * in_spatial_col + in_filt;
								kernel_ind = kern_dim * in_filters * (row_offset + half_kernel_dim) + in_filters * (col_offset + half_kernel_dim) + in_filt;
								if ((in_spatial_row < 0) || (in_spatial_row >= in_spatial_dim) || (in_spatial_col < 0) || (in_spatial_col >= in_spatial_dim)) {
									in_spatial_val = 0;
								}
								else{
									in_spatial_val = input_cpu[input_ind];
								}
								// multiply with conv weight
								// threadIdx.x specifies the output filter id
								// kernel_ind specifies the (x, y, input_channel)
								cpu_output[output_ind] += weights_cpu[out_filt * kernel_size + kernel_ind] * in_spatial_val;
							}
						}
					}
				}
			}
		}
	}

	/* COMPARE RESULTS */
	float gpu_val;
	float cpu_val;
	float eps = 0.0001;
	int err_cnt = 0;
	for (int samp = 0; samp < batch_size; samp++){
		for (int filt = 0; filt < out_filters; filt++){
			for (int i = 0; i < out_spatial_dim; i++){
				for (int j = 0; j < out_spatial_dim; j++){
					output_ind = out_spatial_dim * out_spatial_dim * out_filters * samp + out_spatial_dim * out_filters * i + out_filters * j + filt;
					gpu_val = gpu_output_on_cpu[output_ind];
					cpu_val = cpu_output[output_ind];
					if ( (gpu_val < (cpu_val - eps)) || (gpu_val > (cpu_val + eps)) ){
						printf("ERROR: GPU VALUE DIFFERS FROM CPU\n");
						printf("Occurs at:\nSamp: %d\nFilter: %d\nRow: %d\nCol: %d\n", samp, filt, i, j);
						printf("GPU Value:%f vs. CPU Value:%f\n\n", gpu_val, cpu_val);
						err_cnt++;
					}
					if (err_cnt == 10){
						exit(1);
					}
				}
			}
		}
	}

	/* FREE UP STUFF */

	free(gpu_output_on_cpu);
	free(input_cpu);
	free(weights_cpu);
	free(cpu_output);	

}



int main(int argc, char *argv[]) {

	bool debug = false;

	if (debug){
		testMatMul();
		testTranspose();
		return 0;
	}

	int N_CLASSES = 1000;
	
	// GETTING CLASS METADETA
	char * LABEL_FILENAME = (char *) "/mnt/storage/data/vision/imagenet/2012/id_to_label_mapping.txt";
	char * SYNSET_FILENAME = (char *) "/mnt/storage/data/vision/imagenet/2012/id_to_synset_mapping.txt";
	char * COUNTS_FILENAME = (char *) "/mnt/storage/data/vision/imagenet/2012/id_to_img_count_mapping.txt";
	Class_Metadata * class_metadata = populate_class_info(LABEL_FILENAME, SYNSET_FILENAME, COUNTS_FILENAME, N_CLASSES);
	int total_images = 0;
	for (int i = 0; i < N_CLASSES; i++){
		total_images += (class_metadata -> counts)[i];
	}

	// DEFINING MODEL DIMENSIONS
	int INPUT_DIM = 224;
	int INIT_KERNEL_DIM = 7;
	int INIT_CONV_FILTERS = 64;
	int INIT_CONV_STRIDE = 2;
	int INIT_MAXPOOL_DIM = 3;
	int INIT_MAXPOOL_STRIDE = 2;
	int N_CONV_BLOCKS = 16;
	int * IS_BLOCK_SPATIAL_REDUCTION = (int *) calloc(N_CONV_BLOCKS, sizeof(int));
	// transitions between spatial 56 -> 28 -> 14 -> 7
	// transitions between output depth of 256 -> 512 -> 1024 -> 2048
	int FINAL_DEPTH = 2048;
	IS_BLOCK_SPATIAL_REDUCTION[3] = 1;
	IS_BLOCK_SPATIAL_REDUCTION[7] = 1;
	IS_BLOCK_SPATIAL_REDUCTION[13] = 1;
	Dims * dims = init_dimensions(INPUT_DIM, INIT_KERNEL_DIM, INIT_CONV_FILTERS, INIT_CONV_STRIDE, INIT_MAXPOOL_DIM, INIT_MAXPOOL_STRIDE,
									N_CONV_BLOCKS, IS_BLOCK_SPATIAL_REDUCTION, FINAL_DEPTH, N_CLASSES);


	// declaring curandGenerator
	hiprandGenerator_t gen;
	// INITIALIZING RANDOM NUMBER GENERATOR USED TO INIT WEIGHTS
	hiprandStatus_t status_create = hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
	hiprandStatus_t status_set_seed = hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL);

	// INITIALIZING MODEL
	ResNet * model = init_resnet(dims, &gen);


	// INITIALIZING TRAINING

	// Batch Structure (will be modified every iteration of every epoch)
	
	// given when we generated shards...
	int SHARD_N_IMAGES = 32768;

	int BATCH_SIZE = 128;
	// dimensions of INPUT_DIM X INPUT_DIM x 3 color channels
	int IMAGE_SIZE = INPUT_DIM * INPUT_DIM * 3;
	Batch * batch = init_general_batch(BATCH_SIZE, IMAGE_SIZE, INPUT_DIM, SHARD_N_IMAGES);


	// General Training Structure (holds hyperparameters and pointers to structs which have network values)
	float LEARNING_RATE = 0.0001;
	float WEIGHT_DECAY = 0;
	float MEAN_DECAY = 0.9;
	float VAR_DECAY = 0.999;
	float EPS = 0.0000001;
	float N_EPOCHS = 40;

	Train_ResNet * trainer = init_trainer(model, batch, BATCH_SIZE, LEARNING_RATE, WEIGHT_DECAY, MEAN_DECAY, VAR_DECAY, EPS, N_EPOCHS, total_images);
	

	/* PERFORM TRAINING */


	int iterations_per_epoch = ceil((float) total_images / BATCH_SIZE);

	float *pred;
	int * correct;
	float epoch_n_wrong, batch_n_wrong;
	float epoch_loss, batch_loss, avg_batch_loss, epoch_accuracy, batch_accuracy, val_pred_correct;
	float total_images_per_epoch = BATCH_SIZE * iterations_per_epoch;

	int PRINT_FREQ = 1;

	hipError_t status;

	char * LOSS_FILENAME = (char *) "/mnt/storage/data/vision/imagenet/training_dumps/avg_loss_log.txt";
	FILE * loss_file = fopen(LOSS_FILENAME, "w");

	for (int epoch = 0; epoch < N_EPOCHS; epoch++){
		epoch_loss = 0;
		epoch_n_wrong = 0;
		for (int iter = 0; iter < iterations_per_epoch; iter++){

			printf("************\n");

			/* LOAD NEW BATCH */
			printf("Loading Batch...\n");
			// values go into trainer -> cur_batch -> [images_cpu|images_float_cpu|images|correct_classes_cpu|correct_classes]
			load_new_batch(trainer, class_metadata, trainer -> cur_batch);

			hipDeviceSynchronize();
			status = hipGetLastError();
			//printf("Status after loading batch: %s\n\n", hipGetErrorString(status));
			

			/* DO FORWARD PROP */
			// final predictions go into trainer -> forward_buffer -> [pred|pred_cpu|prediction_label]
			printf("Making Predictions...\n");
			forward_pass(trainer);

			hipDeviceSynchronize();
			status = hipGetLastError();
			//printf("Status after forward pass: %s\n\n", hipGetErrorString(status));
			

			/* RECORD LOSS AND ACCURACY */

			// dimensions of pred: (BATCH_SIZE, N_CLASSES)
			pred = trainer -> forward_buffer -> pred_cpu;
			correct = trainer -> cur_batch -> correct_classes_cpu;
			
			// loss
			batch_loss = 0;
			for (int s = 0; s < BATCH_SIZE; s++){
				batch_loss += -1 * logf(pred[s * N_CLASSES + correct[s]]);
			}
			avg_batch_loss = batch_loss / BATCH_SIZE;
			epoch_loss += batch_loss;

			// accuracy
			batch_n_wrong = 0;
			for (int s = 0; s < BATCH_SIZE; s++){
				val_pred_correct = pred[s * N_CLASSES + correct[s]];
				for (int c = 0; c < N_CLASSES; c++){
					if ((c != correct[s]) && (pred[s * N_CLASSES + c] >= val_pred_correct)){
						batch_n_wrong++;
						break;
					}
				}
			}
			epoch_n_wrong += batch_n_wrong;
			batch_accuracy = 100 * ((float) BATCH_SIZE - batch_n_wrong) / ((float) BATCH_SIZE);

			if (iter % PRINT_FREQ == 0){
				printf("\nEpoch: %d, Batch: %d ----- Avg. Loss: %.4f, Accuracy: %.2f%%\n\n", epoch, iter, avg_batch_loss, batch_accuracy);
			}
			fprintf(loss_file, "%.4f\n", avg_batch_loss);
			fflush(loss_file);


			/* DO BACKPROP */
			printf("Backprop to Compute Derivs...\n");
			backwards_pass(trainer);

			hipDeviceSynchronize();
			status = hipGetLastError();
			//printf("Status after backwards pass: %s\n\n", hipGetErrorString(status));

			/* OPTIMIZE WEIGHTS */
			//printf("Applying Optimizer to Update Params...\n\n");
			update_parameters(trainer);

			hipDeviceSynchronize();
			status = hipGetLastError();
			//printf("Status after updating params: %s\n\n", hipGetErrorString(status));

		}

		(trainer -> loss_per_epoch)[epoch] = epoch_loss;
		epoch_accuracy = (total_images_per_epoch - epoch_n_wrong) / total_images_per_epoch;
		(trainer -> accuracy_per_epoch)[epoch] = epoch_accuracy;

		// reset batch to start from beginning of dataset
		trainer -> cur_batch -> cur_shard_id = -1;
		trainer -> cur_batch -> cur_batch_in_shard = -1;
		trainer -> cur_epoch += 1;

	}

	// DO A FINAL DUMP AFTER MODEL FINISHES (stored at 77777777)
	int FINAL_DUMP_ID = 77777777;
	dump_trainer(FINAL_DUMP_ID, trainer);

	fclose(loss_file);

}
