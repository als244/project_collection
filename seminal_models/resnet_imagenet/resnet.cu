#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hip/hip_runtime.h>
#include <stdint.h>

#include "resnet.h"

#define SM_COUNT 82
#define WARP_PER_SM 4
#define THREAD_PER_WARP 32
#define MAX_THREAD_PER_BLOCK 1024
#define TILE_WIDTH 32
#define BLOCK_ROWS 8
#define CUDA_BATCH_SIZE 32
#define MAX_SHARED_MEMORY 48000
#define MAX_SHARED_MEM_FLOATS 12000
#define MAX_THREAD_PER_BLOCK_INCL_REG 512




// used to hide all print statements for device data
#define TO_PRINT false

#define CUDA_CALL(x) do { if((x)!=hipSuccess) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__);\
    return EXIT_FAILURE;}} while(0)
#define CURAND_CALL(x) do { if((x) != HIPRAND_STATUS_SUCCESS) { \
	printf("Error at %s:%d\n",__FILE__,__LINE__);\
	return EXIT_FAILURE;}} while(0)


/* DECLARING FUNCTIONS HERE */
void testConvolution(int in_spatial_dim, int kern_dim, int in_filters, int out_filters,  int stride, int batch_size, 
																float * input, float * weights, float * biases, float * output);


/* START OF KERNELS/FUNCTIONS */

__global__ void setVal(int size, float val, float *out){
 	int ind = blockDim.x * blockIdx.x + threadIdx.x;
 	if (ind >= size){
 		return;
 	}
 	out[ind] = val;
}

void init_weights_gaussian_device(hiprandGenerator_t * gen, int size, float *X, float mean, float var){
 	float stddev = sqrtf(var);
 	hiprandStatus_t status = hiprandGenerateNormal(*gen, X, (size_t) size, mean, stddev);
 }

// RANDOM NUMBER GENERATOR ON DEVICE CAN'T USE C LIBRARY RAND(), so use cuRAND() library instead...
// __global__ void sample_gaussian(int size, float *X, float mean, float var) {
// 	int i = blockIdx.x * blockDim.x + threadIdx.x;
// 	if (i >= size){
// 		return;
// 	}
// 	if (var == 0){
// 		X[i] = mean;
// 		return;
// 	}
// 	float x = (float)rand() / RAND_MAX;
//   	float y = (float)rand() / RAND_MAX;
//   	float z = sqrtf(-2 * logf(x)) * cosf(2 * M_PI * y);
//   	float std = sqrtf(var);
//   	float val = std * z + mean;
//   	X[i] = val;
// }

// ASSUME 1-D launch
__global__ void addVec(int size, float * A, float * B, float * out){
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= size){
		return;
	}
	out[i] = A[i] + B[i];
}

// GRID has dim (ROWS / TILE_WIDTH, COLS/TILE_WIDTH)
// each BLOCK has dim (TILE_WIDTH, TILE_WIDTH)
__global__ void matMulOptimized(const float *M, const float *N, int m, int k, int n, float *out){
	__shared__ float M_tile[TILE_WIDTH][TILE_WIDTH + 1];
	__shared__ float N_tile[TILE_WIDTH][TILE_WIDTH + 1];

	int block_x = blockIdx.x;
	int block_y = blockIdx.y;

	int thread_x = threadIdx.x;
	int thread_y = threadIdx.y;

	int row_ind = block_x * TILE_WIDTH + thread_x;
	int col_ind = block_y * TILE_WIDTH + thread_y;

	if (row_ind >= m || col_ind >= n){
		return;
	}

	float val = 0;
	for (int phase = 0; phase < ceil((float) k / float(TILE_WIDTH)); phase++) {
		if (phase * TILE_WIDTH + thread_y < k){
			M_tile[thread_x][thread_y] = M[row_ind * k + phase * TILE_WIDTH + thread_y];
		}
		else{
			M_tile[thread_x][thread_y] = 0;
		}
		if (phase * TILE_WIDTH + thread_x < k){
			N_tile[thread_x][thread_y] = N[(phase * TILE_WIDTH + thread_x) * n + col_ind];
		}
		else{
			N_tile[thread_x][thread_y] = 0;
		}

		__syncthreads();

		for (int t = 0; t < TILE_WIDTH; t++){
			val += M_tile[thread_x][t] * N_tile[t][thread_y];
		}
		__syncthreads();
	}
	out[row_ind * n + col_ind] = val;
}


// GRID has dim (ROWS / TILE_WIDTH, COLS/TILE_WIDTH)
// each BLOCK has dim (TILE_WIDTH, TILE_WIDTH)
__global__ void matMul(const float *M, const float *N, int m, int k, int n, float *out){

	
	int row_ind = blockIdx.x * TILE_WIDTH + threadIdx.x;
	int col_ind = blockIdx.y * TILE_WIDTH + threadIdx.y;

	if (row_ind >= m || col_ind >= n){
		return;
	}

	float val = 0;
	for (int z = 0; z < k; z++){
		val += M[row_ind * k + z] * N[z * n + col_ind];
	}
	out[row_ind * n + col_ind] = val;
}


// unoptimized transpose because used rarely...

// grid has dim (ROWS / TILE_WIDTH, COLS/TILE_WIDTH)
// each BLOCK has dim (TILE_WIDTH , BLOCK_ROWS) = # of threads
__global__ void transposeSharedMem(const float *in, int rows, int cols, float * out) {
  __shared__ float tile[TILE_WIDTH][TILE_WIDTH + 1];

  int row_ind = blockIdx.x * TILE_WIDTH + threadIdx.y;
  int col_ind = blockIdx.y * TILE_WIDTH + threadIdx.x;
  
  
  if (col_ind >= cols || row_ind >= rows){
  	return;
  }

  
  // each thread needs to load TILE_WIDTH / BLOCK_ROWS values
  int row_boundary = min(TILE_WIDTH, rows - row_ind);
  for (int j = 0; j < row_boundary; j += BLOCK_ROWS){
     tile[threadIdx.y + j][threadIdx.x] = in[(row_ind+j)*cols + col_ind];
  }

  __syncthreads();

  int col_boundary = min(TILE_WIDTH, cols - col_ind);
  for (int j = 0; j < col_boundary; j += BLOCK_ROWS){
     out[col_ind*rows + row_ind + j] = tile[threadIdx.y + j][threadIdx.x];
  }
}

// grid has dim (ROWS / TILE_WIDTH, COLS/TILE_WIDTH)
// each BLOCK has dim (TILE_WIDTH , TILE_WIDTH) = # of threads
__global__ void transpose(const float *in, int rows, int cols, float * out) {

  int row_ind = blockIdx.x * TILE_WIDTH + threadIdx.x;
  int col_ind = blockIdx.y * TILE_WIDTH + threadIdx.y;
  
  
  if (col_ind >= cols || row_ind >= rows){
  	return;
  }

  out[col_ind * rows + row_ind] = in[row_ind * cols + col_ind];
}


// TRIED TO OPTIMIZE: MAKE SURE THIS WORKS WITH SHARED MEM AND LAUNCH SPECS (FORGOT HOW I DID IT...)
// 48KB is maximum value for shared memory, passed into this kernel as third param <<< gridDim, blockDim, SHARED_MEM_BYTES >>>
// launch grid dimensions as (OUT_SPATIAL_DIM, OUT_SPATIAL_DIM, OUT_FILTER_CHUNK) blocks, and launch with block dim as (out_filt_rows_shared, sub_batch) threads
// thus 12k floats is max for shared memory per block
// first get as many output filter weights in shared memory as possible, but have separate blocks working on different chunks (OUT_FILTER_CHUNK * out_filt_rows_shared = out_filt)
// then stream samples in batch to compute output value for each sample and output filter. Eac sub_batch will have batch_size / dim(sub_batch) samples to go over
// __global__ void doConvolutionOptimized(const float * input, const float * weights, const float * biases, int spatial_dim, int kern_dim, int in_filters, int out_filters, int stride, int batch_size, float * out){

// 	// will consist of (shared_out_filt_rows X (kern_dim^2 * in_filt) conv_weight matrix
// 	extern __shared__ float shared_mem[];


// 	// (Calling "Kernel" a 3-D obj of weights where there is 2-D conv filter for each input channel)
// 	int kernel_size = (kern_dim * kern_dim * in_filters);

// 	int spatial_row_start = stride * blockIdx.x;
// 	int spatial_col_start = stride * blockIdx.y;
// 	int out_spatial_dim = spatial_dim / stride;

// 	int half_kernel_dim = kern_dim / 2;
// 	int out_filter_id, spatial_row, spatial_col;
// 	float out_val, spatial_val;
// 	out_filter_id = blockIdx.z * blockDim.x + threadIdx.x;
// 	if (out_filter_id >= out_filters){
// 		return;
// 	}

// 	for (int j = 0; j < kernel_size; j++){
// 		shared_mem[threadIdx.x * kernel_size + j] = weights[out_filter_id * kernel_size + j];
// 	}

// 	int samp_per_subbatch = ceil((float) batch_size / blockDim.y);
// 	int samp_start = samp_per_subbatch * threadIdx.y;
// 	int samp_end = min(batch_size, samp_start + samp_per_subbatch);
// 	int kernel_ind;
// 	// probably could be more efficient by reducing number of output filters in shared mem, and adding tiled spatial....
// 	for (int sample_ind = samp_start; sample_ind < samp_end; sample_ind++){
// 		out_val = 0;
// 		for (int row_offset = -half_kernel_dim; row_offset <= half_kernel_dim; row_offset++){
// 			for (int col_offset = -half_kernel_dim; col_offset <= half_kernel_dim; col_offset++){
// 				for (int channel = 0; channel < in_filters; channel++){
						
// 					// compute spatial value
// 					spatial_row = spatial_row_start + row_offset;
// 					spatial_col = spatial_col_start + col_offset;
// 					kernel_ind = kern_dim * in_filters * (row_offset + half_kernel_dim) + in_filters * (col_offset + half_kernel_dim) + channel;
// 					if ((spatial_row < 0) || (spatial_row >= spatial_dim) || (spatial_col < 0) || (spatial_col >= spatial_dim)) {
// 						spatial_val = 0;
// 					}
// 					else{
// 						spatial_val = input[spatial_dim * spatial_dim * in_filters * sample_ind + spatial_dim * in_filters * spatial_row + in_filters * spatial_col + channel];
// 					}

// 					// multiply with conv weight
// 					// threadIdx.x specifies the output filter id
// 					// kernel_ind specifies the (x, y, input_channel)
// 					out_val += shared_mem[threadIdx.x * kernel_size + kernel_ind] * spatial_val;
// 				}
// 			}
// 		}
// 		out[out_spatial_dim * out_spatial_dim * out_filters * sample_ind + out_spatial_dim * out_filters * blockIdx.x + out_filters * blockIdx.y + out_filter_id] = out_val + biases[out_filter_id];
// 	}
// }


// FOR NOW KEEP NAIVE (UN-OPTIMIZED)...
// not bothering with shared memory for now...

// Independent over (output_filter_id, output_spatial_row, output_spatial_col, sample)
// Launch with gridDim (out_spatial_dim, out_spatial_dim, max(1, out_filters / (MAX_THREAD_PER_BLOCK)) and blockDim (batch_size, min(MAX_THREAD_PER_BLOCK / batch_size, output_filters))
// Room to optimize a lot...
__global__ void doConvolution(const float * input, const float * weights, const float * biases, int spatial_dim, int kern_dim, int in_filters, int out_filters, int stride, int batch_size, float * out){

	int out_spatial_row = blockIdx.x;
	int out_spatial_col = blockIdx.y;
	int out_filter_id = blockIdx.z * blockDim.y + threadIdx.y;
	int sample_ind = threadIdx.x;
	int out_spatial_dim = spatial_dim / stride;

	// shoudn't need to check based on launch specs but will anyways
	if ((out_filter_id >= out_filters) || (sample_ind >= batch_size) || (out_spatial_row >= out_spatial_dim) || (out_spatial_col >= out_spatial_dim)) {
		return;
	}

	int in_spatial_row_start = stride * out_spatial_row;
	int in_spatial_col_start = stride * out_spatial_col;

	int half_kernel_dim = kern_dim / 2;
	int in_spatial_row, in_spatial_col, kernel_ind;
	
	// (Calling "Kernel" a 3-D obj of weights where there is 2-D conv filter for each input channel)
	int kernel_size = (kern_dim * kern_dim * in_filters);

	float out_val = 0;
	float in_spatial_val;
	for (int row_offset = -half_kernel_dim; row_offset <= half_kernel_dim; row_offset++){
		for (int col_offset = -half_kernel_dim; col_offset <= half_kernel_dim; col_offset++){
			for (int in_channel = 0; in_channel < in_filters; in_channel++){
						
				// compute spatial value
				in_spatial_row = in_spatial_row_start + row_offset;
				in_spatial_col = in_spatial_col_start + col_offset;
				kernel_ind = kern_dim * in_filters * (row_offset + half_kernel_dim) + in_filters * (col_offset + half_kernel_dim) + in_channel;
				if ((in_spatial_row < 0) || (in_spatial_row >= spatial_dim) || (in_spatial_col < 0) || (in_spatial_col >= spatial_dim)) {
					in_spatial_val = 0;
				}
				else{
					in_spatial_val = input[spatial_dim * spatial_dim * in_filters * sample_ind + spatial_dim * in_filters * in_spatial_row + in_filters * in_spatial_col + in_channel];
				}

				// multiply with conv weight
				// threadIdx.x specifies the output filter id
				// kernel_ind specifies the (x, y, input_channel)
				out_val += weights[out_filter_id * kernel_size + kernel_ind] * in_spatial_val;
			}
		}
	}
	//out[out_spatial_dim * out_spatial_dim * out_filters * sample_ind + out_spatial_dim * out_filters * out_spatial_row + out_filters * out_spatial_col + out_filter_id] = out_val + biases[out_filter_id];
	out[out_spatial_dim * out_spatial_dim * out_filters * sample_ind + out_spatial_dim * out_filters * out_spatial_row + out_filters * out_spatial_col + out_filter_id] = out_val;
}


// FOR NOW KEEP NAIVE (UN-OPTIMIZED)...
// not bothering with shared memory for now...

// Independent over (input filter, input_x, input_y, sample)
// could use shared memory over conv weights...
// Launch with gridDim (spatial_dim, spatial_dim, max(1, input_filters / (MAX_THREAD_PER_BLOCK / batch_size))) and blockDim (batch_size, min(MAX_THREAD_PER_BLOCK / batch_size, input_filters))
// Can parallelize further with reductions, if want to optimize
__global__ void convolutionDerivInput(const float * input, const float * weights, const float * out_deriv, int spatial_dim, int kern_dim, int in_filters, int out_filters, int stride, int batch_size, bool toAdd,
											float * input_deriv){

	int spatial_row = blockIdx.x;
	int spatial_col = blockIdx.y;
	int in_filter_id = blockIdx.z * blockDim.y + threadIdx.y;
	int sample_ind = threadIdx.x;
	// shouldn't need to check based on launch specs, but will anyways...
	if ((spatial_row >= spatial_dim) || (spatial_col >= spatial_dim) || (in_filter_id >= in_filters) || (sample_ind >= batch_size)){
		return;
	}

	int out_spatial_dim = spatial_dim / stride;
	int half_kernel_dim = kern_dim / 2;
	int out_spatial_row_start = spatial_row / stride;
	int out_spatial_col_start = spatial_col / stride;
	int kern_ind, kern_row_ind, kern_col_ind, out_spatial_ind, out_spatial_row, out_spatial_col;
	int kernel_size = (kern_dim * kern_dim * in_filters);
	float out_spatial_val_deriv;
	float total_deriv = 0;
	for (int out_filt_id = 0; out_filt_id < out_filters; out_filt_id++){
		for (int row_offset = -half_kernel_dim; row_offset <= half_kernel_dim; row_offset++){
			for (int col_offset = -half_kernel_dim; col_offset <= half_kernel_dim; col_offset++){
				// compute output spatial value that used the input spatial value
				out_spatial_row = out_spatial_row_start + row_offset;
				out_spatial_col = out_spatial_col_start + col_offset;
				// index of output spatial val (iterate over samples in batch, then rows, then columns, then channels)
				out_spatial_ind = out_spatial_dim * out_spatial_dim * out_filters * sample_ind + out_spatial_dim * out_filters * out_spatial_row + out_filters * out_spatial_col + out_filt_id;

				// get kernel index used to generate out spatial value for corresponding input spatial value
				kern_row_ind = spatial_row - out_spatial_row * stride + half_kernel_dim;
				kern_col_ind = spatial_col - out_spatial_col * stride + half_kernel_dim;
				kern_ind = kern_dim * in_filters * kern_row_ind + in_filters * kern_col_ind + in_filter_id;
				if ((kern_row_ind < 0) || (kern_row_ind >= kern_dim) || (kern_col_ind < 0) || (kern_col_ind >= kern_dim) ||
						(out_spatial_row < 0) || (out_spatial_row >= out_spatial_dim) || (out_spatial_col < 0) || (out_spatial_col >= out_spatial_dim)) {
					out_spatial_val_deriv = 0;
				}
				else{
					out_spatial_val_deriv = weights[out_filt_id * kernel_size + kern_ind] * out_deriv[out_spatial_ind];
				}
				total_deriv += out_spatial_val_deriv;
			}
		}
	}
	int input_spatial_ind = spatial_dim * spatial_dim * in_filters * sample_ind + spatial_dim * in_filters * spatial_row + in_filters * spatial_col + in_filter_id;
	// used because normal backprop + residual adds to deriv
	if (toAdd){
		input_deriv[input_spatial_ind] += total_deriv;
	}
	else{
		input_deriv[input_spatial_ind] = total_deriv;
	}
	
}

// FOR NOW KEEP NAIVE (UN-OPTIMIZED)...
// not bothering with shared memory for now...

// Independent over (input filter, output filter, kern_x, kern_x)
// could use shared memory over input values...
// Launch with gridDim (kern_dim, kern_dim, output_filters) and blockDim (input_filters) [if input_filters > MAX_THREAD_PER_BLOCK switch ordering of input_filters and output_filters in launch]
__global__ void convolutionDerivWeights(const float * input, const float * weights, const float * out_deriv, int spatial_dim, int kern_dim, int in_filters, int out_filters, int stride, int batch_size,
											float * weight_deriv, bool is_block_dim_inp){

	int in_filter_id;
	int out_filter_id;
	if (is_block_dim_inp){
		in_filter_id = threadIdx.x;
		out_filter_id = blockIdx.z;
	}
	else{
		in_filter_id = blockIdx.z;
		out_filter_id = threadIdx.x;
	}
	int kern_row = blockIdx.x;
	int kern_col = blockIdx.y;

	// shouldn't need to check based on launch specs, but will anyways...
	if ((in_filter_id >= in_filters) || (out_filter_id >= out_filters) || (kern_row >= kern_dim) || (kern_col >= kern_dim)){
		return;
	}

	int kern_ind = kern_dim * in_filters * kern_row + in_filters * kern_col + in_filter_id;

	int kernel_size = (kern_dim * kern_dim * in_filters);
	int half_kernel_dim = kern_dim / 2;
	int out_spatial_dim = spatial_dim / stride;
	int in_spatial_row, in_spatial_col, in_spatial_ind, out_spatial_ind;
	float out_spatial_val_deriv;
	float total_deriv = 0;
	for (int s = 0; s < batch_size; s++){
		for (int out_row = 0; out_row < out_spatial_dim; out_row++){
			for (int out_col = 0; out_col < out_spatial_dim; out_col++){

				// given out_row, out_col, kern_row, kern_col => get the input value used to generate output
				in_spatial_row = stride * out_row + kern_row - half_kernel_dim;
				in_spatial_col = stride * out_col + kern_col - half_kernel_dim;

				// accounting for input filter and sample in batch get index into the input values
				in_spatial_ind = spatial_dim * spatial_dim * in_filters * s + spatial_dim * in_filters * in_spatial_row + in_filters * in_spatial_col + in_filter_id;

				// going from sample, out_row, out_col, out_filter to get index into out_deriv values
				out_spatial_ind = out_spatial_dim * out_spatial_dim * out_filters * s + out_spatial_dim * out_filters * out_row + out_filters * out_col + out_filter_id;

				if ((in_spatial_row < 0) || (in_spatial_row >= spatial_dim) || (in_spatial_col < 0) || (in_spatial_col >= spatial_dim)){
					out_spatial_val_deriv = 0;
				}
				else{
					out_spatial_val_deriv = input[in_spatial_ind] * out_deriv[out_spatial_ind];
				}
				total_deriv += out_spatial_val_deriv;
			}
		}
	}
	weight_deriv[kernel_size * out_filter_id + kern_ind] = total_deriv;
}



// FOR NOW KEEP NAIVE (UN-OPTIMIZED)...
// not bothering with shared memory for now...
// Independent over (out_filters)
// Could incorportate into other deriv kernels (weights easily), but separate for semantic clarity
// To optimize can reduce over samples or in_filters...
// Launch with gridDim (output_filter) and blockDim (1)
__global__ void convolutionDerivBiases(const float * input, const float * weights, const float * out_deriv, int spatial_dim, int kern_dim, int in_filters, int out_filters, int stride, int batch_size,
											float * bias_deriv){
	int out_filter_id = blockIdx.x;
	// shouldn't need to check based on launch specs, but will anyways...
	if (out_filter_id >= out_filters){
		return;
	}

	int out_spatial_dim = spatial_dim / stride;
	int out_spatial_ind;
	float total_deriv = 0;
	for (int s = 0; s < batch_size; s++){
		for (int out_row = 0; out_row < out_spatial_dim; out_row++){
			for (int out_col = 0; out_col < out_spatial_dim; out_col++){
				// going from sample, out_row, out_col, out_filter to get index into out_deriv values
				out_spatial_ind = out_spatial_dim * out_spatial_dim * out_filters * s + out_spatial_dim * out_filters * out_row + out_filters * out_col + out_filter_id;
				total_deriv += out_deriv[out_spatial_ind];
			}
		}
	}
	bias_deriv[out_filter_id] = total_deriv;
}


// iterating over each filter separately
// launch with (OUTFILTERS) grid dim and thread dim of 1 (could easily parallelize menas + vars, with reduction, but save for later..)
// could also use shared memory here if want to be faster
// input is the output of convolution
// ASSUME reLU activation function
__global__ void doBatchNormAndActivate(const float * input, const float * gamma, const float * beta,
								int spatial_dim, int filters, int batch_size, float eps, float * means, float * vars, float * normalized_temp, float * normalized, float * activated){

	int filter_id = blockIdx.x * blockDim.x + threadIdx.x;
	if (filter_id >= filters){
		return;
	}

	float mean, var;
	float sum = 0;
	for (int s = 0; s < batch_size; s++){
		for (int i = 0; i < spatial_dim; i++){
			for (int j = 0; j < spatial_dim; j++){
				sum += input[spatial_dim * spatial_dim * filters * s + spatial_dim * filters * i + filters * j + filter_id];
			}
		}
	}

	mean = sum / (batch_size * spatial_dim * spatial_dim);
	means[filter_id] = mean;

	float var_sum = 0;
	int inp_index;
	for (int s = 0; s < batch_size; s++){
		for (int i = 0; i < spatial_dim; i++){
			for (int j = 0; j < spatial_dim; j++){
				inp_index = spatial_dim * spatial_dim * filters * s + spatial_dim * filters * i + filters * j + filter_id;
				var_sum += (input[inp_index] - mean) * (input[inp_index] - mean);
			}
		}
	}

	var = var_sum / (batch_size * spatial_dim * spatial_dim);
	vars[filter_id] = var;

	float normalized_temp_val, normalized_val;
	for (int s = 0; s < batch_size; s++){
		for (int i = 0; i < spatial_dim; i++){
			for (int j = 0; j < spatial_dim; j++){
				inp_index = spatial_dim * spatial_dim * filters * s + spatial_dim * filters * i + filters * j + filter_id;
				normalized_temp_val = (input[inp_index] - mean) / sqrtf(var + eps);
				normalized_temp[inp_index] = normalized_temp_val;
				normalized_val = gamma[filter_id] * normalized_temp_val + beta[filter_id];
				normalized[inp_index] = normalized_val;
				activated[inp_index] = fmaxf(normalized_val, 0); 
			}
		}
	}
}


// iterating over each filter separately
// launch with (OUTFILTERS) grid dim and thread dim of 1 (could easily parallelize menas + vars, with reduction, but save for later..)
// could also use shared memory here if want to be faster
// input is the output of convolution
// ASSUME reLU activation function
__global__ void activationAndBatchNormDeriv(const float * input, const float * gamma, const float * beta, 
									int spatial_dim, int filters, int batch_size, float eps, const float * means, const float * vars, const float * normalized_temp, const float * activated,
									const float * out_layer_deriv, float * normalized_temp_deriv, float * gamma_deriv, float * beta_deriv, float * input_deriv){
	
	
	int filter_id = blockIdx.x * blockDim.x + threadIdx.x;
	if (filter_id >= filters){
		return;
	}

	float n_samples = batch_size * spatial_dim * spatial_dim;
	float gamma_val = gamma[filter_id];
	float mean_val = means[filter_id];
	float var_val = vars[filter_id];

	// first compute dL/activated (relu deriv) and then dL/dNormalized_Temp (== x hat)
	// also can compute dL/dGamma and dL/dBeta (parameters of batch norm)
	int index;
	float dGamma = 0;
	float dBeta = 0;
	float activated_val, out_layer_deriv_val, normalized_temp_val;
	for (int s = 0; s < batch_size; s++){
		for (int i = 0; i < spatial_dim; i++){
			for (int j = 0; j < spatial_dim; j++){
				index = spatial_dim * spatial_dim * filters * s + spatial_dim * filters * i + filters * j + filter_id;
				activated_val = activated[index];
				if (activated_val <= 0){
					normalized_temp_deriv[index] = 0;
				}
				else{
					out_layer_deriv_val = out_layer_deriv[index];
					normalized_temp_val = normalized_temp[index];
					normalized_temp_deriv[index] = out_layer_deriv_val * gamma_val;
					dGamma += out_layer_deriv_val * normalized_temp_val;
					dBeta += out_layer_deriv_val;
				}
			}
		}
	}

	// save down dGamma and dBeta so optimzer can update parameters
	gamma_deriv[filter_id] = dGamma;
	beta_deriv[filter_id] = dBeta;

	// compute dL/dVar and most of dL/dMean
	float dVar = 0;
	float dMean = 0;
	float partial_var_deriv = 0; 
	float norm_temp_deriv_val;
	float filt_var_three_halfs_power = -0.5 * powf(var_val + eps, -1.5);
	float filt_var_recip_sqrt = -1.0 / sqrtf(var_val + eps);
	for (int s = 0; s < batch_size; s++){
		for (int i = 0; i < spatial_dim; i++){
			for (int j = 0; j < spatial_dim; j++){
				index = spatial_dim * spatial_dim * filters * s + spatial_dim * filters * i + filters * j + filter_id;
				norm_temp_deriv_val = normalized_temp_deriv[index];
				dVar += norm_temp_deriv_val * (input[index] - mean_val) * filt_var_three_halfs_power;
				dMean += norm_temp_deriv_val * filt_var_recip_sqrt;
				partial_var_deriv += -2 * (input[index] - mean_val);
			}
		}
	}

	// finish off dL/dMean
	dMean += dVar * partial_var_deriv / n_samples;

	// compute dL/dX (aka w.r.t. to input to batch norm which is typically the output of a conv)
	// saving input_deriv so backprop can continue to previous layer
	for (int s = 0; s < batch_size; s++){
		for (int i = 0; i < spatial_dim; i++){
			for (int j = 0; j < spatial_dim; j++){
				index = spatial_dim * spatial_dim * filters * s + spatial_dim * filters * i + filters * j + filter_id;
				input_deriv[index] = normalized_temp_deriv[index] * filt_var_recip_sqrt + dVar * (2 * (input[index] - mean_val)) / n_samples + dMean / n_samples;
			}
		}
	}
}



// assume grid launch of (SPATIAL_OUT_DIM, SPATIAL_OUT_DIM) and block dim of (FILTERS)
// could parallelize over batches as well, but probably ok. 
// *runs into issues if #filters greater than threads per block
__global__ void doMaxPool(const float * input, int kern_dim, int stride, int batch_size, int * max_inds, float * out){

	int filter_id = threadIdx.x;

	// know this because of launch specification
	int filters = blockDim.x;
	int in_spatial_dim = stride * gridDim.x;
	int out_spatial_dim = gridDim.x;

	int spatial_row_start = stride * blockIdx.x;
	int spatial_col_start = stride * blockIdx.y;

	int half_kernel_dim = kern_dim / 2;

	float max_val, inp_val;
	int spatial_row, spatial_col, max_ind, inp_ind, out_ind;
	for (int s = 0; s < batch_size; s++){
		max_val = -1;
		max_ind = -1;
		for (int row_off = -half_kernel_dim; row_off <= half_kernel_dim; row_off++){
			for (int col_off = -half_kernel_dim; col_off <= half_kernel_dim; col_off++){
				spatial_row = spatial_row_start + row_off;
				spatial_col = spatial_col_start + col_off;
				if ((spatial_row < 0) || (spatial_row >= in_spatial_dim) || (spatial_col < 0) || (spatial_col >= in_spatial_dim)){
					continue;
				}
				inp_ind = in_spatial_dim * in_spatial_dim * filters * s + in_spatial_dim * filters * spatial_row + filters * spatial_col + filter_id;
				inp_val = input[inp_ind];
				if (inp_val > max_val){
					max_val = inp_val;
					max_ind = inp_ind;
				}
			}
		}
		out_ind = out_spatial_dim * out_spatial_dim * filters * s + out_spatial_dim * filters * blockIdx.x + filters * blockIdx.y + filter_id;
		max_inds[out_ind] = max_ind;
		out[out_ind] = max_val;
	}
}

// assume grid launch of (SPATIAL_OUT_DIM, SPATIAL_OUT_DIM, OUT_FILTERS) and block dim of (BATCH_SIZE)
// max_inds_populated is mapping from max_pool_out_index -> associated max_index of input (populated from forward pass)
// also assume max_pool_inp_deriv is populated with all 0's to begin with and we overwrite non-zero values
__global__ void maxPoolDeriv(const int *max_inds_populated, const float *out_deriv, int kern_dim, int in_spatial_dim, int stride, int filters, int batch_size, float * max_pool_inp_deriv){

	int out_spatial_dim = in_spatial_dim / stride;

	int out_spatial_row = blockIdx.x;
	int out_spatial_col = blockIdx.y;
	int out_filter_id = blockIdx.z;
	int sample_ind = threadIdx.x;

	// based on launch spec should be ok, but check anyways
	if ((out_spatial_row >= out_spatial_dim) || (out_spatial_col >= out_spatial_dim) || (out_filter_id >= filters) || (sample_ind >= batch_size)){
		return;
	}

	int out_ind = out_spatial_dim * out_spatial_dim * filters * sample_ind + out_spatial_dim * filters * out_spatial_row + filters * out_spatial_col + out_filter_id;
	int max_ind_for_out = max_inds_populated[out_ind];

	max_pool_inp_deriv[max_ind_for_out] = out_deriv[out_ind];
}


// assume grid launch of (# Filters) and block dim of (batch size)
// could parallelize over batches as well, but probably ok. 
// *runs into issues if #filters greater than threads per block
__global__ void doFilterAvgPool(const float * input, int spatial_dim, float * out){

	int filter_id = blockIdx.x;
	int sample_ind = threadIdx.x;

	// know this because of launch specification
	int filters = blockDim.x;

	float sum = 0;
	for (int row = 0; row < spatial_dim; row++){
		for (int col = 0; col < spatial_dim; col++){
			sum += input[spatial_dim * spatial_dim * filters * sample_ind + spatial_dim * filters * row + filters * col + filter_id];
		}
	}

	float avg_val = sum / (spatial_dim * spatial_dim);
	out[filters * sample_ind + filter_id] = avg_val;
}

// assume grid launch of (# Filters) and block dim of (batch size)
// could parallelize over batches as well, but probably ok. 
// *runs into issues if #filters greater than threads per block
__global__ void filterAvgPoolDeriv(const float * pooled_deriv, int filters, int batch_size, int spatial_dim, float * out){

	int filter_id = blockIdx.x;
	int sample_ind = threadIdx.x;

	// unnecessary because of launch conditions, but putting anyways...
	if ((filter_id >= filters) || (sample_ind >= batch_size)){
		return;
	}

	// indexing into (N, 2048) = (batch_size, filters) matrix 
	float pooled_filt_deriv = pooled_deriv[sample_ind * filters + filter_id];
	float avg_pooled_filt_deriv = pooled_filt_deriv / (spatial_dim * spatial_dim);

	// populating the pre-pooled conv block output
	for (int row = 0; row < spatial_dim; row++){
		for (int col = 0; col < spatial_dim; col++){
			out[spatial_dim * spatial_dim * filters * sample_ind + spatial_dim * filters * row + filters * col + filter_id] = avg_pooled_filt_deriv;
		}
	}
}



// hardcoded conv kernel for initial 7x7, stride 2, 64 output filter convolutional layer...
// launching (14, 112, BATCH_SIZE) dim blocks where each block has 112/14=8 phases to utilize shared memory. Each block will have dim (64).
// Each block will contribute 16 unique spatial inds * 64 output filters * 32 Batch Size to the output of layer
// each phase loads stride new rows into shared memory, then multiples new spatial shared_mem with conv_weights, accounting for conv weight col permuation 

/* MAY OR MAY NOT WORK... (commented becuase not used...) */

// __global__ void optimized_init_conv(const float * input, const float * weights, float * out){

// 	__shared__ float conv_weights[64][147];
// 	__shared__ float spatial_vals[147];

// 	// index
// 	int output_filter = threadIdx.x;
// 	int sample_ind = blockIdx.z;

// 	// assume weights are in order of outfilter 0: [R_0,0, B_0,0, G_0,0, R_0,1, G_0,1, B_0,1....R_6,6, G_6,6, B_6,6], outfilter 1: [...], ...., outfilter 63: [...]
// 	for (int kernel_ind = 0; kernel_ind < 147; kernel_ind++){
// 		conv_weights[output_filter][kernel_ind] = weights[output_filter * 147 + kernel_ind];
// 	}o

// 	// 2 * vals because stride of 2
// 	int spatial_row_start = (224 / blockDim.x) * blockIdx.x;
// 	int spatial_col_start = 2 * blockIdx.y;
// 	int spatial_row, spatial_col, kernel_ind;
// 	int half_kernel_dim = 3;
// 	for (int row_offset = -half_kernel_dim; row_offset <= half_kernel_dim;  row_offset++){
// 		for (int col_offset = -half_kernel_dim; col_offset <= half_kernel_dim; col_offset++){
// 			for (int channel = 0; channel < 3; channel++){
// 				spatial_row = spatial_row_start + row_offset;
// 				spatial_col = spatial_col_start + col_offset;
// 				kernel_ind = 7 * 3 * (row_offset + half_kernel_dim) + 3 * (col_offset + half_kernel_dim) + channel;
// 				if ((spatial_row < 0) || (spatial_row >= 224) || (spatial_col < 0) || (spatial_col >= 224)) {
// 					spatial_vals[kernel_ind] = 0;
// 				}
// 				else{
// 					spatial_vals[kernel_ind] = input[224 * 224 * 3 * sample_ind + 224 * 3 * spatial_row + 3 * spatial_col + channel];
// 				}
// 			}
// 		}
// 	}

// 	__syncthreads();

// 	float val = 0;
// 	int circular_row = 0;
// 	int out_spatial_row = (112 / blockDim.x) * blockIdx.x;
// 	int out_spatial_col = blockIdx.y;
// 	int new_top_row = 0;
// 	for (int phase = 0; phase < 8; phase++){

// 		// compute matrix mult to get (output_filt x batch_size) result. this is for a single receptive field across depth and batches
// 		// iterative over phases to get multiple receptive fields and exploit spatial locality
// 		val = 0;
// 		for (int kern_row = 0; kern_row < 7; kern_row++){
// 			for (int kern_col = 0; kern_col < 7; kern_col++){
// 				for (int ch = 0; ch < 3; ch++){
// 					circular_row = (kern_row + 2 * phase) % 7;
// 					val += conv_weights[output_filter][7 * 3 * kern_row + 3 * kern_col + ch] * spatial_vals[7 * 3 * circular_row + 3 * kern_col + ch];
// 				}
// 			}
// 		}

// 		out[112 * 112 * 64 * sample_ind + 112 * 64 * out_spatial_row + 64 * out_spatial_col + output_filter] = val;

// 		__syncthreads();

// 		int row_to_replace, replace_ind;
// 		for (int i = 1; i <= 2; i++){
// 			row_to_replace = (2 * phase) + i % 7;
// 			spatial_row = spatial_row_start + half_kernel_dim + 2 * phase + i; 
// 			for (int col_offset = -half_kernel_dim; col_offset <= half_kernel_dim; col_offset++){
// 				for (int channel = 0; channel < 3; channel++){
// 					spatial_col = spatial_col_start + col_offset;
// 					replace_ind = 7 * 3 * row_to_replace + 3 * (col_offset + half_kernel_dim) + channel;
// 					if ((spatial_row < 0) || (spatial_row >= 224) || (spatial_col < 0) || (spatial_col >= 224)) {
// 						spatial_vals[replace_ind][sample_ind] = 0;
// 					}
// 					else{
// 						spatial_vals[replace_ind][sample_ind] = input[224 * 224 * 3 * sample_ind + 224 * 3 * spatial_row + 3 * spatial_col + channel];
// 					}
// 				}
// 			}
// 		}
// 		out_spatial_row++;

// 		__syncthreads();
// 	}
// }



// assume pass in 1-D block with batch size blocks and 1 thread per block
// could exploit more parallelism here but shouldnt be bottleneck for now...
// assume X is a matrix where # rows = batch size and # columns = output dim
__global__ void softMax(const float * X, int batch_size, int output_len, float * out){
  int i = blockIdx.x;
  if (i < batch_size){
    float sum = 0;
    for (int j = 0; j < output_len; j++){
      sum += __expf(X[i * output_len + j]);
    }
    for (int j = 0; j < output_len; j++){
      out[i * output_len + j] = __expf(X[i * output_len + j]) / sum;
    }
  }
}

// launch with gridDim (output_dim) and threadDim (batch_size)
__global__ void averageDerivOverBatchSize(float * output_deriv, int output_dim, int batch_size){

	int output_class = blockIdx.x;
	int sample_ind = threadIdx.x;

	// shouldn't happen because of launch spec but check anyways...
	if ((output_class >= output_dim) || (sample_ind >= batch_size)){
		return;
	}
	output_deriv[sample_ind * output_dim + output_class] /= batch_size;
}


// launch with gridDim = (batch_size), blockDim = (1)
__global__ void crossEntropyDeriv(float * output_deriv, const int * correct_classes, int output_dim, int batch_size){
	int i = blockIdx.x;
	if (i < batch_size){
		output_deriv[i * output_dim + correct_classes[i]] -= 1;
	}
}

// assume large 1-D launch
__global__ void updateMeans(int size, const float * gradients, float base_mean_decay, float * prev_means, int loc_ind){
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i >= size){
		return;
	}
	if (isnan(gradients[i])){
		printf("ERROR in Update Means for Parameter at location: %d\nGradient is NAN at index: %d...keeping same running mean\n\n", loc_ind, i);
		return;
	}
	if (isinf(gradients[i])){
		printf("ERROR in Update Means for Parameter at location: %d\nGradient is INF at index: %d...keeping same running mean\n\n", loc_ind, i);
		return;
	}
	prev_means[i] = base_mean_decay * prev_means[i] + (1 - base_mean_decay) * gradients[i];
	
}

// assume large 1-D launch
__global__ void updateVars(int size, const float * gradients, float base_var_decay, float * prev_vars, int loc_ind){
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i >= size){
		return;
	}
	float grad = gradients[i];
	if (isnan(grad)){
		printf("ERROR in Update Vars for Parameter at location: %d\nGradient is NAN at index: %d...keeping same running var\n", loc_ind, i);
		return;
	}
	if (isinf(grad)){
		printf("ERROR in Update Vars for Parameter at location: %d\nGradient is INF at index: %d...keeping same running var\n", loc_ind, i);
		return;
	}
	prev_vars[i] = base_var_decay * prev_vars[i] + (1 - base_var_decay) * grad * grad;
}

// assume large 1-D launch
__global__ void updateParams(int size, float * model_params, const float * means, const float * vars, float learning_rate, float cur_mean_decay, float cur_var_decay, float eps, int loc_ind){
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i >= size){
		return;
	}
	float bias_corrected_mean = means[i] / (1 - cur_mean_decay);
	float bias_corrected_var = vars[i] / (1 - cur_var_decay);
	float old_model_param = model_params[i];
	model_params[i] = model_params[i] - learning_rate * bias_corrected_mean / (sqrtf(bias_corrected_var) + eps);
	if (isnan(model_params[i])){
		printf("ERROR: for Parameter at location: %d\nto NAN at index: %d...resetting to prev value\n", loc_ind, i);
		model_params[i] = old_model_param;
		return;
	}
	if (isinf(model_params[i])){
		printf("ERROR: for Parameter at location: %d\nto INF at index: %d...resetting to prev value\n", loc_ind, i);
		model_params[i] = old_model_param;
		return;
	}
}

/* INITIALIZE CORE MODEL STRUCTURES */

Dims * init_dimensions(int input, int init_kernel_dim, int init_conv_filters, int init_conv_stride, int init_maxpool_dim, int init_maxpool_stride, 
							int n_conv_blocks, int * is_block_spatial_reduction, int final_depth, int output){
	
	Dims * dims = (Dims *) malloc(sizeof(Dims));
	dims -> input = input;
	dims -> init_kernel_dim = init_kernel_dim;
	dims -> init_conv_filters = init_conv_filters;
	dims -> init_conv_stride = init_conv_stride;
	dims -> init_maxpool_dim = init_maxpool_dim;
	dims -> init_maxpool_stride = init_maxpool_stride;
	dims -> n_conv_blocks = n_conv_blocks;
	dims -> is_block_spatial_reduction = is_block_spatial_reduction;
	dims -> final_depth = final_depth;
	dims -> output = output;

	return dims;
}

BatchNorm * init_batch_norm(int spatial_dim, int depth, bool is_zero){
	
	BatchNorm * batch_norm = (BatchNorm *) malloc(sizeof(BatchNorm));

	batch_norm -> spatial_dim = spatial_dim;
	batch_norm -> depth = depth;

	float * gamma, * beta;

	hipMalloc(&gamma, depth * sizeof(float));
	hipMemset(gamma, 0, depth * sizeof(float));
	if (!is_zero){
		setVal <<< ceil((float) depth / MAX_THREAD_PER_BLOCK), MAX_THREAD_PER_BLOCK >>> (depth, 1.0, gamma);
	}

	hipMalloc(&beta, depth * sizeof(float));
	hipMemset(beta, 0, depth * sizeof(float));

	batch_norm -> gamma = gamma;
	batch_norm -> beta = beta;

	return batch_norm;

}

ConvBlock * init_conv_block(int incoming_filters, int incoming_spatial_dim, int reduced_depth, int expanded_depth, int stride, hiprandGenerator_t * gen, bool is_zero){
	
	ConvBlock * conv_block = (ConvBlock *) malloc(sizeof(ConvBlock));
	conv_block -> incoming_filters = incoming_filters;
	conv_block -> incoming_spatial_dim = incoming_spatial_dim;
	conv_block -> reduced_depth = reduced_depth;
	conv_block -> expanded_depth = expanded_depth;
	conv_block -> stride = stride;

	float * depth_reduction, *spatial, *depth_expansion;
	float * bias_depth_reduction, * bias_spatial, * bias_depth_expansion;
	int depth_reduction_size, spatial_size, depth_expansion_size;
	int bias_depth_reduction_size, bias_spatial_size, bias_depth_expansion_size;
	float depth_reduction_fan_in, spatial_fan_in, depth_expansion_fan_in;

	BatchNorm *norm_depth_reduction, *norm_spatial, *norm_residual_added;

	depth_reduction_size = incoming_filters * reduced_depth;
	depth_reduction_fan_in = incoming_spatial_dim * incoming_spatial_dim * incoming_filters;
	hipMalloc(&depth_reduction, depth_reduction_size * sizeof(float));
	hipMemset(depth_reduction, 0, depth_reduction_size * sizeof(float));
	if (!is_zero){
		init_weights_gaussian_device(gen, depth_reduction_size, depth_reduction, 0, 2.0 / depth_reduction_fan_in);
	}

	bias_depth_reduction_size = reduced_depth;
	hipMalloc(&bias_depth_reduction, bias_depth_reduction_size * sizeof(float));
	hipMemset(bias_depth_reduction, 0, bias_depth_reduction_size * sizeof(float));

	norm_depth_reduction = init_batch_norm(incoming_spatial_dim, reduced_depth, is_zero);


	spatial_size = reduced_depth * reduced_depth * 3 * 3;
	spatial_fan_in = incoming_spatial_dim * incoming_spatial_dim * reduced_depth;
	hipMalloc(&spatial, spatial_size * sizeof(float));
	hipMemset(spatial, 0, spatial_size * sizeof(float));
	if (!is_zero){
		init_weights_gaussian_device(gen, spatial_size, spatial, 0, 2.0 / spatial_fan_in);
	}

	bias_spatial_size = reduced_depth;
	hipMalloc(&bias_spatial, bias_spatial_size * sizeof(float));
	hipMemset(bias_spatial, 0, bias_spatial_size * sizeof(float));

	// the spatial decrease happens at middle 3x3 layer, to the last layer of stride block will receive lower spatial dim input
	if (stride == 2){
		incoming_spatial_dim /= 2;
	}
	norm_spatial = init_batch_norm(incoming_spatial_dim, reduced_depth, is_zero);

	depth_expansion_size = expanded_depth * reduced_depth;
	depth_expansion_fan_in = incoming_spatial_dim * incoming_spatial_dim * reduced_depth;
	hipMalloc(&depth_expansion, depth_expansion_size * sizeof(float));
	hipMemset(depth_expansion, 0, depth_expansion_size * sizeof(float));
	if (!is_zero){
		init_weights_gaussian_device(gen, depth_expansion_size, depth_expansion, 0, 2.0 / depth_expansion_fan_in);
	}

	bias_depth_expansion_size = expanded_depth;
	hipMalloc(&bias_depth_expansion, bias_depth_expansion_size * sizeof(float));
	hipMemset(bias_depth_expansion, 0, bias_depth_expansion_size * sizeof(float));

	conv_block -> depth_reduction = depth_reduction;
	conv_block -> bias_depth_reduction = bias_depth_reduction;
	conv_block -> norm_depth_reduction = norm_depth_reduction;

	conv_block -> spatial = spatial;
	conv_block -> bias_spatial = bias_spatial;
	conv_block -> norm_spatial = norm_spatial;


	conv_block -> depth_expansion = depth_expansion;
	conv_block -> bias_depth_expansion = bias_depth_expansion;

	float * projection, *bias_projection;
	int projection_size;
	if (stride == 2){
		projection_size = 3 * 3 * incoming_filters * expanded_depth;
	}
	else{
		projection_size = incoming_filters * expanded_depth;
	}

	// assuming only project when depths are different (all projections in resnet-50 this way)
	// could later change to adapt to just spatial transform...
	if (incoming_filters != expanded_depth){
		hipMalloc(&projection, projection_size * sizeof(float));
		hipMemset(projection, 0, projection_size * sizeof(float));
		if (!is_zero){
			init_weights_gaussian_device(gen, projection_size, projection, 0, 2.0 / incoming_filters);
		}
		hipMalloc(&bias_projection, expanded_depth * sizeof(float));
		hipMemset(bias_projection, 0, expanded_depth * sizeof(float));
	}
	else{
		projection = NULL;
		bias_projection = NULL;
	}

	conv_block -> projection = projection;
	conv_block -> bias_projection = bias_projection;


	norm_residual_added = init_batch_norm(incoming_spatial_dim, expanded_depth, is_zero);
	conv_block -> norm_residual_added = norm_residual_added;

	return conv_block;
}

Params * init_model_parameters(Dims * model_dims, hiprandGenerator_t * gen, bool is_zero){

	Params * params = (Params *) malloc(sizeof(Params));

	// dimensions unpacked
	int input_dim = model_dims -> input;
	int n_conv_blocks = model_dims -> n_conv_blocks;
	int init_kernel_dim = model_dims -> init_kernel_dim;
	int init_conv_filters = model_dims -> init_conv_filters;
	int * is_block_spatial_reduction = model_dims -> is_block_spatial_reduction;
	int output_dim = model_dims -> output;

	// init array to hold pointers to weights
	// 3 * 4 weight arrays per conv block (weights, biases, gamma, beta per layer in block) + 4 * inital + fully connected + 4 * 2 projections
	// ignoring biases + batch norm weights for now...
	int n_locations = 13 + 12 * n_conv_blocks;
	params -> n_locations = n_locations;

	float ** locations = (float **) malloc(n_locations * sizeof(float *));
	int * sizes = (int *) malloc(n_locations * sizeof(int));
	// tracking location ind as we start allocating...
	


	// init first 7 * 7 conv_layer
	float * init_conv_layer;
	int init_conv_size = init_kernel_dim * init_kernel_dim * init_conv_filters * 3;
	float init_conv_fan_in = 3 * input_dim * input_dim;
	hipError_t malloc_err = hipMalloc(&init_conv_layer,  init_conv_size * sizeof(float));
	hipError_t memset_err = hipMemset(init_conv_layer, 0, init_conv_size * sizeof(float));
	if (!is_zero){
		init_weights_gaussian_device(gen, init_conv_size, init_conv_layer, 0, 2.0 / init_conv_fan_in);
	}
	params -> init_conv_layer = init_conv_layer;
	int loc_ind = 0;
	locations[loc_ind] = init_conv_layer;
	sizes[loc_ind] = init_kernel_dim * init_kernel_dim * init_conv_filters * 3;
	loc_ind++;

	float * bias_init_conv;
	hipMalloc(&bias_init_conv, init_conv_filters * sizeof(float));
	hipMemset(bias_init_conv, 0, init_conv_filters * sizeof(float));

	params -> bias_init_conv = bias_init_conv;

	locations[loc_ind] = bias_init_conv;
	sizes[loc_ind] = init_conv_filters;
	loc_ind++;

	BatchNorm * norm_init_conv = init_batch_norm(input_dim / model_dims -> init_conv_stride, init_conv_filters, is_zero);
	params -> norm_init_conv = norm_init_conv;

	locations[loc_ind] = norm_init_conv -> gamma;
	sizes[loc_ind] = init_conv_filters;
	loc_ind++;

	locations[loc_ind] = norm_init_conv -> beta;
	sizes[loc_ind] = init_conv_filters;
	loc_ind++;
	

	// init conv blocks
	ConvBlock ** conv_blocks = (ConvBlock **) malloc(n_conv_blocks * sizeof(ConvBlock *));
	int incoming_filters = init_conv_filters;
	// assume stride 2 initial conv layer then stride 2 pool before entering conv_blocks
	int incoming_spatial_dim = input_dim / 4;
	int stride = 1;
	int reduced_depth = init_conv_filters;
	int expanded_depth = 4 * init_conv_filters;
	for (int i = 0; i < n_conv_blocks; i++){
		if (is_block_spatial_reduction[i] == 1){
			stride = 2;
			reduced_depth *= 2;
			expanded_depth *= 2;
		}
		else{
			stride = 1;
		}
		conv_blocks[i] = init_conv_block(incoming_filters, incoming_spatial_dim, reduced_depth, expanded_depth, stride, gen, is_zero);
		locations[loc_ind] = conv_blocks[i] -> depth_reduction;
		sizes[loc_ind] = incoming_filters * reduced_depth;
		loc_ind++;
		locations[loc_ind] = conv_blocks[i] -> bias_depth_reduction;
		sizes[loc_ind] = reduced_depth;
		loc_ind++;
		locations[loc_ind] = conv_blocks[i] -> norm_depth_reduction -> gamma;
		sizes[loc_ind] = reduced_depth;
		loc_ind++;
		locations[loc_ind] = conv_blocks[i] -> norm_depth_reduction -> beta;
		sizes[loc_ind] = reduced_depth;
		loc_ind++;

		locations[loc_ind] = conv_blocks[i] -> spatial;
		sizes[loc_ind] = reduced_depth * reduced_depth * 3 * 3;
		loc_ind++;
		locations[loc_ind] = conv_blocks[i] -> bias_spatial;
		sizes[loc_ind] = reduced_depth;
		loc_ind++;
		locations[loc_ind] = conv_blocks[i] -> norm_spatial -> gamma;
		sizes[loc_ind] = reduced_depth;
		loc_ind++;
		locations[loc_ind] = conv_blocks[i] -> norm_spatial -> beta;
		sizes[loc_ind] = reduced_depth;
		loc_ind++;

		locations[loc_ind] = conv_blocks[i] -> depth_expansion;
		sizes[loc_ind] = expanded_depth * reduced_depth;
		loc_ind++;
		locations[loc_ind] = conv_blocks[i] -> bias_depth_expansion;
		sizes[loc_ind] = expanded_depth;
		loc_ind++;
		
		// if the block needed a projection to make input dim = output dim
		if (conv_blocks[i] -> projection){
			locations[loc_ind] = conv_blocks[i] -> projection;
			if (stride == 2){
				sizes[loc_ind] = 3 * 3 * incoming_filters * expanded_depth;
			}
			else{
				sizes[loc_ind] = incoming_filters * expanded_depth;
			}
			loc_ind++;
			locations[loc_ind] = conv_blocks[i] -> bias_projection;
			sizes[loc_ind] = expanded_depth;
			loc_ind++;
		}

		locations[loc_ind] = conv_blocks[i] -> norm_residual_added -> gamma;
		sizes[loc_ind] = expanded_depth;
		loc_ind++;
		locations[loc_ind] = conv_blocks[i] -> norm_residual_added -> beta;
		sizes[loc_ind] = expanded_depth;
		loc_ind++;


		// after stride 2 block then reduce spatial dim for next block
		if (is_block_spatial_reduction[i] == 1){
			incoming_spatial_dim /= 2;
		}
		incoming_filters = expanded_depth;
	}
	params -> conv_blocks = conv_blocks;

	float * fully_connected;
	// here expanded depth is the last layer's filters which will go through average pool before FC layer
	// expanded depth should equal dims -> final_depth
	int fully_connected_size = expanded_depth * output_dim;
	float fully_connected_fan_in = expanded_depth;
	hipMalloc(&fully_connected, fully_connected_size * sizeof(float));
	hipMemset(fully_connected, 0, fully_connected_size * sizeof(float));
	if (!is_zero){
		init_weights_gaussian_device(gen, fully_connected_size, fully_connected, 0, 2.0 / fully_connected_fan_in);
	}

	params -> fully_connected = fully_connected;
	locations[loc_ind] = fully_connected;
	sizes[loc_ind] = expanded_depth * output_dim;

	params -> locations = locations;
	params -> sizes = sizes;

	return params;
}

ResNet * init_resnet(Dims * dims, hiprandGenerator_t * gen){
	ResNet * model = (ResNet *) malloc(sizeof(ResNet));
	model -> dims = dims;
	Params * params = init_model_parameters(dims, gen, false);
	model -> params = params;
	return model;
}


/* INITIALIZE TRAINING STRUCTURES */

Cache_BatchNorm * init_cache_batchnorm(int input_size, int feature_size){
	Cache_BatchNorm * cache_batchnorm = (Cache_BatchNorm *) malloc(sizeof(Cache_BatchNorm));

	cache_batchnorm -> input_size = input_size;
	cache_batchnorm -> feature_size = feature_size;

	float * means, *vars, *normalized_temp, *normalized;

	hipMalloc(&means, feature_size * sizeof(float));
	hipMalloc(&vars, feature_size * sizeof(float));
	hipMalloc(&normalized_temp, input_size * sizeof(float));
	hipMalloc(&normalized, input_size * sizeof(float));

	cache_batchnorm -> means = means;
	cache_batchnorm -> vars = vars;
	cache_batchnorm -> normalized_temp = normalized_temp;
	cache_batchnorm -> normalized = normalized;

	return cache_batchnorm;
}

Activation_ConvBlock * init_activation_convblock(ConvBlock * conv_block, int batch_size){
	Activation_ConvBlock * activation_conv_block = (Activation_ConvBlock *) malloc(sizeof(Activation_ConvBlock));

	int incoming_filters = conv_block -> incoming_filters;
	int incoming_spatial_dim = conv_block -> incoming_spatial_dim;
	int stride = conv_block -> stride;
	int reduced_depth = conv_block -> reduced_depth;
	int expanded_depth = conv_block -> expanded_depth;

	activation_conv_block -> incoming_filters = incoming_filters;
	activation_conv_block -> incoming_spatial_dim = incoming_spatial_dim;
	activation_conv_block -> reduced_depth = reduced_depth;
	activation_conv_block -> expanded_depth = expanded_depth;
	activation_conv_block -> stride = stride;

	float * post_reduced, *post_spatial, *post_expanded, *transformed_residual, *output, *output_activated;
	float * post_reduced_activated, *post_spatial_activated;
	int post_reduced_size, post_spatial_size, output_size;
	Cache_BatchNorm * norm_post_reduced, *norm_post_spatial, *norm_post_residual_added;
	

	post_reduced_size = reduced_depth * incoming_spatial_dim * incoming_spatial_dim * batch_size;
	hipMalloc(&post_reduced, post_reduced_size * sizeof(float));
	activation_conv_block -> post_reduced = post_reduced;

	norm_post_reduced = init_cache_batchnorm(post_reduced_size, reduced_depth);
	activation_conv_block -> norm_post_reduced = norm_post_reduced;

	hipMalloc(&post_reduced_activated, post_reduced_size * sizeof(float));
	activation_conv_block -> post_reduced_activated = post_reduced_activated;

	post_spatial_size = reduced_depth * incoming_spatial_dim * incoming_spatial_dim / (stride * stride) * batch_size;
	hipMalloc(&post_spatial, post_spatial_size * sizeof(float));
	activation_conv_block -> post_spatial = post_spatial;

	norm_post_spatial = init_cache_batchnorm(post_spatial_size, reduced_depth);
	activation_conv_block -> norm_post_spatial = norm_post_spatial;

	hipMalloc(&post_spatial_activated, post_spatial_size * sizeof(float));
	activation_conv_block -> post_spatial_activated = post_spatial_activated;

	output_size = expanded_depth * incoming_spatial_dim * incoming_spatial_dim / (stride * stride) * batch_size;
	
	hipMalloc(&post_expanded, output_size * sizeof(float));
	activation_conv_block -> post_expanded = post_expanded;

	// only allocate space if transformed, otherwise it will be assumed to be identity of input
	transformed_residual = NULL;
	if (incoming_filters != expanded_depth){
		hipMalloc(&transformed_residual, output_size * sizeof(float));
	}
	activation_conv_block -> transformed_residual = transformed_residual;

	hipMalloc(&output, output_size * sizeof(float));
	activation_conv_block -> output = output;

	norm_post_residual_added = init_cache_batchnorm(output_size, expanded_depth);
	activation_conv_block -> norm_post_residual_added = norm_post_residual_added;

	hipMalloc(&output_activated, output_size * sizeof(float));
	activation_conv_block -> output_activated = output_activated;

	return activation_conv_block;
}

Activations * init_activations(Dims * dims, ConvBlock ** conv_blocks, int batch_size){
	
	Activations * activations = (Activations *) malloc(sizeof(Activations));

	int input_dim = dims -> input;
	int init_conv_filters = dims -> init_conv_filters;
	int init_conv_stride = dims -> init_conv_stride;
	int maxpool_stride = dims -> init_maxpool_stride;

	float * init_conv_applied;
	int init_conv_applied_size = init_conv_filters * input_dim * input_dim / (init_conv_stride * init_conv_stride) * batch_size; 
	hipMalloc(&init_conv_applied, init_conv_applied_size * sizeof(float));
	activations -> init_conv_applied = init_conv_applied;

	Cache_BatchNorm * norm_init_conv = init_cache_batchnorm(init_conv_applied_size, init_conv_filters);
	activations -> norm_init_conv = norm_init_conv;

	float * init_conv_activated;
	hipMalloc(&init_conv_activated, init_conv_applied_size * sizeof(float));
	activations -> init_conv_activated = init_conv_activated;

	int init_convblock_input_size = init_conv_filters * input_dim * input_dim / (init_conv_stride * init_conv_stride) / (maxpool_stride * maxpool_stride) * batch_size;

	int * max_inds;
	hipMalloc(&max_inds, init_convblock_input_size * sizeof(int));
	activations -> max_inds = max_inds;

	float *init_convblock_input;
	hipMalloc(&init_convblock_input, init_convblock_input_size * sizeof(float));
	activations -> init_convblock_input = init_convblock_input;

	int n_conv_blocks = dims -> n_conv_blocks;

	Activation_ConvBlock ** activation_conv_blocks = (Activation_ConvBlock **) malloc(n_conv_blocks * sizeof(Activation_ConvBlock *));
	for (int i = 0; i < n_conv_blocks; i++){
		ConvBlock * conv_block = conv_blocks[i];
		activation_conv_blocks[i] = init_activation_convblock(conv_block, batch_size);
	}

	activations -> activation_conv_blocks = activation_conv_blocks;
	activations -> n_conv_blocks = n_conv_blocks;

	int final_depth = dims -> final_depth;
	float * final_conv_output_pooled;
	int final_conv_output_pooled_size = final_depth * batch_size;
	hipMalloc(&final_conv_output_pooled, final_conv_output_pooled_size * sizeof(float));
	activations -> final_conv_output_pooled = final_conv_output_pooled;

	int output_dim = dims -> output;
	int output_size = output_dim * batch_size;

	float * linear_output;
	hipMalloc(&linear_output, output_size * sizeof(float));
	activations -> linear_output = linear_output;

	return activations;
}


Forward_Buffer * init_forward_buffer(Dims * dims, ConvBlock ** conv_blocks, int batch_size){

	Forward_Buffer * forward_buffer = (Forward_Buffer *) malloc(sizeof(Forward_Buffer));

	forward_buffer -> activations = init_activations(dims, conv_blocks, batch_size);

	int output_dim = dims -> output;
	int output_size = output_dim * batch_size;

	float * pred;
	hipMalloc(&pred, output_size * batch_size * sizeof(float));
	forward_buffer -> pred = pred;

	// will be copied to cpu to be able to print values and compute loss on cpu side
	float * pred_cpu = (float *) malloc(output_size * batch_size * sizeof(float));
	forward_buffer -> pred_cpu = pred_cpu;

	return forward_buffer;
}


Backprop_Buffer * init_backprop_buffer(Dims * dims, ConvBlock ** conv_blocks, int batch_size){

	Backprop_Buffer * backprop_buffer = (Backprop_Buffer *) malloc(sizeof(Backprop_Buffer));

	int output_dim = dims -> output;
	int output_size = output_dim * batch_size;

	float * output_layer_deriv;
	hipMalloc(&output_layer_deriv, output_size * sizeof(float));
	backprop_buffer -> output_layer_deriv = output_layer_deriv;

	backprop_buffer -> param_derivs = init_model_parameters(dims, NULL, true);
	backprop_buffer -> prev_means = init_model_parameters(dims, NULL, true);
	backprop_buffer -> prev_vars = init_model_parameters(dims, NULL, true);
	backprop_buffer -> activation_derivs = init_activations(dims, conv_blocks, batch_size);

	return backprop_buffer;
}


Train_ResNet * init_trainer(ResNet * model, Batch * cur_batch, int batch_size, float learning_rate, float mean_decay, float var_decay, float eps, int n_epochs){
	Train_ResNet * trainer = (Train_ResNet *) malloc(sizeof(Train_ResNet));

	trainer -> model = model;

	trainer -> cur_batch = cur_batch;
	trainer -> batch_size = batch_size;

	Dims * dims = model -> dims;
	ConvBlock ** conv_blocks = model -> params -> conv_blocks;
	trainer -> forward_buffer = init_forward_buffer(dims, conv_blocks, batch_size);
	trainer -> backprop_buffer = init_backprop_buffer(dims, conv_blocks, batch_size);

	trainer -> learning_rate = learning_rate;
	trainer -> base_mean_decay = mean_decay;
	trainer -> base_var_decay = var_decay;
	trainer -> cur_mean_decay = 1;
	trainer -> cur_var_decay = 1;
	trainer -> eps = eps;

	trainer -> n_epochs = n_epochs;

	trainer -> loss_per_epoch = (float *) calloc(n_epochs, sizeof(float));
	trainer -> accuracy_per_epoch = (float *) calloc(n_epochs, sizeof(float));

	return trainer;
}

Batch * init_general_batch(int n_images, int image_size, int image_dim, int shard_n_images){
	Batch * batch = (Batch *) malloc(sizeof(Batch));

	batch -> n_images = n_images;
	// in resnet-50 will be 224 * 224 * 3
	batch -> image_size = image_size;
	batch -> image_dim = image_dim;
	float * images_float_cpu;
	// load batch by first brining into cpu, pinned memory
	hipError_t status_images_pinned = hipHostMalloc((float **)&images_float_cpu, (size_t) n_images * (size_t) image_size * sizeof(float));
	batch -> images_float_cpu = images_float_cpu;
	
	// allocate memory on gpu so that after loaded on cpu can bring in
	// will be converting from uint8 on CPU to float on GPU
	float * images;
	hipMalloc(&images, (size_t) n_images * (size_t) image_size * sizeof(float));
	batch -> images = images;

	// pinned memory for correct_classes_cpu
	int * correct_classes_cpu;
	hipError_t status_classes_pinned = hipHostMalloc((int **)&correct_classes_cpu, n_images * sizeof(int));
	batch -> correct_classes_cpu = correct_classes_cpu;

	int * correct_classes;
	hipMalloc(&correct_classes, n_images * sizeof(int));
	batch -> correct_classes = correct_classes;

	batch -> cur_shard_id = -1;
	batch -> cur_batch_in_shard = -1;
	batch -> shard_n_images = shard_n_images;
	batch -> full_shard_images = (float *) malloc((size_t) shard_n_images * (size_t) image_size * sizeof(float));
	batch -> full_shard_correct_classes = (int *) malloc(shard_n_images * sizeof(int));

	return batch;
}

// (if this takes too long, can do it in parallel with separate process on cpu)
// ASSUMING shard_n_images % batch_size = 0
void load_new_batch(Class_Metadata * class_metadata, Batch * batch_buffer){
	
	int batch_size = batch_buffer -> n_images;
	int image_size = batch_buffer -> image_size;
	size_t total_pixels = (size_t) batch_size * (size_t) image_size;
	
	float * full_shard_images = batch_buffer -> full_shard_images;
	int * full_shard_correct_classes = batch_buffer -> full_shard_correct_classes;	

	float * images_float_cpu = batch_buffer -> images_float_cpu;
	float * images = batch_buffer -> images;

	int * correct_classes_cpu = batch_buffer -> correct_classes_cpu;
	int * correct_classes = batch_buffer -> correct_classes;

	int cur_shard_id = batch_buffer -> cur_shard_id;
	int cur_batch_in_shard = batch_buffer -> cur_batch_in_shard;
	int shard_n_images = batch_buffer -> shard_n_images;



	int start_img_num = cur_batch_in_shard * batch_size;
	int n_read;
	// cur_shard_id = -1 implies first iteration
	if ((cur_shard_id == -1) || (start_img_num >= shard_n_images)) {

		// update new shard id
		cur_shard_id += 1;
		batch_buffer -> cur_shard_id = cur_shard_id;

		// load new shard into RAM
		char shard_images_filepath[100];
		sprintf(shard_images_filepath, "/mnt/storage/data/vision/imagenet/2012/train_data_shards/%03d.images", cur_shard_id);
		shard_images_filepath[67] = '\0';
		FILE * shard_images_file = fopen(shard_images_filepath, "rb");
		n_read = fread(full_shard_images, sizeof(float), ((size_t) shard_n_images) * ((size_t) image_size), shard_images_file);
		fclose(shard_images_file);


		char shard_labels_filepath[100];
		sprintf(shard_labels_filepath, "/mnt/storage/data/vision/imagenet/2012/train_data_shards/%03d.labels", cur_shard_id);
		shard_labels_filepath[67] = '\0';
		FILE * shard_labels_file = fopen(shard_labels_filepath, "rb");
		n_read = fread(full_shard_correct_classes, sizeof(int), shard_n_images, shard_labels_file);
		fclose(shard_labels_file);

		// reset cur batch in shard to 0
		cur_batch_in_shard = 0;
		batch_buffer -> cur_batch_in_shard = cur_batch_in_shard;
	}

	// load current batch
	memcpy(images_float_cpu, full_shard_images + cur_batch_in_shard * total_pixels, total_pixels * sizeof(float));
	memcpy(correct_classes_cpu, full_shard_correct_classes + cur_batch_in_shard * batch_size, batch_size * sizeof(int));

	
	/* SAVING BATCH TO FILES FOR INSPECTION... */
	// FILE * test_images_file = fopen("images.buffer", "wb");
	// fwrite(images_float_cpu, sizeof(float), total_pixels, test_images_file);
	// fclose(test_images_file);

	// FILE * test_labels_file = fopen("labels.buffer", "wb");
	// fwrite(correct_classes_cpu, sizeof(int), (size_t) batch_size, test_labels_file);
	// fclose(test_labels_file);

	// copy current batch to GPU

	hipMemcpy(images, images_float_cpu, total_pixels * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(correct_classes, correct_classes_cpu, batch_size * sizeof(int), hipMemcpyHostToDevice);

	// update cur batch for next iteration of loading
	cur_batch_in_shard++;
	batch_buffer -> cur_batch_in_shard = cur_batch_in_shard;

}


// READ CLASSES AND LABELS!
// reading a text file line by line into a buffer
// pre-allocate buffer and specify type
void text_file_to_buffer(void * buffer, char * filename, const char * type){

	char ** my_text_buffer = (char **) buffer;
	int * my_int_buffer = (int *) buffer;
	
	FILE * fp;
    char * line = NULL;
    size_t len = 0;

    fp = fopen(filename, "r");
    if (fp == NULL)
        exit(EXIT_FAILURE);
    int cnt = 0;
    while (getline(&line, &len, fp) != -1) {
    	if (strcmp(type, "TEXT") == 0){
        	my_text_buffer[cnt] = strdup(line);
        }
        else if (strcmp(type, "INT") == 0){
        	my_int_buffer[cnt] = atoi(line);
        }
        else{
        	// pass
        }
        cnt++;
    }

    fclose(fp);
    if (line){
    	free(line);
    }
}

Class_Metadata * populate_class_info(char * label_filename, char * synset_filename, char * class_size_filename, int n_classes){
	
	Class_Metadata * classes = (Class_Metadata *) malloc(sizeof(Class_Metadata));

	char ** labels = (char **) malloc(n_classes * sizeof(char *));
	char ** synsets = (char **) malloc(n_classes * sizeof(char *));
	int * counts = (int *) malloc(n_classes * sizeof(int));

	text_file_to_buffer(labels, label_filename, "TEXT");
	text_file_to_buffer(synsets, synset_filename, "TEXT");
	text_file_to_buffer(counts, class_size_filename, "INT");

	classes -> labels = labels;
	classes -> synsets = synsets;
	classes -> counts = counts;
	classes -> n_classes = n_classes;

	return classes;
}


/* PREP AND LAUNCHING CUDA KERNELS! */


// tried to write optimized conv kernel. will wait to debug...
// void prepareAndDoConvolutionOptimized(int in_spatial_dim, int kern_dim, int in_filters, int out_filters,  int stride, int batch_size, 
// 																float * input, float * weights, float * biases, float * output){

// 	int out_filter_row_size = kern_dim * kern_dim * in_filters;
// 	int max_out_filter_rows = MAX_SHARED_MEM_FLOATS / out_filter_row_size;
// 	int out_filter_chunks = ceil((float) out_filters / max_out_filter_rows);
// 	int shared_mem_size = out_filter_row_size * max_out_filter_rows;
// 	int out_spatial_dim = in_spatial_dim / stride;
// 	int max_subatch_size = MAX_THREAD_PER_BLOCK / max_out_filter_rows;

// 	dim3 gridDimConv(out_spatial_dim, out_spatial_dim, out_filter_chunks);
// 	dim3 blockDimConv(max_out_filter_rows, max_subatch_size);

// 	doConvolution <<< gridDimConv, blockDimConv, shared_mem_size >>> (input, weights, biases, in_spatial_dim, kern_dim, in_filters, out_filters, stride, batch_size, output);

// }


void prepareAndDoConvolution(int in_spatial_dim, int kern_dim, int in_filters, int out_filters,  int stride, int batch_size, 
																float * input, float * weights, float * biases, float * output){
	int out_spatial_dim = in_spatial_dim / stride;
	int out_filters_block = min(MAX_THREAD_PER_BLOCK / batch_size, out_filters);
	int out_filters_grid = max(1, (int) ceil((float) out_filters / (float) out_filters_block));

	dim3 gridDimConv(out_spatial_dim, out_spatial_dim, out_filters_grid);
	dim3 blockDimConv(batch_size, out_filters_block);

	doConvolution <<< gridDimConv, blockDimConv>>> (input, weights, biases, in_spatial_dim, kern_dim, in_filters, out_filters, stride, batch_size, output);
}


void prepreAndDoConvolutionDeriv(int in_spatial_dim, int kern_dim, int in_filters, int out_filters, int stride, int batch_size, bool toAdd,
												float * input, float * weights, float * out_deriv,
												float * input_deriv, float * weight_deriv, float * bias_deriv, bool toComputeInputDeriv){
	
	// first layer conv doesn't take deriv w.r.t input
	int in_filters_block = min(MAX_THREAD_PER_BLOCK / batch_size, in_filters);
	int in_filters_grid = max(1, (int) ceil((float) in_filters / (float) in_filters_block));

	dim3 gridDimDerivInput(in_spatial_dim, in_spatial_dim, in_filters_grid);
	dim3 blockDimDerivInput(batch_size, in_filters_block);
	if (toComputeInputDeriv){
		convolutionDerivInput <<< gridDimDerivInput, blockDimDerivInput >>> (input, weights, out_deriv, in_spatial_dim, kern_dim, in_filters, out_filters, stride, batch_size, toAdd, input_deriv);
	}

	int block_dim, grid_dim;
	bool is_block_dim_inp;
	if (in_filters > MAX_THREAD_PER_BLOCK){
		block_dim = out_filters;
		grid_dim = in_filters;
		is_block_dim_inp = false;
	}
	else{
		block_dim = in_filters;
		grid_dim = out_filters;
		is_block_dim_inp = true;
	}
	
	dim3 gridDimDerivWeights(kern_dim, kern_dim, grid_dim);
	dim3 blockDimDerivWeights(block_dim);
	convolutionDerivWeights <<< gridDimDerivWeights, blockDimDerivWeights >>> (input, weights, out_deriv, in_spatial_dim, kern_dim, in_filters, out_filters, stride, batch_size, weight_deriv, is_block_dim_inp);

	//convolutionDerivBiases <<< out_filters, 1 >>> (input, weights, out_deriv, in_spatial_dim, kern_dim, in_filters, out_filters, stride, batch_size, bias_deriv);
	
}

void prepareAndDoBatchNormAndActivate(BatchNorm * batch_norm_params, Cache_BatchNorm * batch_norm_cache, int batch_size, float eps, float * input, float * activated_out){
	// reading values from batch norm params
	int filters = batch_norm_params -> depth;
	int spatial_dim = batch_norm_params -> spatial_dim;
	float * gamma = batch_norm_params -> gamma;
	float * beta = batch_norm_params -> beta;

	// read the output device pointers from batch_norm_cache
	float * means_out = batch_norm_cache -> means;
	float * vars_out = batch_norm_cache -> vars;
	float * normalized_temp_out = batch_norm_cache -> normalized_temp;
	float * normalized_out = batch_norm_cache -> normalized;

	int num_threads = min(MAX_THREAD_PER_BLOCK_INCL_REG, filters);
	int num_blocks = 1;
	if (filters > num_threads){
		num_blocks = ceil((float) filters / (float) MAX_THREAD_PER_BLOCK_INCL_REG);
	}
	doBatchNormAndActivate<<< num_blocks, num_threads >>> (input, gamma, beta, spatial_dim, filters, batch_size, eps, means_out, vars_out, normalized_temp_out, normalized_out, activated_out);
}

void prepareAndDoActivationAndBatchNormDeriv(BatchNorm * batch_norm_params, Cache_BatchNorm * batch_norm_cache, BatchNorm * batch_norm_param_derivs, Cache_BatchNorm * batch_norm_cache_derivs, 
																								int batch_size, float eps, float * input, float * activated, float * out_layer_deriv, float * input_deriv){
	int filters = batch_norm_params -> depth;
	int spatial_dim = batch_norm_params -> spatial_dim;
	float * gamma = batch_norm_params -> gamma;
	float * beta = batch_norm_params -> beta;
	float * means = batch_norm_cache -> means;
	float * vars = batch_norm_cache -> vars;
	float * normalized_temp = batch_norm_cache -> normalized_temp;

	float * normalized_temp_deriv = batch_norm_cache_derivs -> normalized_temp;
	float * gamma_deriv = batch_norm_param_derivs -> gamma;
	float * beta_deriv = batch_norm_param_derivs -> beta;

	int num_threads = min(MAX_THREAD_PER_BLOCK_INCL_REG, filters);
	int num_blocks = 1;
	if (filters > num_threads){
		num_blocks = ceil((float) filters / (float) MAX_THREAD_PER_BLOCK_INCL_REG);
	}
	activationAndBatchNormDeriv <<< num_blocks, num_threads >>> (input, gamma, beta, spatial_dim, filters, batch_size, eps, means, vars, normalized_temp, activated, out_layer_deriv, normalized_temp_deriv, gamma_deriv, beta_deriv, input_deriv);


}

void prepareAndDoMatMulLeftTranspose(const float * left_orig, const float * right, int left_orig_rows, int left_orig_cols, int right_rows, int right_cols, float * out){
	float * temp_left;
	hipMalloc(&temp_left, left_orig_rows * left_orig_cols * sizeof(float));

	dim3 gridDimTranspose(ceil((float) left_orig_rows / TILE_WIDTH), ceil((float)left_orig_cols / TILE_WIDTH));
	dim3 blockDimTranspose(TILE_WIDTH, TILE_WIDTH);
	transpose <<< gridDimTranspose, blockDimTranspose >>> (left_orig, left_orig_rows, left_orig_cols, temp_left);

	dim3 gridDimMatMul(ceil((float) left_orig_cols / TILE_WIDTH), ceil((float) right_cols / TILE_WIDTH));
	dim3 blockDimMatMul(TILE_WIDTH, TILE_WIDTH);
	matMul <<< gridDimMatMul, blockDimMatMul >>> (temp_left, right, left_orig_cols, right_rows, right_cols, out);
	hipFree(temp_left);
}

void prepareAndDoMatMulRightTranspose(const float * left, const float * right_orig, int left_rows, int left_cols, int right_orig_rows, int right_orig_cols, float * out){
	float * temp_right;
	hipMalloc(&temp_right, right_orig_rows * right_orig_cols * sizeof(float));
	
	dim3 gridDimTranspose(ceil((float) right_orig_rows / TILE_WIDTH), ceil((float)right_orig_cols / TILE_WIDTH));
	dim3 blockDimTranspose(TILE_WIDTH, TILE_WIDTH);

	transpose <<< gridDimTranspose, blockDimTranspose >>> (right_orig, right_orig_rows, right_orig_cols, temp_right);
	
	dim3 gridDimMatMul(ceil((float) left_rows / TILE_WIDTH), ceil((float) right_orig_rows / TILE_WIDTH));
	dim3 blockDimMatMul(TILE_WIDTH, TILE_WIDTH);
	matMul <<< gridDimMatMul, blockDimMatMul >>> (left, temp_right, left_rows, left_cols, right_orig_rows, out);
	hipFree(temp_right);
}

void printDeviceData(const char * name_of_variable, float * device_variable, int size){
	bool print = TO_PRINT;
	if (print){
		float * cpu_data = (float *) malloc(size * sizeof(float));
		hipMemcpy(cpu_data, device_variable, size * sizeof(float), hipMemcpyDeviceToHost);
		printf("VARIABLE NAME: %s\n\n", name_of_variable);
		printf("DATA:\n");
		for (int i = 0; i < size; i++){
			printf("%d: %f\n", i, cpu_data[i]);
		}
		printf("\n\n\n");
		free(cpu_data);
	}
}

void forward_pass(Train_ResNet * trainer){

	Dims * dims = trainer -> model -> dims;

	float eps = trainer -> eps;
	int batch_size = trainer -> batch_size;

	float * input = trainer -> cur_batch -> images;
	float * first_conv = trainer -> model -> params -> init_conv_layer;
	float * first_conv_bias = trainer -> model -> params -> bias_init_conv;
	float * first_conv_output = trainer -> forward_buffer -> activations -> init_conv_applied;
	// first apply the convolutions
	// launch grid dimensions as (OUT_SPATIAL_DIM, OUT_SPATIAL_DIM, OUT_FILTER_CHUNK) blocks, and launch with block dim as (out_filt_rows_shared, sub_batch) threads
	
	// 3 colors
	int init_in_filters = 3;
	int init_spatial_dim = dims -> input;
	int init_kernel_dim = dims -> init_kernel_dim;
	int init_out_filters = dims -> init_conv_filters;
	int init_stride = dims -> init_conv_stride;
	int init_out_spatial_dim = init_spatial_dim / init_stride;

	prepareAndDoConvolution(init_spatial_dim, init_kernel_dim, init_in_filters, init_out_filters, init_stride, batch_size, input, first_conv, first_conv_bias, first_conv_output);

	int print_size = 10;
	printDeviceData("INIT CONV APPLIED", first_conv_output, print_size);


	BatchNorm * norm_init_conv_params = trainer -> model -> params -> norm_init_conv;
	Cache_BatchNorm * norm_init_conv_cache = trainer -> forward_buffer -> activations -> norm_init_conv;
	float * init_activated = trainer -> forward_buffer -> activations -> init_conv_activated;

	prepareAndDoBatchNormAndActivate(norm_init_conv_params, norm_init_conv_cache, batch_size, eps, first_conv_output, init_activated);

	printDeviceData("INIT CONV ACTIVATED", init_activated, print_size);

	int init_maxpool_dim = dims -> init_maxpool_dim;
	int init_maxpool_stride = dims -> init_maxpool_stride;
	int init_maxpool_out_dim = init_out_spatial_dim / init_maxpool_stride;
	float * init_convblock_input = trainer -> forward_buffer -> activations -> init_convblock_input;
	int * max_ind_buff = trainer -> forward_buffer -> activations -> max_inds;

	dim3 gridDimMaxPool(init_maxpool_out_dim, init_maxpool_out_dim);
	doMaxPool <<< gridDimMaxPool , init_out_filters >>> (init_activated, init_maxpool_dim, init_maxpool_stride, batch_size, max_ind_buff, init_convblock_input);

	printDeviceData("MAX POOL OUTPUT", init_convblock_input, print_size);

	/* NOW CAN MOVE ONTO TO CONV_BLOCK LAYERS! */

	int n_conv_blocks = dims -> n_conv_blocks;

	
	ConvBlock ** params_conv_blocks = trainer -> model -> params -> conv_blocks;
	Activation_ConvBlock ** activation_conv_blocks = trainer -> forward_buffer -> activations -> activation_conv_blocks;
	ConvBlock * cur_conv_block_params;
	Activation_ConvBlock * cur_conv_block_activation;
	int in_spatial_dim, kern_dim, in_filters, out_filters, stride, out_spatial_dim, total_size_conv_block_output;

	float * conv_block_input = init_convblock_input;
	float *conv_input, * conv_weights, * conv_biases, * conv_output, *norm_input, * norm_output, * conv_block_output;
	float *projection_weights, *projection_biases, *transformed_residual;
	BatchNorm * cur_batch_norm_params;
	Cache_BatchNorm * cur_batch_norm_cache;
	for (int i = 0; i < n_conv_blocks; i++){
		cur_conv_block_params = params_conv_blocks[i];
		cur_conv_block_activation = activation_conv_blocks[i];

		// do first 1x1 depth_reduce convolution
		in_spatial_dim = cur_conv_block_params -> incoming_spatial_dim;
		in_filters = cur_conv_block_params -> incoming_filters;
		out_filters = cur_conv_block_params -> reduced_depth;
		kern_dim = 1;
		stride = 1;
		// either intialized first time above loop from the maxpool
		// every other block will be the normalized, activated output of previous conv block (previous iteration output) 
		conv_input = conv_block_input;
		conv_weights = cur_conv_block_params -> depth_reduction;
		conv_biases = cur_conv_block_params -> bias_depth_reduction;
		conv_output = cur_conv_block_activation -> post_reduced;

		prepareAndDoConvolution(in_spatial_dim, kern_dim, in_filters, out_filters, stride, batch_size, conv_input, conv_weights, conv_biases, conv_output);

		printDeviceData("REDUCED CONV APPLIED", conv_output, print_size);

		norm_input = conv_output;
		cur_batch_norm_params = cur_conv_block_params -> norm_depth_reduction;
		cur_batch_norm_cache = cur_conv_block_activation -> norm_post_reduced;
		norm_output = cur_conv_block_activation -> post_reduced_activated;

		prepareAndDoBatchNormAndActivate(cur_batch_norm_params, cur_batch_norm_cache, batch_size, eps, norm_input, norm_output);

		printDeviceData("REDUCED CONV NORM & ACTIVATED", norm_output, print_size);

		// do 3x3 spatial convolution

		// same as in first conv
		in_spatial_dim = cur_conv_block_params -> incoming_spatial_dim;
		// now is output filters of 1st conv, which is reduced depth filters
		in_filters = cur_conv_block_params -> reduced_depth;
		// keeps depth the same, just spatial conv
		out_filters = cur_conv_block_params -> reduced_depth;
		kern_dim = 3;
		// if stride is occurring in conv block happens at this kernel
		stride = cur_conv_block_params -> stride;
		conv_input = norm_output;
		conv_weights = cur_conv_block_params -> spatial;
		conv_biases = cur_conv_block_params -> bias_spatial;
		conv_output = cur_conv_block_activation -> post_spatial;

		prepareAndDoConvolution(in_spatial_dim, kern_dim, in_filters, out_filters, stride, batch_size, conv_input, conv_weights, conv_biases, conv_output);

		printDeviceData("SPATIAL CONV APPLIED", conv_output, print_size);

		norm_input = conv_output;
		cur_batch_norm_params = cur_conv_block_params -> norm_spatial;
		cur_batch_norm_cache = cur_conv_block_activation -> norm_post_spatial;
		norm_output = cur_conv_block_activation -> post_spatial_activated;

		prepareAndDoBatchNormAndActivate(cur_batch_norm_params, cur_batch_norm_cache, batch_size, eps, norm_input, norm_output);

		printDeviceData("SPATIAL CONV NORM & ACTIVATED", norm_output, print_size);

		// do 1x1 depth expansion convolution

		// if stride happened now would need to take that into account
		in_spatial_dim = (cur_conv_block_params -> incoming_spatial_dim) / (cur_conv_block_params -> stride);
		// prev 3x3 conv kept out filters as reduced depth
		in_filters = cur_conv_block_params -> reduced_depth;
		// now creating expanded depth out filters
		out_filters = cur_conv_block_params -> expanded_depth;
		kern_dim = 1;
		stride = 1;
		conv_input = norm_output;
		conv_weights = cur_conv_block_params -> depth_expansion;
		conv_biases = cur_conv_block_params -> bias_depth_expansion;
		conv_output = cur_conv_block_activation -> post_expanded;

		prepareAndDoConvolution(in_spatial_dim, kern_dim, in_filters, out_filters, stride, batch_size, conv_input, conv_weights, conv_biases, conv_output);

		printDeviceData("EXPANDED CONV APPLIED", conv_output, print_size);

		// now need to add identity of conv_block_input (if same dimensions), or project=convolve (different dimensions) and add to conv_output
		// projection is a incoming block filters X expanded depth matrix
		// if stride of 2 in additon to depth change, then 3x3 kernel with stride 2 applied to block input
		// works as a depth-wise 1x1 convolution where in_filters = incoming_filters and out_filters = expanded_depth

		// already updated
		in_spatial_dim = (cur_conv_block_params -> incoming_spatial_dim);
		out_spatial_dim = (cur_conv_block_params -> incoming_spatial_dim) / (cur_conv_block_params -> stride);
		in_filters = cur_conv_block_params -> incoming_filters;
		out_filters = cur_conv_block_params -> expanded_depth;
		stride = cur_conv_block_params -> stride;
		if (stride == 2){
			kern_dim = 3;
		}
		else{
			kern_dim = 1;
		}
		projection_weights = cur_conv_block_params -> projection;
		projection_biases = cur_conv_block_params -> bias_projection;


		total_size_conv_block_output = out_spatial_dim * out_spatial_dim * out_filters * batch_size;
		conv_block_output = cur_conv_block_activation -> output;
				
		// the conv_block initializer already handled if we need projection, and if so it allocated weights
		// if there is a projection needed we will do convolution with the above parameters
		if (projection_weights){
			// allocated device memory to store output
			transformed_residual = cur_conv_block_activation -> transformed_residual;
			prepareAndDoConvolution(in_spatial_dim, kern_dim, in_filters, out_filters, stride, batch_size, conv_block_input, projection_weights, projection_biases, transformed_residual);
		}
		else{
			// would've been null, so renaming for semantic clarity
			transformed_residual = conv_block_input;
		}

		printDeviceData("(TRANSFORMED) RESIDUAL", transformed_residual, print_size);

		// add identity residual connection (or projected residual connection) to the prior convolutional output
		addVec <<< ceil((float) total_size_conv_block_output / MAX_THREAD_PER_BLOCK), MAX_THREAD_PER_BLOCK >>> (total_size_conv_block_output, conv_output, transformed_residual, conv_block_output);

		printDeviceData("CONV OUTPUT + (TRANSFORMED) RESIDUAL", conv_block_output, print_size);

		norm_input = conv_block_output;
		cur_batch_norm_params = cur_conv_block_params -> norm_residual_added;
		cur_batch_norm_cache = cur_conv_block_activation -> norm_post_residual_added;
		norm_output = cur_conv_block_activation -> output_activated;

		prepareAndDoBatchNormAndActivate(cur_batch_norm_params, cur_batch_norm_cache, batch_size, eps, norm_input, norm_output);

		printDeviceData("CONV BLOCK OUTPUT NORM & ACTIVATED", norm_output, print_size);

		// prepare for next block...
		conv_block_input = norm_output;
	}

	int final_filters = dims -> final_depth;
	int final_spatial_dim = params_conv_blocks[n_conv_blocks - 1] -> incoming_spatial_dim;
	float * final_conv_block_output = activation_conv_blocks[n_conv_blocks - 1] -> output_activated;
	float * final_avg_pool_values = trainer -> forward_buffer -> activations -> final_conv_output_pooled;

	// NEED TO DO AVERAGE POOL OF LAST LAYER to go from (batch_size, 7, 7, 2048) to (batch size, 1, 1, 2048)

	// format of output is each row is a sample and has a row size of 2048
	doFilterAvgPool <<< (final_filters), (batch_size) >>> (final_conv_block_output, final_spatial_dim, final_avg_pool_values);

	printDeviceData("FINAL AVG POOL VALUES", final_avg_pool_values, print_size);


	// APPLY FULLY CONNECTED LAYER BETWEEN (2048, 1000)
	float * fc_weights = trainer -> model -> params -> fully_connected;
	float * fc_output = trainer -> forward_buffer -> activations -> linear_output;
	int output_dim = dims -> output;

	// matrix multiply between (N, 2048) and fc weights of (2048, 1000), yields output of (N, 1000)
	// output is each row is a unique sample

	// GRID has dim (OUT_ROWS / TILE_WIDTH, OUT_COLS/TILE_WIDTH)
	// each BLOCK has dim (TILE_WIDTH, TILE_WIDTH)
	dim3 gridDimFCOutput(ceil((float) batch_size / TILE_WIDTH), ceil((float) output_dim / TILE_WIDTH));
	dim3 blockDimFCOutput(TILE_WIDTH, TILE_WIDTH);

	matMul <<< (gridDimFCOutput), (blockDimFCOutput) >>> (final_avg_pool_values, fc_weights, batch_size, final_filters, output_dim, fc_output);

	printDeviceData("FULLY CONNECTED WEIGHTS", fc_weights, print_size);
	printDeviceData("FULLY CONNECTED OUTPUT", fc_output, print_size);

	// DO SOFTMAX
	float * pred = trainer -> forward_buffer -> pred;
	softMax <<< (batch_size), (1) >>> (fc_output, batch_size, output_dim, pred);

	printDeviceData("SOFTMAX PREDICTIONS", pred, print_size);

	// FINISH UP BY POPULATING PREDICTIONS ONTO CPU
	float * pred_cpu = trainer -> forward_buffer -> pred_cpu;
	hipMemcpy(pred_cpu, pred, batch_size * output_dim * sizeof(float), hipMemcpyDeviceToHost);
}

void backwards_pass(Train_ResNet * trainer){
	
	Dims * dims = trainer -> model -> dims;
	int batch_size = trainer -> batch_size;
	int output_dim = dims -> output;
	float eps = trainer -> eps;
	Activations * activations = trainer -> forward_buffer -> activations;
	Params * model_params = trainer -> model -> params;
	Backprop_Buffer * backprop_buffer = trainer -> backprop_buffer;
	Params * param_derivs = backprop_buffer -> param_derivs;
	Activations * activation_derivs = backprop_buffer -> activation_derivs;

	int print_size = 10;

	/* STEP 1: LAST LAYER DERIVATIVE */

	// layer has output_dim * batch_size values
	// End of network was: fully connected layer -> softmax
	// Derivative of cross entropy loss w.r.t to fully connected values is: s - y where s is softmax value
	// thus copy softmax values and subtract 1 from the correct index (we know labels y are 0 except correct label of 1)
	int * correct_classes = trainer -> cur_batch -> correct_classes;
	float * pred = trainer -> forward_buffer -> pred;
	float * output_layer_deriv = backprop_buffer -> output_layer_deriv;
	hipMemcpy(output_layer_deriv, pred, batch_size * output_dim * sizeof(float), hipMemcpyDeviceToDevice);

	crossEntropyDeriv <<< (batch_size), (1) >>> (output_layer_deriv, correct_classes, output_dim, batch_size);

	// divide by the batch size because loss is sum across all batches...
	// NOT SURE IF WE WANT TO DO AVERAGE HERE OR NOT...?
	averageDerivOverBatchSize <<< output_dim, batch_size >>> (output_layer_deriv, output_dim, batch_size);

	printDeviceData("CROSS ENTROPY DERIV", output_layer_deriv, print_size);

	/* STEP 2: FC WEIGHT DERIV AND FINAL AVG POOL (SECOND LAST ACTIVTION LAYER) DERIVATIVE */

	// TODO: MAKE SURE THE DIMENSIONS ARE CORRECT ORDER...

	// FC WEIGHTS (2048, 1000) DERIV = matMul(transpose(final_conv_output_pooled), output_layer_deriv)
	int final_depth = dims -> final_depth;
	float * fc_deriv = param_derivs -> fully_connected;
	float * final_conv_output_pooled = activations -> final_conv_output_pooled;
	prepareAndDoMatMulLeftTranspose(final_conv_output_pooled, output_layer_deriv, batch_size, final_depth, batch_size, output_dim, fc_deriv);

	printDeviceData("FC WEIGHT DERIV", fc_deriv, print_size);

	// FINAL AVG POOL (N, 2048) DERIV = matMul(output_layer_deriv, transpose(FC Weight))
	float * fc_weights = model_params -> fully_connected;
	float * final_avg_pool_deriv = activation_derivs -> final_conv_output_pooled;
	prepareAndDoMatMulRightTranspose(output_layer_deriv, fc_weights, batch_size, output_dim, final_depth, output_dim, final_avg_pool_deriv);

	printDeviceData("FINAL AVG POOL ACTIVATION DERIV", final_avg_pool_deriv, print_size);


	/* CONV BLOCK DATA FROM FORWARD PASS */
	int n_conv_blocks = dims -> n_conv_blocks;
	Activation_ConvBlock ** activation_conv_blocks = activations -> activation_conv_blocks;
	ConvBlock ** conv_block_params = model_params -> conv_blocks;

	/* CONV BLOCK DERIV BUFFERS */
	Activation_ConvBlock ** activation_conv_blocks_derivs = activation_derivs -> activation_conv_blocks;
	ConvBlock ** conv_block_param_derivs = param_derivs -> conv_blocks;


	int final_spatial_dim = conv_block_params[n_conv_blocks - 1] -> incoming_spatial_dim;
	
	/* STEP 3: AVG POOL DERIV */

	// get the location for the deriv of final conv block output
	float * final_conv_block_output_deriv = activation_conv_blocks_derivs[n_conv_blocks - 1] -> output_activated;
	// using final_avg_pool_deriv (batch_size, 2048) to populate final_conv_block_output_deriv (batch_size, 7, 7, 2048)
	// each expanded (prior to pooling) spatial index takes on value of given filter's avg_pool_deriv / (spatial_dim^2)
	filterAvgPoolDeriv <<< (final_depth), (batch_size) >>> (final_avg_pool_deriv, final_depth, batch_size, final_spatial_dim, final_conv_block_output_deriv);

	printDeviceData("FINAL CONV BLOCK OUTPUT ACTIVATION DERIV", final_conv_block_output_deriv, print_size);

	
	/* STEP 4: CONV BLOCK & BATCH NORM DERIVS  */
	

	// we are starting with deriv of last conv block output...

	// To go backwards for each block we:
		// 1.) Get deriv of batch norm for residual added to expanded conv output (with respect to both its own parameters and also the input to batch norm = expanded conv output)
		// 2.) Get deriv projection filter & transformed (if there is a projection of residual, otherwise both derivs are 1)
		// 3.) Multiply the deriv of input to batch norm * deriv of transformed residual and add to the deriv of first layer of conv block (= batch norm output of prior block)
		// 4.) Get deriv of expanded convolution & deriv of input to expanded convolution (= batch norm output of spatial conv)
		// 5.) Get deriv of batch norm for spatial conv output (with respect to both its own parameters and also the input to batch norm = spatial conv output)
		// 6.) Get deriv of sptial convolution & deriv of input to spatial convolution (= batch norm output of reduced conv)
		// 7.) Get deriv of batch norm for reduced conv output (with respect to both its own parameters and also the input to batch norm = reduced conv output)
		// 8.) Get deriv of reduced convolution & deriv of input to reduced convolution, which is the first layer of conv block (= batch norm output of prior conv block)
		// Items 3.) and 8.) provide the derivative used to repeat process for prior block

	

	// will update these variables throughout loop to pass to batch norm deriv
	float *bn_input, *bn_activated, *bn_out_layer_deriv, *bn_input_deriv;
	BatchNorm *cur_batch_norm_params, *cur_batch_norm_param_derivs;
	Cache_BatchNorm *cur_batch_norm_cache, *cur_batch_norm_cache_derivs;

	// will update these every iteration through conv_blocks
	ConvBlock * cur_conv_block_params, *cur_conv_block_param_derivs;
	Activation_ConvBlock * cur_conv_block_activation, *cur_conv_block_activation_derivs;

	// will update these within every iteration through conv_blocks
	// because multiple convolutions per block, but keep params same for easy calls to functions
	int in_spatial_dim, kern_dim, in_filters, out_filters, stride;
	float *conv_input, *conv_weight, *conv_out_deriv;
	float *conv_input_deriv, *conv_weight_deriv, *conv_bias_deriv;


	// STARTING POINT FROM BACKPROP COMING FROM UPSTREAM LAYERS IS AT LAST CONV BLOCK ACTIVATION -> OUTPUT_ACTIVATED
	float *conv_block_input, *conv_block_input_deriv;

	// extra temp variables
	int total_size;

	for (int i = n_conv_blocks - 1; i >= 0; i--){

		// residual deriv and normal backprop deriv added to this
		if (i == 0){
			conv_block_input = activations -> init_convblock_input;
			conv_block_input_deriv = activation_derivs -> init_convblock_input;
		}
		else{
			conv_block_input = activation_conv_blocks[i - 1] -> output_activated;
			conv_block_input_deriv = activation_conv_blocks_derivs[i - 1] -> output_activated;
		}

		// getting current conv block parameters and buffers to hold derivs
		cur_conv_block_params = conv_block_params[i];
		cur_conv_block_param_derivs = conv_block_param_derivs[i];

		// getting current conv block activation values and buffers to hold derivs
		cur_conv_block_activation = activation_conv_blocks[i];
		cur_conv_block_activation_derivs = activation_conv_blocks_derivs[i];

		/* 1: Conv Block Output Activation and Batch Norm Derivs */

		// update the current batch norm layer pointers
		cur_batch_norm_params = cur_conv_block_params -> norm_residual_added;
		cur_batch_norm_param_derivs = cur_conv_block_param_derivs -> norm_residual_added;

		cur_batch_norm_cache = cur_conv_block_activation -> norm_post_residual_added;
		cur_batch_norm_cache_derivs = cur_conv_block_activation_derivs -> norm_post_residual_added;

		// fill in details about backprop I/O
		// dL/dBN_Output (given)
		bn_out_layer_deriv = activation_conv_blocks_derivs[i] -> output_activated;
		// dL/dBN_Input (to fill in)
		bn_input_deriv = cur_conv_block_activation_derivs -> output;
		// input to batch norm layer from forward pass
		bn_input = cur_conv_block_activation -> output;
		// activated output of batch norm layer from forward pass
		bn_activated = cur_conv_block_activation -> output_activated;
		
		prepareAndDoActivationAndBatchNormDeriv(cur_batch_norm_params, cur_batch_norm_cache, cur_batch_norm_param_derivs, cur_batch_norm_cache_derivs,
																						batch_size, eps, bn_input, bn_activated, bn_out_layer_deriv, bn_input_deriv);

		printDeviceData("CONV BLOCK OUTPUT ACTIVATION & NORM DERIV", bn_input_deriv, print_size);

		/* 2: (Transformed) Residual Derivs & Chained/Added to Conv Block Input Deriv (= prior_block_output_deriv) */

		// check if there is a projection (aka convolution over depth/kern_dim=1 or possibly stride=2/kern_dim=3), otherwise the projection deriv is 1
		// If there is a projection need to compute derivative of the projection convolution kernel weights and deriv w.r.t. projection convolution input=conv_block_input=prior_block_output_activated
		if (cur_conv_block_params -> projection){


			// CONVOLUTION DIMENSIONS
			in_spatial_dim = (cur_conv_block_params -> incoming_spatial_dim);
			in_filters = cur_conv_block_params -> incoming_filters;
			out_filters = cur_conv_block_params -> expanded_depth;
			stride = cur_conv_block_params -> stride;
			if (stride == 2){
				kern_dim = 3;
			}
			else{
				kern_dim = 1;
			}


			// CONVOLUTION FORWARD DATA
			// transformed residual convolution input is the value at first step of conv block => activated output from previous block
			conv_input = conv_block_input;
			conv_weight = cur_conv_block_params -> projection;
			// from backprop
			conv_out_deriv = cur_conv_block_activation_derivs -> output;

			// CONVOLUTION BACKWARDS DERIV DATA BUFFERS
			// because residual
			conv_input_deriv = conv_block_input_deriv;
			conv_weight_deriv = cur_conv_block_param_derivs -> projection;
			conv_bias_deriv = cur_conv_block_param_derivs -> bias_projection;

			prepreAndDoConvolutionDeriv(in_spatial_dim, kern_dim, in_filters, out_filters, stride, batch_size, false,
													conv_input, conv_weight, conv_out_deriv,
													conv_input_deriv, conv_weight_deriv, conv_bias_deriv, true);

			printDeviceData("PROJECTED CONV INPUT DERIV", conv_input_deriv, print_size);
			printDeviceData("PROJECTED CONV WEIGHT DERIV", conv_weight_deriv, print_size);
			printDeviceData("PROJECTED CONV BIAS DERIV", conv_bias_deriv, print_size);
		}
		else{
			total_size = batch_size * (cur_conv_block_params -> incoming_spatial_dim) * (cur_conv_block_params -> incoming_spatial_dim) * (cur_conv_block_params -> incoming_filters);
			addVec <<< ceil((float) total_size / MAX_THREAD_PER_BLOCK), MAX_THREAD_PER_BLOCK >>> (total_size, conv_block_input_deriv, cur_conv_block_activation_derivs -> output, conv_block_input_deriv);
		}
		

		/* 3: Expanded Convolution Derivs */

		// CONVOLUTION DIMENSIONS
		in_spatial_dim = (cur_conv_block_params -> incoming_spatial_dim) / (cur_conv_block_params -> stride);
		in_filters = cur_conv_block_params -> reduced_depth;
		out_filters = cur_conv_block_params -> expanded_depth;
		stride = 1;
		kern_dim = 1;

		// CONVOLUTION FORWARD DATA
		conv_input = cur_conv_block_activation -> post_spatial_activated;
		conv_weight = cur_conv_block_params -> depth_expansion;
		// from backprop
		conv_out_deriv = cur_conv_block_activation_derivs -> output;

		// CONVOLUTION BACKWARDS DERIV DATA BUFFERS
		// because residual
		conv_input_deriv = cur_conv_block_activation_derivs -> post_spatial_activated;
		conv_weight_deriv = cur_conv_block_param_derivs -> depth_expansion;
		conv_bias_deriv = cur_conv_block_param_derivs -> bias_depth_expansion;

		prepreAndDoConvolutionDeriv(in_spatial_dim, kern_dim, in_filters, out_filters, stride, batch_size, false,
													conv_input, conv_weight, conv_out_deriv,
													conv_input_deriv, conv_weight_deriv, conv_bias_deriv, true);
		
		printDeviceData("EXPANDED CONV INPUT DERIV", conv_input_deriv, print_size);
		printDeviceData("EXPANDED CONV WEIGHT DERIV", conv_weight_deriv, print_size);
		printDeviceData("EXPANDED CONV BIAS DERIV", conv_bias_deriv, print_size);


		/* 4: Spatial Convolution Activation and Batch Norm Derivs */

		// update the current batch norm layer pointers
		cur_batch_norm_params = cur_conv_block_params -> norm_spatial;
		cur_batch_norm_param_derivs = cur_conv_block_param_derivs -> norm_spatial;

		cur_batch_norm_cache = cur_conv_block_activation -> norm_post_spatial;
		cur_batch_norm_cache_derivs = cur_conv_block_activation_derivs -> norm_post_spatial;

		// fill in details about backprop I/O
		// dL/dBN_Output (given)
		bn_out_layer_deriv = cur_conv_block_activation_derivs -> post_spatial_activated;
		// dL/dBN_Input (to fill in)
		bn_input_deriv = cur_conv_block_activation_derivs -> post_spatial;
		// input to batch norm layer from forward pass
		bn_input = cur_conv_block_activation -> post_spatial;
		// activated output of batch norm layer from forward pass
		bn_activated = cur_conv_block_activation -> post_spatial_activated;
		
		prepareAndDoActivationAndBatchNormDeriv(cur_batch_norm_params, cur_batch_norm_cache, cur_batch_norm_param_derivs, cur_batch_norm_cache_derivs,
																						batch_size, eps, bn_input, bn_activated, bn_out_layer_deriv, bn_input_deriv);

		printDeviceData("SPATIAL ACTIVATION & BATCH NORM DERIV", bn_input_deriv, print_size);

		/* 5: Spatial Convolution Derivs */

		// CONVOLUTION DIMENSIONS
		in_spatial_dim = cur_conv_block_params -> incoming_spatial_dim;
		in_filters = cur_conv_block_params -> reduced_depth;
		out_filters = cur_conv_block_params -> reduced_depth;
		stride = cur_conv_block_params -> stride;
		kern_dim = 3;

		// CONVOLUTION FORWARD DATA
		conv_input = cur_conv_block_activation -> post_reduced_activated;
		conv_weight = cur_conv_block_params -> spatial;
		// from backprop
		conv_out_deriv = cur_conv_block_activation_derivs -> post_spatial;

		// CONVOLUTION BACKWARDS DERIV DATA BUFFERS
		// because residual
		conv_input_deriv = cur_conv_block_activation_derivs -> post_reduced_activated;
		conv_weight_deriv = cur_conv_block_param_derivs -> spatial;
		conv_bias_deriv = cur_conv_block_param_derivs -> bias_spatial;

		prepreAndDoConvolutionDeriv(in_spatial_dim, kern_dim, in_filters, out_filters, stride, batch_size, false,
													conv_input, conv_weight, conv_out_deriv,
													conv_input_deriv, conv_weight_deriv, conv_bias_deriv, true);

		printDeviceData("SPATIAL CONV INPUT DERIV", conv_input_deriv, print_size);
		printDeviceData("SPATIAL CONV WEIGHT DERIV", conv_weight_deriv, print_size);
		printDeviceData("SPATIAL CONV BIAS DERIV", conv_bias_deriv, print_size);


		/* 6: Reduced Convolution Activation and Batch Norm Derivs */

		// update the current batch norm layer pointers
		cur_batch_norm_params = cur_conv_block_params -> norm_depth_reduction;
		cur_batch_norm_param_derivs = cur_conv_block_param_derivs -> norm_depth_reduction;

		cur_batch_norm_cache = cur_conv_block_activation -> norm_post_reduced;
		cur_batch_norm_cache_derivs = cur_conv_block_activation_derivs -> norm_post_reduced;

		// fill in details about backprop I/O
		// dL/dBN_Output (given)
		bn_out_layer_deriv = cur_conv_block_activation_derivs -> post_reduced_activated;
		// dL/dBN_Input (to fill in)
		bn_input_deriv = cur_conv_block_activation_derivs -> post_reduced;
		// input to batch norm layer from forward pass
		bn_input = cur_conv_block_activation -> post_reduced;
		// activated output of batch norm layer from forward pass
		bn_activated = cur_conv_block_activation -> post_reduced_activated;
		
		prepareAndDoActivationAndBatchNormDeriv(cur_batch_norm_params, cur_batch_norm_cache, cur_batch_norm_param_derivs, cur_batch_norm_cache_derivs,
																						batch_size, eps, bn_input, bn_activated, bn_out_layer_deriv, bn_input_deriv);

		printDeviceData("REDUCED ACTIVATION & BATCH NORM DERIV", bn_input_deriv, print_size);

		/* 7: Reduced Convolution Derivs */


		// CONVOLUTION DIMENSIONS
		in_spatial_dim = cur_conv_block_params -> incoming_spatial_dim;
		in_filters = cur_conv_block_params -> incoming_filters;
		out_filters = cur_conv_block_params -> reduced_depth;
		stride = 1;
		kern_dim = 1;

		// CONVOLUTION FORWARD DATA
		conv_input = conv_block_input;
		conv_weight = cur_conv_block_params -> depth_reduction;
		// from backprop
		conv_out_deriv = cur_conv_block_activation_derivs -> post_reduced;

		// CONVOLUTION BACKWARDS DERIV DATA BUFFERS
		// because residual
		conv_input_deriv = conv_block_input_deriv;
		conv_weight_deriv = cur_conv_block_param_derivs -> depth_reduction;
		conv_bias_deriv = cur_conv_block_param_derivs -> bias_depth_reduction;

		prepreAndDoConvolutionDeriv(in_spatial_dim, kern_dim, in_filters, out_filters, stride, batch_size, true,
													conv_input, conv_weight, conv_out_deriv,
													conv_input_deriv, conv_weight_deriv, conv_bias_deriv, true);

		printDeviceData("REDUCED CONV INPUT DERIV", conv_input_deriv, print_size);
		printDeviceData("REDUCED CONV WEIGHT DERIV", conv_weight_deriv, print_size);
		printDeviceData("REDUCED CONV BIAS DERIV", conv_bias_deriv, print_size);

	}


	/* STEP 5: MAX POOL DERIV */

	// maxpool dimensions (used in forward pass)
	int maxpool_kern_dim = dims -> init_maxpool_dim;
	int maxpool_stride = dims -> init_maxpool_stride;
	int maxpool_in_spatial_dim = dims -> input / dims -> init_conv_stride;
	int maxpool_out_spatial_dim = maxpool_in_spatial_dim / maxpool_stride;
	int maxpool_filters = dims -> init_conv_filters;

	// backprop up through the init convblock input has been done. the gradient is at:
	float * maxpool_out_deriv = activation_derivs -> init_convblock_input;

	// getting the max inds cached from forward pass to easily do backprop
	int * max_inds = activations -> max_inds;

	// populating the gradient of input to max_pool located at:
	float * maxpool_inp_deriv = activation_derivs -> init_conv_activated;
	// ensure that gradient has 0's, so that maxPoolDeriv kernel can overwrite only at max ind locations
	int maxpool_inp_size = maxpool_in_spatial_dim * maxpool_in_spatial_dim * maxpool_filters * batch_size;
	hipMemset(maxpool_inp_deriv, 0, maxpool_inp_size * sizeof(float));

	dim3 gridDimMaxPoolDeriv(maxpool_out_spatial_dim, maxpool_out_spatial_dim, maxpool_filters);
	dim3 blockDimMaxPoolDeriv(batch_size);

	// compute max pool deriv (i.e. populate maxpool_inp_deriv)
	maxPoolDeriv <<< gridDimMaxPoolDeriv, blockDimMaxPoolDeriv >>> (max_inds, maxpool_out_deriv, maxpool_kern_dim, maxpool_in_spatial_dim, maxpool_stride, maxpool_filters, batch_size, maxpool_inp_deriv);

	printDeviceData("MAX POOL INPUT ACTIVATION DERIV", maxpool_inp_deriv, print_size);

	/* STEP 6: INIT BATCH NORM & CONV DERIV */

	// BACKPROP OVER THE BATCH NORM OF FIRST CONV LAYER

	// update the current batch norm layer pointers
	cur_batch_norm_params = model_params -> norm_init_conv;
	cur_batch_norm_param_derivs = param_derivs -> norm_init_conv;

	cur_batch_norm_cache = activations -> norm_init_conv;
	cur_batch_norm_cache_derivs = activation_derivs -> norm_init_conv;

	// fill in details about backprop I/O
	// dL/dBN_Output (given)
	bn_out_layer_deriv = activation_derivs -> init_conv_activated;
	// dL/dBN_Input (to fill in)
	bn_input_deriv = activation_derivs -> init_conv_applied;
	// input to batch norm layer from forward pass
	bn_input = activations -> init_conv_applied;
	// activated output of batch norm layer from forward pass
	bn_activated = activations -> init_conv_activated;
		
	prepareAndDoActivationAndBatchNormDeriv(cur_batch_norm_params, cur_batch_norm_cache, cur_batch_norm_param_derivs, cur_batch_norm_cache_derivs,
																						batch_size, eps, bn_input, bn_activated, bn_out_layer_deriv, bn_input_deriv);

	printDeviceData("INIT CONV ACTIVATION & BATCH NORM DERIV", bn_input_deriv, print_size);

	// BACKPROP OVER FIRST CONV LAYER

	// CONVOLUTION DIMENSIONS
	// hardcoded to 3 for the colors
	in_filters = 3;
	out_filters = dims -> init_conv_filters;
	in_spatial_dim = dims -> input;
	stride = dims -> init_conv_stride;
	kern_dim = dims -> init_kernel_dim;

	// CONVOLUTION FORWARD DATA
	conv_input = trainer -> cur_batch -> images;
	conv_weight = model_params -> init_conv_layer;
	// from backprop
	conv_out_deriv = activation_derivs -> init_conv_applied;

	// CONVOLUTION BACKWARDS DERIV DATA BUFFERS
	// because residual
	conv_input_deriv = NULL;
	conv_weight_deriv = param_derivs -> init_conv_layer;
	conv_bias_deriv = param_derivs -> bias_init_conv;

	prepreAndDoConvolutionDeriv(in_spatial_dim, kern_dim, in_filters, out_filters, stride, batch_size, false,
													conv_input, conv_weight, conv_out_deriv,
													conv_input_deriv, conv_weight_deriv, conv_bias_deriv, false);

	printDeviceData("INIT CONV WEIGHT DERIV", conv_weight_deriv, print_size);
	printDeviceData("INIT CONV BIAS DERIV", conv_bias_deriv, print_size);
}	


// doing ADAM optimizer
void update_parameters(Train_ResNet * trainer){
	
	float learning_rate = trainer -> learning_rate;
	float base_mean_decay = trainer -> base_mean_decay;
	float base_var_decay = trainer -> base_var_decay;
	// update the running decays here...
	float cur_mean_decay = trainer -> cur_mean_decay * base_mean_decay;
	float cur_var_decay = trainer -> cur_var_decay * base_mean_decay;
	float eps = trainer -> eps;

	Params * model_params = trainer -> model -> params;
	float ** model_params_locations = model_params -> locations;
	int * param_sizes = model_params -> sizes;
	int n_locations = model_params -> n_locations;

	// values calculated from backprop, will reset these before returning
	Params * current_gradients = trainer -> backprop_buffer -> param_derivs;
	float ** current_gradient_locations = current_gradients -> locations;
	
	// running history values that the optimizer needs, will update these before returning
	Params * prev_grad_means = trainer -> backprop_buffer -> prev_means;
	float ** prev_grad_means_locations = prev_grad_means -> locations;
	Params * prev_grad_vars = trainer -> backprop_buffer -> prev_vars;
	float ** prev_grad_vars_locations = prev_grad_vars -> locations;

	int param_size;
	float *model_location, *grad_location, * mean_location, * var_location;
	
	for (int i = 0; i < n_locations; i++){
		param_size = param_sizes[i];
		model_location = model_params_locations[i];
		grad_location = current_gradient_locations[i];
		mean_location = prev_grad_means_locations[i];
		var_location = prev_grad_vars_locations[i];

		updateMeans <<< ceil((float) param_size / MAX_THREAD_PER_BLOCK), MAX_THREAD_PER_BLOCK >>> (param_size, grad_location, base_mean_decay, mean_location, i);
		updateVars <<< ceil((float) param_size / MAX_THREAD_PER_BLOCK), MAX_THREAD_PER_BLOCK >>> (param_size, grad_location, base_var_decay, var_location, i);
		updateParams <<< ceil((float) param_size / MAX_THREAD_PER_BLOCK), MAX_THREAD_PER_BLOCK >>> (param_size, model_location, mean_location, var_location, learning_rate, cur_mean_decay, cur_var_decay, eps, i);

		hipMemset(grad_location, 0, param_size * sizeof(float));
	}

	// reset images and classes before next hipMemcpy
	size_t batch_size = (size_t) trainer -> batch_size;
	size_t image_size = (size_t) trainer -> cur_batch -> image_size;
	hipMemset(trainer -> cur_batch -> images, 0, batch_size * image_size * sizeof(float));
	hipMemset(trainer -> cur_batch -> correct_classes, 0, batch_size * sizeof(int));
}

void testTranspose(){

	int orig_rows = max(1, rand() % 2048);
	int orig_cols = max(1, rand() % 2048);

	float * origMat_host = (float *) malloc(orig_rows * orig_cols * sizeof(float));
	for (int i = 0; i < orig_rows; i++){
		for (int j = 0; j < orig_cols; j++){
			origMat_host[i * orig_cols + j] = ((float)(rand())/(float)(RAND_MAX));
		}
	}

	float * devOrigMat;
	hipMalloc(&devOrigMat, orig_rows * orig_cols * sizeof(float));
	hipMemcpy(devOrigMat, origMat_host, orig_rows * orig_cols * sizeof(float), hipMemcpyHostToDevice);

	float * devTrans;
	hipMalloc(&devTrans, orig_cols * orig_rows * sizeof(float));

	dim3 gridDimTranspose(ceil((float) orig_rows / TILE_WIDTH), ceil((float) orig_cols / TILE_WIDTH));
	dim3 blockDimTranspose(TILE_WIDTH, TILE_WIDTH);
	transpose <<< gridDimTranspose, blockDimTranspose >>> (devOrigMat, orig_rows, orig_cols, devTrans);

	float *matTrans_host = (float *) malloc(orig_cols * orig_rows * sizeof(float));

	hipMemcpy(matTrans_host, devTrans, orig_cols * orig_rows * sizeof(float), hipMemcpyDeviceToHost);

	hipFree(devOrigMat);
	hipFree(devTrans);

	for (int i = 0; i < orig_cols; i++){
		for (int j = 0; j < orig_rows; j++){
			if (origMat_host[j * orig_cols + i] != matTrans_host[i * orig_rows + j]){
				printf("TRANSPOSE ERROR: @ original row: %d, original col: %d\n", j, i);
			}
		}
	}

	free(origMat_host);
	free(matTrans_host);
}


void testMatMul(){

	int m = max(1, rand() % 512);
	int k = max(1, rand() % 512);
	int n = max(1, rand() % 512);

	float * A_host = (float *) malloc(m * k * sizeof(float));
	float * B_host = (float *) malloc(k * n * sizeof(float));
	float * C_host = (float *) calloc(m * n, sizeof(float));

	for (int i = 0; i < m; i++){
		for (int j = 0; j < k; j++){
			A_host[i * k + j] = ((float)(rand())/(float)(RAND_MAX));
		}
	}

	for (int i = 0; i < k; i++){
		for (int j = 0; j < n; j++){
			B_host[i * n + j] = ((float)(rand())/(float)(RAND_MAX));
		}
	}

	for (int i = 0; i < m; i++){
		for (int j = 0; j < n; j++){
			for (int c = 0; c < k; c++){
				C_host[i * n + j] += A_host[i * k + c] * B_host[c * n + j];
			}
		}
	}

	float * A_dev, *B_dev, *C_dev;
	hipMalloc(&A_dev, m * k * sizeof(float));
	hipMemcpy(A_dev, A_host, m * k * sizeof(float), hipMemcpyHostToDevice);

	hipMalloc(&B_dev, k * n * sizeof(float));
	hipMemcpy(B_dev, B_host, k * n * sizeof(float), hipMemcpyHostToDevice);


	hipMalloc(&C_dev, m * n * sizeof(float));


	dim3 gridDimMatMul(ceil((float) m / TILE_WIDTH), ceil((float) n / TILE_WIDTH));
	dim3 blockDimMatMul(TILE_WIDTH, TILE_WIDTH);

	matMul <<< gridDimMatMul, blockDimMatMul >>> (A_dev, B_dev, m, k, n, C_dev);

	float * C_kern_result = (float *) malloc(m * n * sizeof(float));

	hipMemcpy(C_kern_result, C_dev, m * n * sizeof(float), hipMemcpyDeviceToHost);

	for (int i = 0; i < m; i++){
		for (int j = 0; j < n; j++){
			if (C_kern_result[i * n + j] != C_host[i * n + j]){
				printf("MatMul ERROR: @ row: %d, col: %d\n", j, i);
				printf("CPU Result: %f\n", C_host[i * n + j]);
				printf("GPU Result: %f\n\n", C_kern_result[i * n + j]);
			}
		}
	}

	hipFree(A_dev);
	hipFree(B_dev);
	hipFree(C_dev);

	free(A_host);
	free(B_host);
	free(C_host);
	free(C_kern_result);

}

void testConvolution(int in_spatial_dim, int kern_dim, int in_filters, int out_filters,  int stride, int batch_size, 
																float * input, float * weights, float * biases, float * output){

	printf("\n\n* TESTING THE CONVOLUTION KERNEL *\n\n");
	/* FIRST DO COMPUTATION ON GPU */

	int out_spatial_dim = in_spatial_dim / stride;
	int out_filters_block = min(MAX_THREAD_PER_BLOCK / batch_size, out_filters);
	int out_filters_grid = max(1, (int) ceil((float) out_filters / (float) out_filters_block));

	printf("Conv Params -- Batch Size: %d, In Spatial: %d, Stride: %d, Kern Dim: %d, In Filters: %d, Out Filters %d\n", batch_size, in_spatial_dim, stride, kern_dim, in_filters, out_filters);
	printf("Launch Params -- Out Filters Block: %d, Out Filters Grid: %d\n", out_filters_block, out_filters_grid);
	dim3 gridDimConv(out_spatial_dim, out_spatial_dim, out_filters_grid);
	dim3 blockDimConv(batch_size, out_filters_block);

	printf("Computing Convolution on GPU...\n");

	doConvolution <<< gridDimConv, blockDimConv>>> (input, weights, biases, in_spatial_dim, kern_dim, in_filters, out_filters, stride, batch_size, output);

	hipDeviceSynchronize();

	float * gpu_output_on_cpu = (float *) malloc(batch_size * out_spatial_dim * out_spatial_dim * out_filters * sizeof(float));

	hipMemcpy(gpu_output_on_cpu, output, batch_size * out_spatial_dim * out_spatial_dim * out_filters * sizeof(float), hipMemcpyDeviceToHost);

	/* DO COMPUTATION ON CPU */
	
	// COPYING VALUES FROM GPU TO THE CPU...
	float * input_cpu = (float *) malloc(batch_size * in_spatial_dim * in_spatial_dim * in_filters * sizeof(float));
	hipMemcpy(input_cpu, input, batch_size * in_spatial_dim * in_spatial_dim * in_filters * sizeof(float), hipMemcpyDeviceToHost);

	float * weights_cpu = (float *) malloc(kern_dim * kern_dim * in_filters * out_filters * sizeof(float));
	hipMemcpy(weights_cpu, weights, kern_dim * kern_dim * in_filters * out_filters * sizeof(float), hipMemcpyDeviceToHost);

	float * biases_cpu = (float *) malloc(out_filters * sizeof(float));
	hipMemcpy(biases_cpu, biases, out_filters * sizeof(float), hipMemcpyDeviceToHost);

	float * cpu_output = (float *) malloc(batch_size * out_spatial_dim * out_spatial_dim * out_filters * sizeof(float));

	printf("Computing Convolution on CPU...\n");

	int output_ind, in_spatial_row_start, in_spatial_col_start, in_spatial_row, in_spatial_col, input_ind, kernel_ind;
	int half_kernel_dim = kern_dim / 2;
	int kernel_size = in_filters * kern_dim * kern_dim;
	float in_spatial_val;
	for (int samp = 0; samp < batch_size; samp++){
		for (int out_filt = 0; out_filt < out_filters; out_filt++){
			for (int out_i = 0; out_i < out_spatial_dim; out_i++){
				for (int out_j = 0; out_j < out_spatial_dim; out_j++){
					output_ind = out_spatial_dim * out_spatial_dim * out_filters * samp + out_spatial_dim * out_filters * out_i + out_filters * out_j + out_filt;
					cpu_output[output_ind] = 0;
					in_spatial_row_start = out_i * stride;
					in_spatial_col_start = out_j * stride;
					for (int in_filt = 0; in_filt < in_filters; in_filt++){
						for (int row_offset = -half_kernel_dim; row_offset <= half_kernel_dim; row_offset++){
							for (int col_offset = -half_kernel_dim; col_offset <= half_kernel_dim; col_offset++){
								// compute spatial value
								in_spatial_row = in_spatial_row_start + row_offset;
								in_spatial_col = in_spatial_col_start + col_offset;
								input_ind = in_spatial_dim * in_spatial_dim * in_filters * samp + in_spatial_dim * in_filters * in_spatial_row + in_filters * in_spatial_col + in_filt;
								kernel_ind = kern_dim * in_filters * (row_offset + half_kernel_dim) + in_filters * (col_offset + half_kernel_dim) + in_filt;
								if ((in_spatial_row < 0) || (in_spatial_row >= in_spatial_dim) || (in_spatial_col < 0) || (in_spatial_col >= in_spatial_dim)) {
									in_spatial_val = 0;
								}
								else{
									in_spatial_val = input_cpu[input_ind];
								}
								// multiply with conv weight
								// threadIdx.x specifies the output filter id
								// kernel_ind specifies the (x, y, input_channel)
								cpu_output[output_ind] += weights_cpu[out_filt * kernel_size + kernel_ind] * in_spatial_val;
							}
						}
					}
					cpu_output[output_ind] += biases_cpu[out_filt];
				}
			}
		}
	}

	/* COMPARE RESULTS */
	float gpu_val;
	float cpu_val;
	float eps = 0.0001;
	int err_cnt = 0;
	for (int samp = 0; samp < batch_size; samp++){
		for (int filt = 0; filt < out_filters; filt++){
			for (int i = 0; i < out_spatial_dim; i++){
				for (int j = 0; j < out_spatial_dim; j++){
					output_ind = out_spatial_dim * out_spatial_dim * out_filters * samp + out_spatial_dim * out_filters * i + out_filters * j + filt;
					gpu_val = gpu_output_on_cpu[output_ind];
					cpu_val = cpu_output[output_ind];
					if ( (gpu_val < (cpu_val - eps)) || (gpu_val > (cpu_val + eps)) ){
						printf("ERROR: GPU VALUE DIFFERS FROM CPU\n");
						printf("Occurs at:\nSamp: %d\nFilter: %d\nRow: %d\nCol: %d\n", samp, filt, i, j);
						printf("GPU Value:%f vs. CPU Value:%f\n\n", gpu_val, cpu_val);
						err_cnt++;
					}
					if (err_cnt == 10){
						exit(1);
					}
				}
			}
		}
	}

	/* FREE UP STUFF */

	free(gpu_output_on_cpu);
	free(input_cpu);
	free(weights_cpu);
	free(biases_cpu);
	free(cpu_output);	

}



int main(int argc, char *argv[]) {

	bool debug = false;

	if (debug){
		testMatMul();
		testTranspose();
		return 0;
	}

	int N_CLASSES = 1000;
	
	// GETTING CLASS METADETA
	char * LABEL_FILENAME = (char *) "/mnt/storage/data/vision/imagenet/2012/id_to_label_mapping.txt";
	char * SYNSET_FILENAME = (char *) "/mnt/storage/data/vision/imagenet/2012/id_to_synset_mapping.txt";
	char * COUNTS_FILENAME = (char *) "/mnt/storage/data/vision/imagenet/2012/id_to_img_count_mapping.txt";
	Class_Metadata * class_metadata = populate_class_info(LABEL_FILENAME, SYNSET_FILENAME, COUNTS_FILENAME, N_CLASSES);
	int total_images = 0;
	for (int i = 0; i < N_CLASSES; i++){
		total_images += (class_metadata -> counts)[i];
	}

	// DEFINING MODEL DIMENSIONS
	int INPUT_DIM = 224;
	int INIT_KERNEL_DIM = 7;
	int INIT_CONV_FILTERS = 64;
	int INIT_CONV_STRIDE = 2;
	int INIT_MAXPOOL_DIM = 3;
	int INIT_MAXPOOL_STRIDE = 2;
	int N_CONV_BLOCKS = 16;
	int * IS_BLOCK_SPATIAL_REDUCTION = (int *) calloc(N_CONV_BLOCKS, sizeof(int));
	// transitions between spatial 56 -> 28 -> 14 -> 7
	// transitions between output depth of 256 -> 512 -> 1024 -> 2048
	int FINAL_DEPTH = 2048;
	IS_BLOCK_SPATIAL_REDUCTION[3] = 1;
	IS_BLOCK_SPATIAL_REDUCTION[7] = 1;
	IS_BLOCK_SPATIAL_REDUCTION[13] = 1;
	Dims * dims = init_dimensions(INPUT_DIM, INIT_KERNEL_DIM, INIT_CONV_FILTERS, INIT_CONV_STRIDE, INIT_MAXPOOL_DIM, INIT_MAXPOOL_STRIDE,
									N_CONV_BLOCKS, IS_BLOCK_SPATIAL_REDUCTION, FINAL_DEPTH, N_CLASSES);


	// declaring curandGenerator
	hiprandGenerator_t gen;
	// INITIALIZING RANDOM NUMBER GENERATOR USED TO INIT WEIGHTS
	hiprandStatus_t status_create = hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
	hiprandStatus_t status_set_seed = hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL);

	// INITIALIZING MODEL
	ResNet * model = init_resnet(dims, &gen);


	// INITIALIZING TRAINING

	// Batch Structure (will be modified every iteration of every epoch)
	
	// given when we generated shards...
	int SHARD_N_IMAGES = 32768;

	int BATCH_SIZE = 32;
	// dimensions of INPUT_DIM X INPUT_DIM x 3 color channels
	int IMAGE_SIZE = INPUT_DIM * INPUT_DIM * 3;
	Batch * batch = init_general_batch(BATCH_SIZE, IMAGE_SIZE, INPUT_DIM, SHARD_N_IMAGES);


	// General Training Structure (holds hyperparameters and pointers to structs which have network values)
	float LEARNING_RATE = 0.0001;
	float MEAN_DECAY = 0.9;
	float VAR_DECAY = 0.999;
	float EPS = 0.00000001;
	float N_EPOCHS = 1;

	Train_ResNet * trainer = init_trainer(model, batch, BATCH_SIZE, LEARNING_RATE, MEAN_DECAY, VAR_DECAY, EPS, N_EPOCHS);
	

	/* PERFORM TRAINING */


	int iterations_per_epoch = ceil((float) total_images / BATCH_SIZE);

	float *pred;
	int * correct;
	float epoch_n_wrong, batch_n_wrong;
	float epoch_loss, batch_loss, avg_batch_loss, epoch_accuracy, batch_accuracy, val_pred_correct;
	float total_images_per_epoch = BATCH_SIZE * iterations_per_epoch;

	int PRINT_FREQ = 1;

	hipError_t status;

	for (int epoch = 0; epoch < N_EPOCHS; epoch++){
		epoch_loss = 0;
		epoch_n_wrong = 0;
		for (int iter = 0; iter < iterations_per_epoch; iter++){

			printf("************\n");

			/* LOAD NEW BATCH */
			printf("Loading Batch...\n");
			// values go into trainer -> cur_batch -> [images_cpu|images_float_cpu|images|correct_classes_cpu|correct_classes]
			load_new_batch(class_metadata, trainer -> cur_batch);

			hipDeviceSynchronize();
			status = hipGetLastError();
			//printf("Status after loading batch: %s\n\n", hipGetErrorString(status));
			

			/* DO FORWARD PROP */
			// final predictions go into trainer -> forward_buffer -> [pred|pred_cpu|prediction_label]
			printf("Making Predictions...\n");
			forward_pass(trainer);

			hipDeviceSynchronize();
			status = hipGetLastError();
			//printf("Status after forward pass: %s\n\n", hipGetErrorString(status));
			

			/* RECORD LOSS AND ACCURACY */

			// dimensions of pred: (N_CLASSES, BATCH_SIZE)
			pred = trainer -> forward_buffer -> pred_cpu;
			correct = trainer -> cur_batch -> correct_classes_cpu;
			
			// loss
			batch_loss = 0;
			for (int s = 0; s < BATCH_SIZE; s++){
				batch_loss += -1 * logf(pred[correct[s] * BATCH_SIZE + s]);
			}
			avg_batch_loss = batch_loss / BATCH_SIZE;
			epoch_loss += batch_loss;

			// accuracy
			batch_n_wrong = 0;
			for (int s = 0; s < BATCH_SIZE; s++){
				val_pred_correct = pred[correct[s] * BATCH_SIZE + s];
				for (int c = 0; c < N_CLASSES; c++){
					if ((c != correct[s]) && (pred[c * BATCH_SIZE + s] >= val_pred_correct)){
						batch_n_wrong++;
						break;
					}
				}
			}
			epoch_n_wrong += batch_n_wrong;
			batch_accuracy = 100 * ((float) BATCH_SIZE - batch_n_wrong) / ((float) BATCH_SIZE);

			if (iter % PRINT_FREQ == 0){
				printf("\nEpoch: %d, Batch: %d ----- Avg. Loss: %.4f, Accuracy: %.2f\n\n", epoch, iter, avg_batch_loss, batch_accuracy);
			}

			/* DO BACKPROP */
			printf("Backprop to Compute Derivs...\n");
			backwards_pass(trainer);

			hipDeviceSynchronize();
			status = hipGetLastError();
			//printf("Status after backwards pass: %s\n\n", hipGetErrorString(status));

			/* OPTIMIZE WEIGHTS */
			printf("Applying Optimizer to Update Params...\n\n");
			update_parameters(trainer);

			hipDeviceSynchronize();
			status = hipGetLastError();
			//printf("Status after updating params: %s\n\n", hipGetErrorString(status));

		}

		(trainer -> loss_per_epoch)[epoch] = epoch_loss;
		epoch_accuracy = (total_images_per_epoch - epoch_n_wrong) / total_images_per_epoch;
		(trainer -> accuracy_per_epoch)[epoch] = epoch_accuracy;

	}

}